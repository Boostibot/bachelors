#include "hip/hip_runtime.h"
#define JMAPI __host__ __device__ static inline

#include <>
#include "kernel.h"
#include <cmath>
#include <stdio.h>
#include <assert.h>

#define MOD(x, mod) (((x) % (mod) + (mod)) % (mod))

enum {
    DO_DEBUG = 0,
    MAP_GRAD_PHI = 0,
    MAP_GRAD_T = 1,
    MAP_REACTION = 2,
    MAP_ANISO_FACTOR = 3,
};

__host__ __device__ real_t* map_at(real_t* map, int x, int y, Allen_Cahn_Params params)
{
    int x_mod = MOD(x, params.mesh_size_x);
    int y_mod = MOD(y, params.mesh_size_y);

    return &map[x_mod + y_mod*params.mesh_size_x];
}

__host__ __device__ real_t f0(real_t phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}

__host__ __device__ real_t allen_cahn_reaction_term_1(real_t phi, real_t T, real_t xi, Allen_Cahn_Params params)
{
    real_t mK = 1;
	return (params.a*f0(phi) - params.b*params.beta*xi*(T - params.Tm))*mK;
}

__host__ __device__ real_t allen_cahn_reaction_term_2(real_t phi, real_t T, real_t xi, real_t grad_phi_x, real_t grad_phi_y, Allen_Cahn_Params params)
{
    real_t mK = 1;
	real_t grad_val = hypot(grad_phi_x, grad_phi_y);
	return (params.a*f0(phi) - params.b*params.beta*xi*xi*grad_val*(T - params.Tm))*mK;
}

#define ECHOF(x) printf(#x": %f\n", (x))

#define DO_ANISOTROPY

__device__ real_t g(real_t theta, real_t theta0, real_t m, real_t S)
{
    #ifdef DO_ANISOTROPY
        return 1.0f - S*cosf(m*(theta - theta0));
    #else
        return 1;
    #endif
}

#define PI 3.14159265359
#define TAU (2*PI)

__global__ void allen_cahn_simulate(real_t* Phi_map_next, real_t* T_map_next, real_t* Phi_map, real_t* T_map, Allen_Cahn_Maps maps, Allen_Cahn_Params params, size_t iter)
{
    real_t dx = (real_t) params.L0 / params.mesh_size_x;
    real_t dy = (real_t) params.L0 / params.mesh_size_y;
    real_t mK = dx * dy;

    real_t a = params.a;
    real_t b = params.b;
    real_t alpha = params.alpha;
    real_t beta = params.beta;
    real_t xi = params.xi;
    real_t Tm = params.Tm;
    real_t L = params.L; //Latent heat, not L0 (sym size) ! 
    real_t dt = params.dt;
    real_t S = params.S; //anisotrophy strength
    real_t m = params.m; //anisotrophy frequency (?)
    real_t theta0 = params.theta0;
    bool do_anisotropy = params.do_anisotropy;

    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < params.mesh_size_x; x += blockDim.x * gridDim.x) 
    {
        for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < params.mesh_size_x; y += blockDim.y * gridDim.y) 
        {
            real_t T = *map_at(T_map, x, y, params);
            real_t Phi = *map_at(Phi_map, x, y, params);

            real_t Phi_U = *map_at(Phi_map, x, y + 1, params);
            real_t Phi_D = *map_at(Phi_map, x, y - 1, params);
            real_t Phi_R = *map_at(Phi_map, x + 1, y, params);
            real_t Phi_L = *map_at(Phi_map, x - 1, y, params);

            real_t T_U = *map_at(T_map, x, y + 1, params);
            real_t T_D = *map_at(T_map, x, y - 1, params);
            real_t T_R = *map_at(T_map, x + 1, y, params);
            real_t T_L = *map_at(T_map, x - 1, y, params);

            real_t grad_T_x = dy*(T_R - T_L);
            real_t grad_T_y = dx*(T_U - T_D);

            real_t grad_Phi_x = dy*(Phi_R - Phi_L);
            real_t grad_Phi_y = dx*(Phi_U - Phi_D);

            real_t grad_T_norm = hypotf(grad_T_x, grad_T_y);
            real_t grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);

            real_t g_theta = 1;
            real_t grad_Phi_y_norm = grad_Phi_y / grad_Phi_norm;
            if(do_anisotropy)
            {
                //prevent nans
                if(grad_Phi_norm > 0.0001)
                {
                    real_t theta = asinf(grad_Phi_y_norm);
                    g_theta = 1.0f - S*cosf(m*theta + theta0);
                }
            }

            real_t int_K_laplace_T   = dy/dx*(T_L - 2*T + T_R)     + dx/dy*(T_D - 2*T + T_U);
            real_t int_K_laplace_Phi = dy/dx*(Phi_L - 2*Phi + Phi_R) + dx/dy*(Phi_D - 2*Phi + Phi_U);
            real_t int_K_f = g_theta*a*mK*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm/2;

            real_t int_K_dt_Phi = g_theta/alpha*int_K_laplace_Phi + 1/(xi*xi * alpha)*int_K_f;
            real_t int_K_dt_T = int_K_laplace_T + L*int_K_dt_Phi;

            real_t dt_Phi = 1/mK*int_K_dt_Phi;
            real_t dt_T = 1/mK*int_K_dt_T;

            real_t Phi_next = Phi + dt*dt_Phi;
            real_t T_next = T + dt*dt_T;
        
            if(DO_DEBUG)
            {
                if(maps.debug_request[MAP_GRAD_PHI])
                    *map_at(maps.debug_maps[MAP_GRAD_PHI], x, y, params) = hypotf(Phi_R - Phi_L, Phi_U - Phi_D);

                if(maps.debug_request[MAP_GRAD_T])
                    *map_at(maps.debug_maps[MAP_GRAD_T], x, y, params) = hypotf(T_R - T_L, T_U - T_D);

                if(maps.debug_request[MAP_REACTION])
                    *map_at(maps.debug_maps[MAP_REACTION], x, y, params) = int_K_f / mK;

                if(maps.debug_request[MAP_ANISO_FACTOR])
                    *map_at(maps.debug_maps[MAP_ANISO_FACTOR], x, y, params) = g_theta;
            }

            *map_at(Phi_map_next, x, y, params) = Phi_next;
            *map_at(T_map_next, x, y, params) = T_next;
        }
    }
}

#define CUDA_ERR_AND(err) (err) != hipSuccess ? (err) :

extern "C" hipError_t kernel_step(Allen_Cahn_Maps* maps, Allen_Cahn_Params params, int device_processor_count, size_t iter)
{
    real_t* Phi_map_next = maps->Phi[(iter + 1) % 2];
    real_t* Phi_map = maps->Phi[(iter) % 2];
    
    real_t* T_map_next = maps->T[(iter + 1) % 2];
    real_t* T_map = maps->T[(iter) % 2];

    dim3 bs(64, 1);
    dim3 grid(device_processor_count, 1);
    allen_cahn_simulate<<<grid, bs>>>(Phi_map_next, T_map_next, Phi_map, T_map, *maps, params, iter);

    memset(maps->debug_names, 0, sizeof maps->debug_names); 
    #define ASSIGN_MAP_NAME(maps, name) \
        memcpy((maps)->debug_names[(name)], #name, sizeof(#name));
    ASSIGN_MAP_NAME(maps, MAP_GRAD_PHI);
    ASSIGN_MAP_NAME(maps, MAP_GRAD_T);
    ASSIGN_MAP_NAME(maps, MAP_REACTION);
    ASSIGN_MAP_NAME(maps, MAP_ANISO_FACTOR);

    hipError_t out = hipSuccess;
    out = CUDA_ERR_AND(out) hipGetLastError();
    out = CUDA_ERR_AND(out) hipDeviceSynchronize();
    return out;
}