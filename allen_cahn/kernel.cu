#include "hip/hip_runtime.h"
#define SHARED __host__ __device__

#include <>
#include "kernel.h"
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <stdint.h>
#include <hip/hip_runtime.h>


#define MOD(x, mod) (((x) % (mod) + (mod)) % (mod))
#define MAX(a, b)   ((a) > (b) ? (a) : (b))
#define MIN(a, b)   ((a) < (b) ? (a) : (b))
#define PI          ((Real) 3.14159265359)
#define TAU         (2*PI)
#define CUDA_ERR_AND(err) (err) != hipSuccess ? (err) :
#define ECHOF(x)    printf(#x ": " REAL_FMT "\n", (x))

#include <hip/hip_runtime.h>
#include <stdarg.h>
static bool _test_cuda_(hipError_t error, const char* expression, int line, const char* file, const char* format, ...)
{
    if(error != hipSuccess)
    {
        printf("CUDA_TEST(%s) failed with %s! %s:%i\n", expression, hipGetErrorString(error), file, line);
        if(format != NULL && strlen(format) != 0)
        {
            va_list args;
            va_start(args, format);
            vprintf(format, args);
            va_end(args);
            printf("\n");
        }
    }
    fflush(stdout);
    return error == hipSuccess;
}

static void _test(const char* expression, int line, const char* file, const char* format, ...)
{
    if(format != NULL && strlen(format) != 0)
    {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
        printf("\n");
    }
    else
    {
        printf("TEST(%s) failed! %s:%i\n", expression, file, line);
    }

    fflush(stdout);
}

#define CUDA_TEST(status, ...) (_test_cuda_((status), #status,  __LINE__, __FILE__, "" __VA_ARGS__) ? (void) 0 : abort())
#define TEST(x, ...)           ((x) ? (void) 0 : (_test(#x,  __LINE__, __FILE__, "" __VA_ARGS__), abort()))

#ifdef NDEBUG
    #define CUDA_DEBUG_TEST(status, ...) (0 ? printf("" __VA_ARGS__) : (status))
#else
    #define CUDA_DEBUG_TEST(status, ...) CUDA_TEST(status, __VA_ARGS__)
#endif

struct Cuda_Info {
    int device_id;
    hipDeviceProp_t prop;
};

Cuda_Info cuda_one_time_setup()
{
    static bool was_setup = false;
    static Cuda_Info info = {0};

    if(was_setup == false)
    {
        enum {MAX_DEVICES = 100};
        hipDeviceProp_t devices[MAX_DEVICES] = {0};
        double scores[MAX_DEVICES] = {0};
        double peak_memory[MAX_DEVICES] = {0};
        
        int nDevices = 0;
        CUDA_TEST(hipGetDeviceCount(&nDevices));
        if(nDevices > MAX_DEVICES)
        {
            assert(false && "wow this should probably not happen!");
            nDevices = MAX_DEVICES;
        }
        TEST(nDevices > 0, "Didnt find any CUDA capable devices. Stopping.");

        for (int i = 0; i < nDevices; i++) 
            CUDA_DEBUG_TEST(hipGetDeviceProperties(&devices[i], i));

        //compute maximum in each tracked category to
        // be able to properly select the best device for
        // the job!
        hipDeviceProp_t max_prop = {0};
        double max_peak_memory = 0;
        for (int i = 0; i < nDevices; i++) 
        {
            hipDeviceProp_t prop = devices[i];
            max_prop.warpSize = MAX(max_prop.warpSize, prop.warpSize);
            max_prop.multiProcessorCount = MAX(max_prop.multiProcessorCount, prop.multiProcessorCount);
            max_prop.concurrentKernels = MAX(max_prop.concurrentKernels, prop.concurrentKernels);
            max_prop.memoryClockRate = MAX(max_prop.memoryClockRate, prop.memoryClockRate);
            max_prop.memoryBusWidth = MAX(max_prop.memoryBusWidth, prop.memoryBusWidth);
            max_prop.totalGlobalMem = MAX(max_prop.totalGlobalMem, prop.totalGlobalMem);
            max_prop.sharedMemPerBlock = MAX(max_prop.sharedMemPerBlock, prop.sharedMemPerBlock);
            peak_memory[i] = 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6;

            max_peak_memory = MAX(max_peak_memory, peak_memory[i]);
        }

        double max_score = 0;
        int max_score_i = 0;
        for (int i = 0; i < nDevices; i++) 
        {
            hipDeviceProp_t prop = devices[i];
            double score = 0
                + 0.40 * prop.warpSize/max_prop.warpSize
                + 0.40 * prop.multiProcessorCount/max_prop.multiProcessorCount
                + 0.05 * prop.concurrentKernels/max_prop.concurrentKernels
                + 0.05 * peak_memory[i]/max_peak_memory
                + 0.05 * prop.totalGlobalMem/max_prop.totalGlobalMem
                + 0.05 * prop.sharedMemPerBlock/max_prop.sharedMemPerBlock
                ;

            scores[i] = score;
            if(max_score < score)
            {
                max_score = score;
                max_score_i = i;
            }
        }
        hipDeviceProp_t selected = devices[max_score_i];
        info.prop = selected;
        info.device_id = max_score_i;
        was_setup = true;
        CUDA_TEST(hipSetDevice(info.device_id));

        printf("Listing devices below (%d):\n", nDevices);
        for (int i = 0; i < nDevices; i++)
            printf("%i > %s (score: %lf) %s\n", i, devices[i].name, scores[i], i == max_score_i ? "[selected]" : "");

        printf("Selected %s:\n", selected.name);
        printf("  Multi Processor count: %i\n", selected.multiProcessorCount);
        printf("  Warp-size: %d\n", selected.warpSize);
        printf("  Memory Clock Rate (MHz): %d\n", selected.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n", selected.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n", peak_memory[max_score_i]);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(selected.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(selected.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", selected.minor, selected.major);
        printf("  Concurrent kernels: %s\n", selected.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",selected.deviceOverlap ? "yes" : "no");
    }

    return info;
}

enum {
    REALLOC_COPY = 1,
    REALLOC_ZERO = 2,
};

typedef struct Memory_Format {
    const char* unit;
    size_t unit_value;
    double fraction;

    int whole;
    int remainder;
} Memory_Format;

Memory_Format get_memory_format(size_t bytes)
{
    size_t B  = (size_t) 1;
    size_t KB = (size_t) 1024;
    size_t MB = (size_t) 1024*1024;
    size_t GB = (size_t) 1024*1024*1024;
    size_t TB = (size_t) 1024*1024*1024*1024;

    Memory_Format out = {0};
    out.unit = "";
    out.unit_value = 1;
    if(bytes >= TB)
    {
        out.unit = "TB";
        out.unit_value = TB;
    }
    else if(bytes >= GB)
    {
        out.unit = "GB";
        out.unit_value = GB;
    }
    else if(bytes >= MB)
    {
        out.unit = "MB";
        out.unit_value = MB;
    }
    else if(bytes >= KB)
    {
        out.unit = "KB";
        out.unit_value = KB;
    }
    else
    {
        out.unit = "B";
        out.unit_value = B;
    }

    out.fraction = (double) bytes / (double) out.unit_value;
    out.whole = (int) (bytes / out.unit_value);
    out.remainder = (int) (bytes / out.unit_value);

    return out;
}

#define MEMORY_FMT "%.2lf%s"
#define MEMORY_PRINT(bytes) get_memory_format((bytes)).fraction, get_memory_format((bytes)).unit

void* _cuda_realloc(void* old_ptr, size_t new_size, size_t old_size, int flags, const char* file, int line)
{
    printf("CUDA realloc " MEMORY_FMT "-> " MEMORY_FMT " %s:%i\n",
            MEMORY_PRINT(old_size), 
            MEMORY_PRINT(new_size),
            file, line);

    static int64_t used_bytes = 0;
    void* new_ptr = NULL;
    if(new_size != 0)
    {
        Cuda_Info info = cuda_one_time_setup();
        CUDA_TEST(hipMalloc(&new_ptr, new_size), 
            "Out of CUDA memory! Requested " MEMORY_FMT ". Using " MEMORY_FMT " / " MEMORY_FMT ". %s:%i", 
            MEMORY_PRINT(new_size), 
            MEMORY_PRINT(used_bytes), 
            MEMORY_PRINT(info.prop.totalGlobalMem),
            file, line);

        size_t min_size = MIN(old_size, new_size);
        if((flags & REALLOC_ZERO) && !(flags & REALLOC_COPY))
            CUDA_DEBUG_TEST(hipMemset(new_ptr, 0, new_size));
        else
        {
            if(flags & REALLOC_COPY)
                CUDA_DEBUG_TEST(hipMemcpy(new_ptr, old_ptr, min_size, hipMemcpyDeviceToDevice));
            if(flags & REALLOC_ZERO)
                CUDA_DEBUG_TEST(hipMemset((uint8_t*) new_ptr + min_size, 0, new_size - min_size));
        }
    }


    CUDA_DEBUG_TEST(hipFree(old_ptr), 
        "Invalid pointer passed to cuda_realloc! %s:%i", file, line);

    used_bytes += (int64_t) new_size - (int64_t) old_size;
    assert(used_bytes >= 0);
    return new_ptr;
}

void _cuda_realloc_in_place(void** ptr_ptr, size_t new_size, size_t old_size, int flags, const char* file, int line)
{
    *ptr_ptr = _cuda_realloc(*ptr_ptr, new_size, old_size, flags, file, line);
}

#define cuda_realloc(old_ptr, new_size, old_size, flags)          _cuda_realloc(old_ptr, new_size, old_size, flags, __FILE__, __LINE__)
#define cuda_realloc_in_place(ptr_ptr, new_size, old_size, flags) _cuda_realloc_in_place(ptr_ptr, new_size, old_size, flags, __FILE__, __LINE__)

SHARED Real* at_mod(Real* map, int x, int y, int n, int m)
{
    int x_mod = MOD(x, m);
    int y_mod = MOD(y, n);

    return &map[x_mod + y_mod*m];
}

SHARED Real f0(Real phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}


int explicit_solver_resize(Explicit_Solver* solver, int n, int m)
{
    size_t N = (size_t)m*(size_t)n;
    size_t N_old = (size_t)solver->m*(size_t)solver->n;
    if(solver->m != m || solver->n != n)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real* debug_maps = (Real*) (void*) &solver->debug_maps;
        for(int i = 0; i < sizeof(solver->debug_maps) / sizeof(Real); i++)
            cuda_realloc_in_place((void**) &debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->m = m;
        solver->n = n;
    }

    return EXPLICIT_SOLVER_REQUIRED_HISTORY;
}

void explicit_state_resize(Explicit_State* state, int n, int m)
{
    size_t N = (size_t)m*(size_t)n;
    size_t N_old = (size_t)state->m*(size_t)state->n;
    if(state->m != m || state->n != n)
    {
        cuda_realloc_in_place((void**) &state->F, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &state->U, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        state->m = m;
        state->n = n;
    }
}





enum {
    FLAG_DO_DEBUG = 1,
    FLAG_DO_ANISOTROPHY = 2,
};

template <unsigned FLAGS>
__global__ void allen_cahn_simulate(Real* Phi_map_next, Real* T_map_next, Real* Phi_map, Real* T_map, const Explicit_Solver expli, const Allen_Cahn_Params params, const size_t iter)
{
    Real dx = (Real) params.L0 / params.m;
    Real dy = (Real) params.L0 / params.n;
    Real mK = dx * dy;

    Real a = params.a;
    Real b = params.b;
    Real alpha = params.alpha;
    Real beta = params.beta;
    Real xi = params.xi;
    Real Tm = params.Tm;
    Real L = params.L; //Latent heat, not L0 (sym size) ! 
    Real dt = params.dt;
    Real S = params.S; //anisotrophy strength
    Real m0 = params.m0; //anisotrophy frequency (?)
    Real theta0 = params.theta0;
    int n = expli.n;
    int m = expli.m;

    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < params.m; x += blockDim.x * gridDim.x) 
    {
        for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < params.m; y += blockDim.y * gridDim.y) 
        {
            Real T = T_map[x + y*m];
            Real Phi = Phi_map[x + y*m];

            Real Phi_U = *at_mod(Phi_map, x, y + 1, n, m);
            Real Phi_D = *at_mod(Phi_map, x, y - 1, n, m);
            Real Phi_R = *at_mod(Phi_map, x + 1, y, n, m);
            Real Phi_L = *at_mod(Phi_map, x - 1, y, n, m);

            Real T_U = *at_mod(T_map, x, y + 1, n, m);
            Real T_D = *at_mod(T_map, x, y - 1, n, m);
            Real T_R = *at_mod(T_map, x + 1, y, n, m);
            Real T_L = *at_mod(T_map, x - 1, y, n, m);

            Real grad_T_x = dy*(T_R - T_L);
            Real grad_T_y = dx*(T_U - T_D);

            Real grad_Phi_x = dy*(Phi_R - Phi_L);
            Real grad_Phi_y = dx*(Phi_U - Phi_D);

            Real grad_T_norm = hypotf(grad_T_x, grad_T_y);
            Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);

            Real g_theta = 1;
            if constexpr(FLAGS & FLAG_DO_ANISOTROPHY)
            {
                //prevent nans
                // if(grad_Phi_norm > 0.0001)
                {

                    Real theta = atan2(grad_Phi_y, grad_Phi_x);
                    // Real grad_Phi_y_norm = grad_Phi_y / grad_Phi_norm;
                    // Real theta = asinf(grad_Phi_y_norm);
                    g_theta = 1.0f - S*cosf(m0*theta + theta0);
                }
            }

            Real int_K_laplace_T   = dy/dx*(T_L - 2*T + T_R)       + dx/dy*(T_D - 2*T + T_U);
            Real int_K_laplace_Phi = dy/dx*(Phi_L - 2*Phi + Phi_R) + dx/dy*(Phi_D - 2*Phi + Phi_U);
            Real int_K_f = g_theta*a*mK*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm/2;

            Real int_K_dt_Phi = g_theta/alpha*int_K_laplace_Phi + 1/(xi*xi * alpha)*int_K_f;
            Real int_K_dt_T = int_K_laplace_T + L*int_K_dt_Phi;

            Real dt_Phi = 1/mK*int_K_dt_Phi;
            Real dt_T = 1/mK*int_K_dt_T;

            Real Phi_next = Phi + dt*dt_Phi;
            Real T_next = T + dt*dt_T;
        
            if constexpr(FLAGS & FLAG_DO_DEBUG)
            {
                *at_mod(expli.debug_maps.grad_phi, x, y, n, m) = hypotf(Phi_R - Phi_L, Phi_U - Phi_D);
                *at_mod(expli.debug_maps.grad_T, x, y, n, m) = hypotf(T_R - T_L, T_U - T_D);
                *at_mod(expli.debug_maps.reaction, x, y, n, m) = int_K_f / mK;
                *at_mod(expli.debug_maps.aniso_factor, x, y, n, m) = g_theta;

                #if 0
                if(0)
                {
                    //@TODO: calculate properly!
                    Real* T_map_prev = expli.U[MOD(iter - 1, ALLEN_CAHN_HISTORY)];
                    Real* Phi_map_prev = expli.F[MOD(iter - 1, ALLEN_CAHN_HISTORY)];

                    Real T_prev = *at_mod(T_map_prev, x, y, n, m);
                    Real Phi_prev = *at_mod(Phi_map_prev, x, y, n, m);

                    Real dt_Phi_prev = (Phi - Phi_prev) / dt;
                    Real r_Phi = (dt_Phi_prev - dt_Phi);

                    *at_mod(expli.debug_maps.step_residual, x, y, params) = abs(r_Phi);
                }
                #endif
            }

            Phi_map_next[x + y*m] = Phi_next;
            T_map_next[x + y*m] = T_next;
        }
    }
}

extern "C" void explicit_solver_step(Explicit_Solver* solver, Explicit_State state, Explicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Cuda_Info info = cuda_one_time_setup();
    dim3 bs(64, 1);
    dim3 grid(info.prop.multiProcessorCount, 1);

    Real* Phi_next = next_state.F;
    Real* Phi = state.F;
    
    Real* T_next = next_state.U;
    Real* T = state.U;

    if(params.do_anisotropy && do_debug)
        allen_cahn_simulate<FLAG_DO_ANISOTROPHY | FLAG_DO_DEBUG><<<grid, bs>>>(Phi_next, T_next, Phi, T, *solver, params, iter);
    if(params.do_anisotropy && !do_debug)
        allen_cahn_simulate<FLAG_DO_ANISOTROPHY><<<grid, bs>>>(Phi_next, T_next, Phi, T, *solver, params, iter);
    if(!params.do_anisotropy && do_debug)
        allen_cahn_simulate<FLAG_DO_DEBUG><<<grid, bs>>>(Phi_next, T_next, Phi, T, *solver, params, iter);
    if(!params.do_anisotropy && !do_debug)
        allen_cahn_simulate<0><<<grid, bs>>>(Phi_next, T_next, Phi, T, *solver, params, iter);

    CUDA_DEBUG_TEST(hipGetLastError());
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
}


void explicit_solver_get_maps(Explicit_Solver* solver, Explicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * map_count);

    #define ASSIGN_MAP_NAMED(var_ptr, var_name) \
        if(__map_i < map_count) \
        { \
            maps[__map_i].data = var_ptr; \
            maps[__map_i].name = var_name; \
            maps[__map_i].m = solver->m; \
            maps[__map_i].n = solver->n; \
            __map_i += 1; \
        }\

    #define ASSIGN_MAP(var_ptr) ASSIGN_MAP_NAMED(var_ptr, #var_ptr) 

    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");
    ASSIGN_MAP_NAMED(solver->debug_maps.aniso_factor, "aniso_factor");
    ASSIGN_MAP_NAMED(solver->debug_maps.reaction, "reaction");
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residual, "step_residual");
}


template <typename Function>
__global__ void _kernel_cuda_for_each(int from, int item_count, Function func)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < item_count; i += blockDim.x * gridDim.x) 
        func(from + i);
}

template <typename Function>
void cuda_for(int from, int to, Function func)
{
    Cuda_Info info = cuda_one_time_setup();
    dim3 bs(64, 1);
    dim3 grid(info.prop.multiProcessorCount, 1);

    _kernel_cuda_for_each<<<grid, bs>>>(from, to-from, (Function&&) func);
    CUDA_DEBUG_TEST(hipGetLastError());
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

template <typename Function>
__global__ void _kernel_cuda_for_each_2D(int from_x, int x_size, int from_y, int y_size, Function func)
{
    //@TODO: Whats the optimal loop order? First x or y?
    for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < y_size; y += blockDim.y * gridDim.y) 
        for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < x_size; x += blockDim.x * gridDim.x) 
            func(x + from_x, y + from_y);
}

template <typename Function>
void cuda_for_2D(int from_x, int from_y, int to_x, int to_y, Function func)
{
    Cuda_Info info = cuda_one_time_setup();
    dim3 bs(64, 1);
    dim3 grid(info.prop.multiProcessorCount, 1);
    _kernel_cuda_for_each_2D<<<grid, bs>>>(from_x, to_x-from_x, from_y, to_y-from_y, (Function&&) func);
    CUDA_DEBUG_TEST(hipGetLastError());
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

//Will hand write my own version later. For now we trust in thrust *cymbal*
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>

Real vector_dot_product(const Real *a, const Real *b, int n)
{
  // wrap raw pointers to device memory with device_ptr
  thrust::device_ptr<const Real> d_a(a);
  thrust::device_ptr<const Real> d_b(b);

  // inner_product implements a mathematical dot product
  return thrust::inner_product(d_a, d_a + n, d_b, 0.0);
}

struct Cross_Matrix_Static {
    Real C;
    Real U;
    Real D;
    Real L;
    Real R;
};

struct Cross_Matrix {
    Real* C;
    Real* U;
    Real* D;
    Real* L;
    Real* R;
};

struct Cross_Matrix_Anisotrophy {
    Real* scale;
    Real X;
    Real Y;
    Real C_minus_one;
};

void* cross_matrix_vector_alloced(Real* vector, int n, int m)
{
    if(vector == NULL)
        return NULL;
    (void) n;
    return vector - m;
}

Real* cross_matrix_vector_padded(void* alloced, int n, int m)
{
    if(alloced == NULL)
        return (Real*) NULL;
    (void) n;
    return (Real*) alloced + m;
}

void cross_matrix_vector_pad(Real* vector, int n, int m)
{
    CUDA_DEBUG_TEST(hipMemset(vector - m, 0, sizeof(Real)*m));
    CUDA_DEBUG_TEST(hipMemset(vector + n*m, 0, sizeof(Real)*m));
}

Real* cross_matrix_vector_realloc(Real* vector, int n, int m, int old_n, int old_m)
{
    int new_size = 2*m + n*m;
    int old_size = 2*old_m + old_n*old_m;
    void* old = cross_matrix_vector_alloced(vector, n, m);
    void* new_ = cuda_realloc(old, (size_t) new_size * sizeof(Real), (size_t) old_size * sizeof(Real), REALLOC_ZERO);
    return cross_matrix_vector_padded(new_, n, m);
}

typedef struct Conjugate_Gardient_Params {
    Real epsilon;
    Real tolerance;
    int max_iters;
    bool padded;
} Conjugate_Gardient_Params;

typedef struct Conjugate_Gardient_Convergence {
    Real error;
    int iters;
    bool converged;
} Conjugate_Gardient_Convergence;

void cross_matrix_static_multiply_padded(Real* out, const Cross_Matrix_Static* _A, const Real* x, int n, int m)
{
    Cross_Matrix_Static A = * (Cross_Matrix_Static*)_A;
    cuda_for(0, m*n, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C;
        //@NOTE: No edge logic! We require explicit (m) padding to be added on both sides of x!
        val += x[i+1]*A.R;
        val += x[i-1]*A.L;
        val += x[i+m]*A.U;
        val += x[i-m]*A.D;

        out[i] = val;
    });
}

void cross_matrix_static_multiply_not_padded(Real* out, const Cross_Matrix_Static* _A, const Real* x, int n, int m)
{
    Cross_Matrix_Static A = * (Cross_Matrix_Static*)_A;
    int N = m*n;
    cuda_for(0, N, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C;
        if(i+1 < N)  val += x[i+1]*A.R;
        if(i-1 >= 0) val += x[i-1]*A.L;
        if(i+m < N)  val += x[i+m]*A.U;
        if(i-m >= 0) val += x[i-m]*A.D;

        out[i] = val;
    });
}

void cross_matrix_multiply_padded(Real* out, const Cross_Matrix* _A, const Real* x, int n, int m)
{
    Cross_Matrix A = * (Cross_Matrix*)_A;
    cuda_for(0, m*n, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C[i];
        val += x[i+1]*A.R[i];
        val += x[i-1]*A.L[i];
        val += x[i+m]*A.U[i];
        val += x[i-m]*A.D[i];

        out[i] = val;
    });
}

void cross_matrix_multiply_not_padded(Real* out, const Cross_Matrix* _A, const Real* x, int n, int m)
{
    Cross_Matrix A = * (Cross_Matrix*)_A;
    int N = m*n;
    cuda_for(0, N, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C[i];
        if(i+1 < N)  val += x[i+1]*A.R[i];
        if(i-1 >= 0) val += x[i-1]*A.L[i];
        if(i+m < N)  val += x[i+m]*A.U[i];
        if(i-m >= 0) val += x[i-m]*A.D[i];

        out[i] = val;
    });
}

void cross_matrix_aniso_multiply_padded(Real* out, const Cross_Matrix_Anisotrophy* _A, const Real* x, int n, int m)
{
    Cross_Matrix_Anisotrophy A = * (Cross_Matrix_Anisotrophy*)_A;
    cuda_for(0, m*n, [=]SHARED(int i){
        Real s = A.scale[i];
        Real X = A.X*s;
        Real Y = A.Y*s;
        Real C = 1 + A.C_minus_one*s;

        Real val = 0;
        val += x[i]*C;
        val += x[i+1]*X;
        val += x[i-1]*X;
        val += x[i+m]*Y;
        val += x[i-m]*Y;

        out[i] = val;
    });
}

void cross_matrix_aniso_multiply_not_padded(Real* out, const Cross_Matrix_Anisotrophy* _A, const Real* x, int n, int m)
{
    Cross_Matrix_Anisotrophy A = * (Cross_Matrix_Anisotrophy*)_A;
    int N = m*n;
    cuda_for(0, N, [=]SHARED(int i){
        Real s = A.scale[i];
        Real X = A.X*s;
        Real Y = A.Y*s;
        Real C = 1 + A.C_minus_one*s;

        Real val = 0;
        val += x[i]*C;
        if(i+1 < N)  val += x[i+1]*X;
        if(i-1 >= 0) val += x[i-1]*X;
        if(i+m < N)  val += x[i+m]*Y;
        if(i-m >= 0) val += x[i-m]*Y;

        out[i] = val;
    });
}


typedef void(*Matrix_Mul_Func)(Real* out, const void* A, const Real* x, int n, int m);

Conjugate_Gardient_Convergence cross_matrix_conjugate_gradient_solve(const void* A, Real* x, const Real* b, int n, int m, void* matrix_mul, const Conjugate_Gardient_Params* params_or_null)
{
    Conjugate_Gardient_Convergence out = {0};
    Conjugate_Gardient_Params params = {0};
    params.epsilon = (Real) 1.0e-10;
    params.tolerance = (Real) 1.0e-5;
    params.max_iters = 10;
    params.padded = false;
    if(params_or_null)
        params = *params_or_null;

    int N = m*n;

    //@NOTE: Evil programmer doing evil programming practices
    static int static_n = 0;
    static int static_m = 0;
    static Real* _r = NULL;
    static Real* _p = NULL;
    static Real* _Ap = NULL;
    if(static_n < n || static_m < m)
    {
        _r = cross_matrix_vector_realloc(_r, n, m, static_n, static_m);
        _p = cross_matrix_vector_realloc(_p, n, m, static_n, static_m);
        _Ap = cross_matrix_vector_realloc(_Ap, n, m, static_n, static_m);

        static_n = n;
        static_m = m;
    }
   
    //NVCC seams to struggle with statics in device code
    // (is probably passing them by reference or something)
    Real* r = _r;
    Real* p = _p;
    Real* Ap = _Ap;
    cross_matrix_vector_pad(p, n, m);

    Matrix_Mul_Func matrix_mul_ = (Matrix_Mul_Func) (void*) matrix_mul;

    CUDA_DEBUG_TEST(hipMemset(x, 0, sizeof(Real)*N));
    CUDA_DEBUG_TEST(hipMemcpy(r, b, sizeof(Real)*N, hipMemcpyDeviceToDevice));
    CUDA_DEBUG_TEST(hipMemcpy(p, b, sizeof(Real)*N, hipMemcpyDeviceToDevice));
    
    Real r_dot_r = vector_dot_product(r, r, N);
    int iter = 0;
    for(; iter < params.max_iters; iter++)
    {
        matrix_mul_(Ap, A, p, n, m);
        
        Real p_dot_Ap = vector_dot_product(p, Ap, N);
        Real alpha = r_dot_r / MAX(p_dot_Ap, params.epsilon);
        
        cuda_for(0, N, [=]SHARED(int i){
            x[i] = x[i] + alpha*p[i];
            r[i] = r[i] - alpha*Ap[i];
        });

        Real r_dot_r_new = vector_dot_product(r, r, N);
        if(r_dot_r_new/N < params.tolerance*params.tolerance)
        {
            r_dot_r = r_dot_r_new;
            break;
        }

        Real beta = r_dot_r_new / MAX(r_dot_r, params.epsilon);
        cuda_for(0, N, [=]SHARED(int i){
            p[i] = r[i] + beta*p[i]; 
        });

        r_dot_r = r_dot_r_new;
    }

    out.iters = iter;
    out.converged = iter != params.max_iters;
    out.error = sqrt(r_dot_r/N);
    return out;
} 

void matrix_multiply(Real* output, const Real* A, const Real* B, int A_height, int A_width, int B_height, int B_width)
{
    assert(A_width == B_height);
    for(int y = 0; y < A_height; y++)
    {
        for(int x = 0; x < B_width; x++)
        {
            Real val = 0;
            for(int k = 0; k < A_width; k++)
                val += A[k + y*A_width]*B[x + k*B_width];

            output[x + y*B_width] = val;
        }
    }
}

Real vector_get_dist_norm(const Real* a, const Real* b, int N)
{
    static Real* temp = NULL;
    static int temp_size = 0;
    if(temp_size < N)
    {
        cuda_realloc_in_place((void**) &temp, N*sizeof(Real), temp_size*sizeof(Real), 0);
        temp_size = N;
    }

    Real* t = temp; //Needed for lambda to work (they capture statics differently)
    cuda_for(0, N, [=]SHARED(int i){
        t[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_dot_product(temp, temp, N);
    Real error = sqrt(temp_dot_temp/N);
    return error;
}

bool vector_is_near(const Real* a, const Real* b, Real epsilon, int N)
{
    return vector_get_dist_norm(a, b, N) < epsilon;
}

int semi_implicit_solver_resize(Semi_Implicit_Solver* solver, int n, int m)
{
    if(solver->m != m || solver->n != n)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real** debug_maps = (Real**) (void*) &solver->debug_maps;
        for(int i = 0; i < sizeof(solver->debug_maps) / sizeof(Real); i++)
            debug_maps[i] = cross_matrix_vector_realloc(debug_maps[i], n, m, solver->n, solver->m);

        Real** maps = (Real**) (void*) &solver->maps;
        for(int i = 0; i < sizeof(solver->maps) / sizeof(Real); i++)
            maps[i] = cross_matrix_vector_realloc(maps[i], n, m, solver->n, solver->m);

        solver->m = m;
        solver->n = n;
    }

    return SEMI_IMPLICIT_SOLVER_REQUIRED_HISTORY;
}

void semi_implicit_state_resize(Semi_Implicit_State* state, int n, int m)
{
    if(state->m != m || state->n != n)
    {
        state->F = cross_matrix_vector_realloc(state->F, n, m, state->n, state->m);
        state->U = cross_matrix_vector_realloc(state->U, n, m, state->n, state->m);

        state->m = m;
        state->n = n;
    }
}

extern "C" void semi_implicit_solver_step(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->m;
    Real dy = (Real) params.L0 / solver->n;

    int m = solver->m;
    int n = solver->n;
    int N = m*n;

    Real mK = dx * dy;
    Real a = params.a;
    Real b = params.b;
    Real alpha = params.alpha;
    Real beta = params.beta;
    Real xi = params.xi;
    Real Tm = params.Tm;
    Real L = params.L; 
    Real dt = params.dt;
    Real S = params.S; 
    Real m0 = params.m0; 
    Real theta0 = params.theta0;
    
    Real* F_next = next_state.F;
    Real* U_next = next_state.U;

    Real* F = state.F;
    Real* U = state.U;
    
    Real* b_F = solver->maps.b_F;
    Real* b_U = solver->maps.b_U;

    Cross_Matrix_Anisotrophy A_F = {0};
    A_F.scale = solver->maps.scale;
    A_F.C_minus_one = 2*dt/(alpha*dx*dx) + 2*dt/(alpha*dy*dy);
    A_F.X = -dt/(alpha*dx*dx);
    A_F.Y = -dt/(alpha*dy*dy);

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);

    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        Real T = U[x + y*m];
        Real Phi = F[x + y*m];

        Real Phi_U = *at_mod(F, x, y + 1, m, n);
        Real Phi_D = *at_mod(F, x, y - 1, m, n);
        Real Phi_R = *at_mod(F, x + 1, y, m, n);
        Real Phi_L = *at_mod(F, x - 1, y, m, n);

        Real grad_Phi_x = dy*(Phi_R - Phi_L);
        Real grad_Phi_y = dx*(Phi_U - Phi_D);
        Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);
 
        Real g_theta = 1;
        {
            Real theta = atan2(grad_Phi_y, grad_Phi_x);
            g_theta = 1.0f - S*cosf(m0*theta + theta0);
        }

        // g_theta = 1;

        Real f = g_theta*a*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm/(2*mK);
        A_F.scale[x+y*m] = g_theta;
        b_F[x + y*m] = Phi + dt/(xi*xi*alpha)*f;
    });

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-10;
    solver_params.tolerance = (Real) 1.0e-7;
    solver_params.max_iters = 100;
    solver_params.padded = true;

    //Solve A_F*F_next = b_F
    Conjugate_Gardient_Convergence F_converged = cross_matrix_conjugate_gradient_solve(&A_F, F_next, b_F, m, n, (void*) cross_matrix_aniso_multiply_padded, &solver_params);
    printf("%lli F %s in %i iters with error %lf\n", (long long) iter, F_converged.converged ? "converged" : "diverged", F_converged.iters, F_converged.error);

    //Clamp in valid range. This should reduce the error even more
    // cuda_for(0, N, [=]SHARED(int i){
    //     F_next[i] = MAX(MIN(F_next[i], 1), 0);
    // });

    //Calculate b_U
    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        Real T = *at_mod(U, x, y, n, m);
        Real Phi = *at_mod(F, x, y, n, m);
        Real Phi_next = *at_mod(F_next, x, y, n, m);

        b_U[x + y*m] = T + L*(Phi_next - Phi);
    });

    //Solve A_U*U_next = b_U
    Conjugate_Gardient_Convergence U_converged = cross_matrix_conjugate_gradient_solve(&A_U, U_next, b_U, m, n, (void*) cross_matrix_static_multiply_padded, &solver_params);
    printf("%lli U %s in %i iters with error %lf\n", (long long) iter, U_converged.converged ? "converged" : "diverged", U_converged.iters, U_converged.error);

    if(do_debug)
    {
        Real* AfF = solver->debug_maps.AfF;
        Real* AuU = solver->debug_maps.AuU;
        //Back test
        if(1)
        {
            cross_matrix_aniso_multiply_not_padded(AfF, &A_F, F_next, n, m);
            cross_matrix_static_multiply_not_padded(AuU, &A_U, U_next, n, m);

            Real back_error_F = vector_get_dist_norm(AfF, b_F, N);
            Real back_error_U = vector_get_dist_norm(AuU, b_U, N);
            printf("F:" REAL_FMT " U:" REAL_FMT " Epsilon:" REAL_FMT "\n", back_error_F, back_error_U, solver_params.tolerance*2);
        }

        Real* grad_F = solver->debug_maps.grad_phi;
        Real* grad_U = solver->debug_maps.grad_T;
        cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
            Real T = *at_mod(U, x, y, n, m);
            Real Phi = *at_mod(F, x, y, n, m);

            Real Phi_U = *at_mod(F, x, y + 1, n, m);
            Real Phi_D = *at_mod(F, x, y - 1, n, m);
            Real Phi_R = *at_mod(F, x + 1, y, n, m);
            Real Phi_L = *at_mod(F, x - 1, y, n, m);

            Real T_U = *at_mod(U, x, y + 1, n, m);
            Real T_D = *at_mod(U, x, y - 1, n, m);
            Real T_R = *at_mod(U, x + 1, y, n, m);
            Real T_L = *at_mod(U, x - 1, y, n, m);

            Real grad_Phi_x = (Phi_R - Phi_L);
            Real grad_Phi_y = (Phi_U - Phi_D);
            Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);

            Real grad_T_x = (T_R - T_L);
            Real grad_T_y = (T_U - T_D);
            Real grad_T_norm = hypotf(grad_T_x, grad_T_y);
            
            grad_F[x + y*m] = grad_Phi_norm;
            grad_U[x + y*m] = grad_T_norm;
        });
    }
}

void semi_implicit_solver_get_maps(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * map_count);
    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->maps.b_F, "b_F");           
    ASSIGN_MAP_NAMED(solver->debug_maps.AfF, "AfF");           
    ASSIGN_MAP_NAMED(solver->maps.b_U, "b_U");           
    ASSIGN_MAP_NAMED(solver->debug_maps.AuU, "AuU");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");           
    ASSIGN_MAP_NAMED(solver->maps.scale, "Anisotrophy");  
}

extern "C" void kernel_float_from_double(float* output, const double* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (float) input[i];
    });
}
extern "C" void kernel_double_from_float(double* output, const float* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (double) input[i];
    });
}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify)
{
    if(modify == MODIFY_UPLOAD)
        CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice));
    else
        CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost));
}

extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t size, Sim_Modify modify)
{
    static float* static_device = NULL;
    static size_t static_size = 0;

    if(sizeof(Real) != sizeof(float))
    {
        if(static_size < size)
        {
            cuda_realloc_in_place((void**) &static_device, size*sizeof(float), static_size*sizeof(float), 0);
            static_size = size;
        }

        if(modify == MODIFY_UPLOAD)
        {
            //Upload: host -> static -> device
            CUDA_DEBUG_TEST(hipMemcpy(static_device, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
            kernel_double_from_float((double*) (void*) device_memory, static_device, size);
        }
        else
        {
            //download: device -> static -> host
            kernel_float_from_double(static_device, (double*) (void*) device_memory, size);
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, static_device, size*sizeof(float), hipMemcpyDeviceToHost));
        }
    }
    else
    {
        if(modify == MODIFY_UPLOAD)
            CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
        else
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size*sizeof(float), hipMemcpyDeviceToHost));
    }
}


extern "C" int  sim_solver_reinit(Sim_Solver* solver, Solver_Type type, int n, int m)
{
    if(solver->type != type && solver->type != SOLVER_TYPE_NONE)
        sim_solver_reinit(solver, solver->type, 0, 0);

    int out = 0;
    switch(type) {
        case SOLVER_TYPE_NONE: {
            n = 0;
            m = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            out = explicit_solver_resize(&solver->expli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            out = semi_implicit_solver_resize(&solver->impli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            assert(false);
        } break;

        default: {
            assert(false);
        }
    };

    solver->type = type;
    solver->m = m;
    solver->n = n;
    return out;
}

void sim_state_reinit(Sim_State* states, Solver_Type type, int n, int m)
{
    if(states->type != type && states->type != SOLVER_TYPE_NONE)
        sim_state_reinit(states, states->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            n = 0;
            m = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            explicit_state_resize(&states->expli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_state_resize(&states->impli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            assert(false);
        } break;

        default: {
            assert(false);
        }
    };

    states->type = type;
    states->m = m;
    states->n = n;
}

extern "C" void sim_states_reinit(Sim_State* states, int state_count, Solver_Type type, int n, int m)
{
    for(int i = 0; i < state_count; i++)
        sim_state_reinit(&states[i], type, n, m);
}


void _switch(Solver_Type type)
{
    switch(type) {
        case SOLVER_TYPE_NONE: {

        } break;

        case SOLVER_TYPE_EXPLICIT: {
            
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {

        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {

        } break;

        default: assert(false);
    };
}


extern "C" void sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Allen_Cahn_Params params, bool do_debug)
{

    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            // nothing
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            if(states_count < EXPLICIT_SOLVER_REQUIRED_HISTORY)
                printf("explicit solver requires bigger history\n");
            else
            {
                Sim_State state = states[iter % states_count];
                Sim_State next_state = states[(iter + 1) % states_count];
                if(state.type != SOLVER_TYPE_EXPLICIT || next_state.type != SOLVER_TYPE_EXPLICIT)
                    printf("not matching state provided to explicit solver\n");
                else
                    explicit_solver_step(&solver->expli, state.expli, next_state.expli, params, iter, do_debug);
            }
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            if(states_count < EXPLICIT_SOLVER_REQUIRED_HISTORY)
                printf("explicit solver requires bigger history");
            else
            {
                Sim_State state = states[iter % states_count];
                Sim_State next_state = states[(iter + 1) % states_count];
                if(state.type != SOLVER_TYPE_SEMI_IMPLICIT || next_state.type != SOLVER_TYPE_SEMI_IMPLICIT)
                    printf("not matching state provided to semi implicit solver\n");
                else
                semi_implicit_solver_step(&solver->impli, state.impli, next_state.impli, params, iter, do_debug);
            }
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            assert(false);
        } break;

        default: assert(false);
    };
}

extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count)
{
    if(states_count <= 0 || map_count <= 0)
        return;

    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            //none
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            explicit_solver_get_maps(&solver->expli, states[iter % states_count].expli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_get_maps(&solver->impli, states[iter % states_count].impli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            assert(false);
        } break;

        default: assert(false);
    };
}