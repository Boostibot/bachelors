#include "hip/hip_runtime.h"
#define SHARED __host__ __device__

#include <>
#include "kernel.h"
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <stdint.h>
#include <hip/hip_runtime.h>


#define MOD(x, mod) (((x) % (mod) + (mod)) % (mod))
#define MAX(a, b)   ((a) > (b) ? (a) : (b))
#define MIN(a, b)   ((a) < (b) ? (a) : (b))
#define PI          ((Real) 3.14159265359)
#define TAU         (2*PI)
#define CUDA_ERR_AND(err) (err) != hipSuccess ? (err) :
#define ECHOF(x)    printf(#x ": " REAL_FMT "\n", (x))

#include <hip/hip_runtime.h>
#include <stdarg.h>
static bool _test_cuda_(hipError_t error, const char* expression, int line, const char* file, const char* format, ...)
{
    if(error != hipSuccess)
    {
        printf("CUDA_TEST(%s) failed with %s! %s:%i\n", expression, hipGetErrorString(error), file, line);
        if(format != NULL && strlen(format) != 0)
        {
            va_list args;
            va_start(args, format);
            vprintf(format, args);
            va_end(args);
            printf("\n");
        }
    }
    fflush(stdout);
    return error == hipSuccess;
}

static void _test(const char* expression, int line, const char* file, const char* format, ...)
{
    if(format != NULL && strlen(format) != 0)
    {
        va_list args;
        va_start(args, format);
        vprintf(format, args);
        va_end(args);
        printf("\n");
    }
    else
    {
        printf("TEST(%s) failed! %s:%i\n", expression, file, line);
    }

    fflush(stdout);
}

#define CUDA_TEST(status, ...) (_test_cuda_((status), #status,  __LINE__, __FILE__, "" __VA_ARGS__) ? (void) 0 : abort())
#define TEST(x, ...)           ((x) ? (void) 0 : (_test(#x,  __LINE__, __FILE__, "" __VA_ARGS__), abort()))

#undef NDEBUG

#ifdef NDEBUG
    #define CUDA_DEBUG_TEST(status, ...) (0 ? printf("" __VA_ARGS__) : (status))
#else
    #define CUDA_DEBUG_TEST(status, ...) CUDA_TEST(status, __VA_ARGS__)
#endif

struct Cuda_Info {
    int device_id;
    hipDeviceProp_t prop;
};

Cuda_Info cuda_one_time_setup()
{
    static bool was_setup = false;
    static Cuda_Info info = {0};

    if(was_setup == false)
    {
        enum {MAX_DEVICES = 100};
        hipDeviceProp_t devices[MAX_DEVICES] = {0};
        double scores[MAX_DEVICES] = {0};
        double peak_memory[MAX_DEVICES] = {0};
        
        int nDevices = 0;
        CUDA_TEST(hipGetDeviceCount(&nDevices));
        if(nDevices > MAX_DEVICES)
        {
            assert(false && "wow this should probably not happen!");
            nDevices = MAX_DEVICES;
        }
        TEST(nDevices > 0, "Didnt find any CUDA capable devices. Stopping.");

        for (int i = 0; i < nDevices; i++) 
            CUDA_DEBUG_TEST(hipGetDeviceProperties(&devices[i], i));

        //compute maximum in each tracked category to
        // be able to properly select the best device for
        // the job!
        hipDeviceProp_t max_prop = {0};
        double max_peak_memory = 0;
        for (int i = 0; i < nDevices; i++) 
        {
            hipDeviceProp_t prop = devices[i];
            max_prop.warpSize = MAX(max_prop.warpSize, prop.warpSize);
            max_prop.multiProcessorCount = MAX(max_prop.multiProcessorCount, prop.multiProcessorCount);
            max_prop.concurrentKernels = MAX(max_prop.concurrentKernels, prop.concurrentKernels);
            max_prop.memoryClockRate = MAX(max_prop.memoryClockRate, prop.memoryClockRate);
            max_prop.memoryBusWidth = MAX(max_prop.memoryBusWidth, prop.memoryBusWidth);
            max_prop.totalGlobalMem = MAX(max_prop.totalGlobalMem, prop.totalGlobalMem);
            max_prop.sharedMemPerBlock = MAX(max_prop.sharedMemPerBlock, prop.sharedMemPerBlock);
            peak_memory[i] = 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6;

            max_peak_memory = MAX(max_peak_memory, peak_memory[i]);
        }

        double max_score = 0;
        int max_score_i = 0;
        for (int i = 0; i < nDevices; i++) 
        {
            hipDeviceProp_t prop = devices[i];
            double score = 0
                + 0.40 * prop.warpSize/max_prop.warpSize
                + 0.40 * prop.multiProcessorCount/max_prop.multiProcessorCount
                + 0.05 * prop.concurrentKernels/max_prop.concurrentKernels
                + 0.05 * peak_memory[i]/max_peak_memory
                + 0.05 * prop.totalGlobalMem/max_prop.totalGlobalMem
                + 0.05 * prop.sharedMemPerBlock/max_prop.sharedMemPerBlock
                ;

            scores[i] = score;
            if(max_score < score)
            {
                max_score = score;
                max_score_i = i;
            }
        }
        hipDeviceProp_t selected = devices[max_score_i];
        info.prop = selected;
        info.device_id = max_score_i;
        was_setup = true;
        CUDA_TEST(hipSetDevice(info.device_id));

        printf("Listing devices below (%d):\n", nDevices);
        for (int i = 0; i < nDevices; i++)
            printf("%i > %s (score: %lf) %s\n", i, devices[i].name, scores[i], i == max_score_i ? "[selected]" : "");

        printf("Selected %s:\n", selected.name);
        printf("  Multi Processor count: %i\n", selected.multiProcessorCount);
        printf("  Warp-size: %d\n", selected.warpSize);
        printf("  Memory Clock Rate (MHz): %d\n", selected.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n", selected.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n", peak_memory[max_score_i]);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(selected.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(selected.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", selected.minor, selected.major);
        printf("  Concurrent kernels: %s\n", selected.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",selected.deviceOverlap ? "yes" : "no");
    }

    return info;
}

enum Cuda_For_Flags {
    CUDA_FOR_NONE = 0,
    CUDA_FOR_ASYNC = 1,
};

template <typename Function>
__global__ void _kernel_cuda_for_each(int from, int item_count, Function func)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < item_count; i += blockDim.x * gridDim.x) 
        func(from + i);
}

template <typename Function>
void cuda_for(int from, int to, Function func, int flags = 0)
{
    Cuda_Info info = cuda_one_time_setup();
    dim3 bs(64, 1);
    dim3 grid(info.prop.multiProcessorCount, 1);

    _kernel_cuda_for_each<<<grid, bs>>>(from, to-from, (Function&&) func);
    CUDA_DEBUG_TEST(hipGetLastError());
    if((flags & CUDA_FOR_ASYNC) == 0)
        CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

template <typename Function>
__global__ void _kernel_cuda_for_each_2D(int from_x, int x_size, int from_y, int y_size, Function func)
{
    //@TODO: Whats the optimal loop order? First x or y?
    for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < y_size; y += blockDim.y * gridDim.y) 
        for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < x_size; x += blockDim.x * gridDim.x) 
            func(x + from_x, y + from_y);
}

template <typename Function>
void cuda_for_2D(int from_x, int from_y, int to_x, int to_y, Function func, int flags = 0)
{
    Cuda_Info info = cuda_one_time_setup();
    dim3 bs(64, 1);
    dim3 grid(info.prop.multiProcessorCount, 1);
    _kernel_cuda_for_each_2D<<<grid, bs>>>(from_x, to_x-from_x, from_y, to_y-from_y, (Function&&) func);
    CUDA_DEBUG_TEST(hipGetLastError());
    if((flags & CUDA_FOR_ASYNC) == 0)
        CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

enum {
    REALLOC_COPY = 1,
    REALLOC_ZERO = 2,
};

typedef struct Memory_Format {
    const char* unit;
    size_t unit_value;
    double fraction;

    int whole;
    int remainder;
} Memory_Format;

Memory_Format get_memory_format(size_t bytes)
{
    size_t B  = (size_t) 1;
    size_t KB = (size_t) 1024;
    size_t MB = (size_t) 1024*1024;
    size_t GB = (size_t) 1024*1024*1024;
    size_t TB = (size_t) 1024*1024*1024*1024;

    Memory_Format out = {0};
    out.unit = "";
    out.unit_value = 1;
    if(bytes >= TB)
    {
        out.unit = "TB";
        out.unit_value = TB;
    }
    else if(bytes >= GB)
    {
        out.unit = "GB";
        out.unit_value = GB;
    }
    else if(bytes >= MB)
    {
        out.unit = "MB";
        out.unit_value = MB;
    }
    else if(bytes >= KB)
    {
        out.unit = "KB";
        out.unit_value = KB;
    }
    else
    {
        out.unit = "B";
        out.unit_value = B;
    }

    out.fraction = (double) bytes / (double) out.unit_value;
    out.whole = (int) (bytes / out.unit_value);
    out.remainder = (int) (bytes / out.unit_value);

    return out;
}

#define MEMORY_FMT "%.2lf%s"
#define MEMORY_PRINT(bytes) get_memory_format((bytes)).fraction, get_memory_format((bytes)).unit

void* _cuda_realloc(void* old_ptr, size_t new_size, size_t old_size, int flags, const char* file, int line)
{
    printf("CUDA realloc " MEMORY_FMT "-> " MEMORY_FMT " %s:%i\n",
            MEMORY_PRINT(old_size), 
            MEMORY_PRINT(new_size),
            file, line);

    static int64_t used_bytes = 0;
    void* new_ptr = NULL;
    if(new_size != 0)
    {
        Cuda_Info info = cuda_one_time_setup();
        CUDA_TEST(hipMalloc(&new_ptr, new_size), 
            "Out of CUDA memory! Requested " MEMORY_FMT ". Using " MEMORY_FMT " / " MEMORY_FMT ". %s:%i", 
            MEMORY_PRINT(new_size), 
            MEMORY_PRINT(used_bytes), 
            MEMORY_PRINT(info.prop.totalGlobalMem),
            file, line);

        size_t min_size = MIN(old_size, new_size);
        if((flags & REALLOC_ZERO) && !(flags & REALLOC_COPY))
            CUDA_DEBUG_TEST(hipMemset(new_ptr, 0, new_size));
        else
        {
            if(flags & REALLOC_COPY)
                CUDA_DEBUG_TEST(hipMemcpy(new_ptr, old_ptr, min_size, hipMemcpyDeviceToDevice));
            if(flags & REALLOC_ZERO)
                CUDA_DEBUG_TEST(hipMemset((uint8_t*) new_ptr + min_size, 0, new_size - min_size));
        }
    }


    CUDA_DEBUG_TEST(hipFree(old_ptr), 
        "Invalid pointer passed to cuda_realloc! %s:%i", file, line);

    used_bytes += (int64_t) new_size - (int64_t) old_size;
    assert(used_bytes >= 0);
    return new_ptr;
}

void _cuda_realloc_in_place(void** ptr_ptr, size_t new_size, size_t old_size, int flags, const char* file, int line)
{
    *ptr_ptr = _cuda_realloc(*ptr_ptr, new_size, old_size, flags, file, line);
}

#define cuda_realloc(old_ptr, new_size, old_size, flags)          _cuda_realloc(old_ptr, new_size, old_size, flags, __FILE__, __LINE__)
#define cuda_realloc_in_place(ptr_ptr, new_size, old_size, flags) _cuda_realloc_in_place(ptr_ptr, new_size, old_size, flags, __FILE__, __LINE__)

SHARED Real* at_mod(Real* map, int x, int y, int n, int m)
{
    int x_mod = MOD(x, m);
    int y_mod = MOD(y, n);

    return &map[x_mod + y_mod*m];
}

SHARED Real f0(Real phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}


int explicit_solver_resize(Explicit_Solver* solver, int n, int m)
{
    size_t N = (size_t)m*(size_t)n;
    size_t N_old = (size_t)solver->m*(size_t)solver->n;
    if(solver->m != m || solver->n != n)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real* debug_maps = (Real*) (void*) &solver->debug_maps;
        for(int i = 0; i < sizeof(solver->debug_maps) / sizeof(Real); i++)
            cuda_realloc_in_place((void**) &debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->m = m;
        solver->n = n;
    }

    return EXPLICIT_SOLVER_REQUIRED_HISTORY;
}

void explicit_state_resize(Explicit_State* state, int n, int m)
{
    size_t N = (size_t)m*(size_t)n;
    size_t N_old = (size_t)state->m*(size_t)state->n;
    if(state->m != m || state->n != n)
    {
        cuda_realloc_in_place((void**) &state->F, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &state->U, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        state->m = m;
        state->n = n;
    }
}

union Explicit_Solve_In {
    struct {
        Real Phi;
        Real Phi_U;
        Real Phi_D;
        Real Phi_L;
        Real Phi_R;
        Real T;
        Real T_U;
        Real T_D;
        Real T_L;
        Real T_R;
    };
    Real vals[10];
};
struct Explicit_Solve_Result {
    Real dt_Phi;
    Real dt_T;
};

struct Explicit_Solve_Debug {
    Real grad_Phi;
    Real grad_T;
    Real g_theta;
    Real theta;
    Real reaction_term;
};

SHARED Explicit_Solve_Result allen_cahn_explicit_solve(Explicit_Solve_In input, Allen_Cahn_Params params, Explicit_Solve_Debug* debug_or_null)
{
    Real dx = (Real) params.L0 / params.m;
    Real dy = (Real) params.L0 / params.n;
    Real mK = dx * dy;

    //@NOTE: dont you wish we had odin lang using in C?
    Real a = params.a;
    Real b = params.b;
    Real alpha = params.alpha;
    Real beta = params.beta;
    Real xi = params.xi;
    Real Tm = params.Tm;
    Real L = params.L; //Latent heat, not L0 (sym size) ! 
    // Real dt = params.dt;
    Real S = params.S; //anisotrophy strength
    Real m0 = params.m0; //anisotrophy frequency (?)
    Real theta0 = params.theta0;

    Real Phi = input.Phi;
    Real Phi_U = input.Phi_U;
    Real Phi_D = input.Phi_D;
    Real Phi_L = input.Phi_L;
    Real Phi_R = input.Phi_R;
    Real T = input.T;
    Real T_U = input.T_U;
    Real T_D = input.T_D;
    Real T_L = input.T_L;
    Real T_R = input.T_R;

    Real grad_T_x = dy*(T_R - T_L);
    Real grad_T_y = dx*(T_U - T_D);

    Real grad_Phi_x = dy*(Phi_R - Phi_L);
    Real grad_Phi_y = dx*(Phi_U - Phi_D);

    Real grad_T_norm = hypotf(grad_T_x, grad_T_y);
    Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);

    Real g_theta = 1;
    Real theta = 0;
    //prevent nans
    // if(grad_Phi_norm > 0.0001)
    {

        theta = atan2(grad_Phi_y, grad_Phi_x);
        g_theta = 1.0f - S*cosf(m0*theta + theta0);
    }

    Real int_K_laplace_T   = dy/dx*(T_L - 2*T + T_R)       + dx/dy*(T_D - 2*T + T_U);
    Real int_K_laplace_Phi = dy/dx*(Phi_L - 2*Phi + Phi_R) + dx/dy*(Phi_D - 2*Phi + Phi_U);
    Real int_K_f = g_theta*a*mK*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm/2;

    Real int_K_dt_Phi = g_theta/alpha*int_K_laplace_Phi + 1/(xi*xi * alpha)*int_K_f;
    Real int_K_dt_T = int_K_laplace_T + L*int_K_dt_Phi;

    Real dt_Phi = 1/mK*int_K_dt_Phi;
    Real dt_T = 1/mK*int_K_dt_T;

    if(debug_or_null)
    {
        debug_or_null->grad_Phi = hypotf(Phi_R - Phi_L, Phi_U - Phi_D);
        debug_or_null->grad_T = hypotf(T_R - T_L, T_U - T_D);
        debug_or_null->reaction_term = int_K_f / mK;
        debug_or_null->g_theta = g_theta;
        debug_or_null->theta = g_theta;
    }

    Explicit_Solve_Result out = {dt_Phi, dt_T};
    return out;
}

SHARED Explicit_Solve_In explicit_solve_in_mod(const Real* Phi, const Real* T, int x, int y, int n, int m)
{
    Explicit_Solve_In solve = {0};
    solve.T = T[x + y*m];
    solve.Phi = Phi[x + y*m];

    solve.Phi_U = *at_mod((Real*) Phi, x, y + 1, n, m);
    solve.Phi_D = *at_mod((Real*) Phi, x, y - 1, n, m);
    solve.Phi_R = *at_mod((Real*) Phi, x + 1, y, n, m);
    solve.Phi_L = *at_mod((Real*) Phi, x - 1, y, n, m);

    solve.T_U = *at_mod((Real*) T, x, y + 1, n, m);
    solve.T_D = *at_mod((Real*) T, x, y - 1, n, m);
    solve.T_R = *at_mod((Real*) T, x + 1, y, n, m);
    solve.T_L = *at_mod((Real*) T, x - 1, y, n, m);

    return solve;
}

extern "C" void explicit_solver_newton_step(Explicit_Solver* solver, Explicit_State state, Explicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real* Phi_next = next_state.F;
    Real* Phi = state.F;
    
    Real* T_next = next_state.U;
    Real* T = state.U;

    Explicit_Solver expli = *solver;
    int m = params.m;
    int n = params.n;
    if(do_debug)
    {
        cuda_for_2D(0, 0, params.m, params.n, [=]SHARED(int x, int y){
            Explicit_Solve_Debug debug = {0};
            Explicit_Solve_In input = explicit_solve_in_mod(Phi, T, x, y, n, m);
            Explicit_Solve_Result solved = allen_cahn_explicit_solve(input, params, &debug);

            //Newton update
            Phi_next[x + y*m] = input.Phi + solved.dt_Phi*params.dt;
            T_next[x + y*m] = input.T + solved.dt_T*params.dt;

            expli.debug_maps.grad_phi[x + y*m] = debug.grad_Phi;
            expli.debug_maps.grad_T[x + y*m] = debug.grad_T;
            expli.debug_maps.reaction[x + y*m] = debug.reaction_term;
            expli.debug_maps.aniso_factor[x + y*m] = debug.g_theta;
        });
    }
    else
    {
        cuda_for_2D(0, 0, params.m, params.n, [=]SHARED(int x, int y){
            Explicit_Solve_In input = explicit_solve_in_mod(Phi, T, x, y, n, m);
            Explicit_Solve_Result solved = allen_cahn_explicit_solve(input, params, NULL);

            Phi_next[x + y*m] = input.Phi + solved.dt_Phi*params.dt;
            T_next[x + y*m] = input.T + solved.dt_T*params.dt;
        });
    }
}


void explicit_solver_rk4_dt(Explicit_State* out, Explicit_State base_state, Real blend_factor, Explicit_State half_state, Allen_Cahn_Params params)
{
    int m = params.m;
    int n = params.n;
    Real* out_F = out->F;
    Real* out_U = out->U;

    //If blend fatcor == 0 no point in loading extra memory...
    if(blend_factor == 0)
    {
        cuda_for_2D(0, 0, params.m, params.n, [=]SHARED(int x, int y){
            Explicit_Solve_In input_base = explicit_solve_in_mod(base_state.F, base_state.U, x, y, n, m);
            Explicit_Solve_Result solved = allen_cahn_explicit_solve(input_base, params, NULL);

            out_F[x + y*m] = solved.dt_Phi;
            out_U[x + y*m] = solved.dt_T;
        });
    }
    else
    {
        cuda_for_2D(0, 0, params.m, params.n, [=]SHARED(int x, int y){
            Explicit_Solve_In input_base = explicit_solve_in_mod(base_state.F, base_state.U, x, y, n, m);
            Explicit_Solve_In input_half = explicit_solve_in_mod(half_state.F, half_state.U, x, y, n, m);
            Explicit_Solve_In input_blend = {0};

            for(int i = 0; i < sizeof(input_blend.vals) / sizeof(input_blend.vals[0]); i++)
                input_blend.vals[i] = input_base.vals[i] + blend_factor*input_half.vals[i];

            Explicit_Solve_Result solved = allen_cahn_explicit_solve(input_blend, params, NULL);

            out_F[x + y*m] = solved.dt_Phi;
            out_U[x + y*m] = solved.dt_T;
        });
    }
}

#if 0
static double runge_kutta4(double t_ini, double x_ini, double T, double tau, RK4_Func f, void* context)
{
    double x = x_ini;
    for (double t = t_ini; t <= T; t += tau)
    {
        double k1 = tau*f(t, x, context);
        double k2 = tau*f(t + 0.5f*tau, x + 0.5f*k1, context);
        double k3 = tau*f(t + 0.5f*tau, x + 0.5f*k2, context);
        double k4 = tau*f(t + tau, x + k3, context);
  
        double x_next = x + (k1 + 2*k2 + 2*k3 + k4)/6;
        x = x_next;  
    }
  
    return x;
}
#endif

extern "C" void explicit_solver_rk4_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Explicit_State empty = {0};
    static Explicit_State steps[4] = {0};
    if(steps[0].m != params.m || steps[0].n != params.n)
    {
        for(int i = 0; i < 4; i++)
            explicit_state_resize(&steps[i], params.n, params.n);
    }

    Explicit_State k1 = steps[0];
    Explicit_State k2 = steps[1];
    Explicit_State k3 = steps[2];
    Explicit_State k4 = steps[3];

    Real dt = params.dt;
    explicit_solver_rk4_dt(&k1, state, dt * 0, empty, params);
    explicit_solver_rk4_dt(&k2, state, dt * 0.5, k1, params);
    explicit_solver_rk4_dt(&k3, state, dt * 0.5, k2, params);
    explicit_solver_rk4_dt(&k4, state, dt * 1, k3, params);

    //@TODO: async and explicit sync.
    Real* out_F = next_state->F;
    Real* out_U = next_state->U;
    cuda_for(0, params.n*params.m, [=]SHARED(int i){
        // out_F[i] =  state.F[i] + k1.F[i]*dt;
        // out_U[i] =  state.U[i] + k1.F[i]*dt;
        out_F[i] =  state.F[i] + dt/6*(k1.F[i] + 2*k2.F[i] + 2*k3.F[i] + k4.F[i]);
        out_U[i] =  state.U[i] + dt/6*(k1.U[i] + 2*k2.U[i] + 2*k3.U[i] + k4.U[i]);
    });
}

void explicit_solver_get_maps(Explicit_Solver* solver, Explicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * map_count);

    #define ASSIGN_MAP_NAMED(var_ptr, var_name) \
        if(__map_i < map_count) \
        { \
            maps[__map_i].data = var_ptr; \
            maps[__map_i].name = var_name; \
            maps[__map_i].m = solver->m; \
            maps[__map_i].n = solver->n; \
            __map_i += 1; \
        }\

    #define ASSIGN_MAP(var_ptr) ASSIGN_MAP_NAMED(var_ptr, #var_ptr) 

    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");
    ASSIGN_MAP_NAMED(solver->debug_maps.aniso_factor, "aniso_factor");
    ASSIGN_MAP_NAMED(solver->debug_maps.reaction, "reaction");
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residual, "step_residual");
}



//Will hand write my own version later. For now we trust in thrust *cymbal*
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>

Real vector_dot_product(const Real *a, const Real *b, int n)
{
  // wrap raw pointers to device memory with device_ptr
  thrust::device_ptr<const Real> d_a(a);
  thrust::device_ptr<const Real> d_b(b);

  // inner_product implements a mathematical dot product
  return thrust::inner_product(d_a, d_a + n, d_b, 0.0);
}

struct Cross_Matrix_Static {
    Real C;
    Real U;
    Real D;
    Real L;
    Real R;

    int m;
    int n;
};

struct Cross_Matrix {
    Real* C;
    Real* U;
    Real* D;
    Real* L;
    Real* R;

    int m;
    int n;
};

struct Anisotrophy_Matrix {
    Real* scale;
    Real X;
    Real Y;
    Real C_minus_one;

    int m;
    int n;
};

void* cross_matrix_vector_alloced(Real* vector, int n, int m)
{
    if(vector == NULL)
        return NULL;
    (void) n;
    return vector - m;
}

Real* cross_matrix_vector_padded(void* alloced, int n, int m)
{
    if(alloced == NULL)
        return (Real*) NULL;
    (void) n;
    return (Real*) alloced + m;
}

void cross_matrix_vector_pad(Real* vector, int n, int m)
{
    CUDA_DEBUG_TEST(hipMemset(vector - m, 0, sizeof(Real)*m));
    CUDA_DEBUG_TEST(hipMemset(vector + n*m, 0, sizeof(Real)*m));
}

Real* cross_matrix_vector_realloc(Real* vector, int n, int m, int old_n, int old_m)
{
    int new_size = 2*m + n*m;
    int old_size = 2*old_m + old_n*old_m;
    void* old = cross_matrix_vector_alloced(vector, n, m);
    void* new_ = cuda_realloc(old, (size_t) new_size * sizeof(Real), (size_t) old_size * sizeof(Real), REALLOC_ZERO);
    return cross_matrix_vector_padded(new_, n, m);
}


void cross_matrix_static_multiply(Real* out, const void* _A, const Real* x, int N)
{
    Cross_Matrix_Static A = *(Cross_Matrix_Static*)_A;
    int m = A.m;
    cuda_for(0, N, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C;
        if(i+1 < N)  val += x[i+1]*A.R;
        if(i-1 >= 0) val += x[i-1]*A.L;
        if(i+m < N)  val += x[i+m]*A.U;
        if(i-m >= 0) val += x[i-m]*A.D;

        out[i] = val;
    });
}

void cross_matrix_multiply(Real* out, const void* _A, const Real* x, int N)
{
    Cross_Matrix A = *(Cross_Matrix*)_A;
    int m = A.m;
    cuda_for(0, N, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C[i];
        if(i+1 < N)  val += x[i+1]*A.R[i];
        if(i-1 >= 0) val += x[i-1]*A.L[i];
        if(i+m < N)  val += x[i+m]*A.U[i];
        if(i-m >= 0) val += x[i-m]*A.D[i];

        out[i] = val;
    });
}

void anisotrophy_matrix_multiply(Real* out, const void* _A, const Real* x, int N)
{
    Anisotrophy_Matrix A = * (Anisotrophy_Matrix*)_A;
    int m = A.m;
    cuda_for(0, N, [=]SHARED(int i){
        Real s = A.scale[i];
        Real X = A.X*s;
        Real Y = A.Y*s;
        Real C = 1 + A.C_minus_one*s;

        Real val = 0;
        val += x[i]*C;
        if(i+1 < N)  val += x[i+1]*X;
        if(i-1 >= 0) val += x[i-1]*X;
        if(i+m < N)  val += x[i+m]*Y;
        if(i-m >= 0) val += x[i-m]*Y;

        out[i] = val;
    });
}


typedef struct Conjugate_Gardient_Params {
    Real epsilon;
    Real tolerance;
    int max_iters;

    Real* initial_value_or_null;
} Conjugate_Gardient_Params;

typedef struct Conjugate_Gardient_Convergence {
    Real error;
    int iters;
    bool converged;
} Conjugate_Gardient_Convergence;

typedef void(*Matrix_Vector_Mul_Func)(Real* out, const void* A, const Real* x, int N);

Conjugate_Gardient_Convergence conjugate_gradient_solve(const void* A, Real* x, const Real* b, int N, Matrix_Vector_Mul_Func matrix_mul_func, const Conjugate_Gardient_Params* params_or_null)
{
    Conjugate_Gardient_Params params = {0};
    params.epsilon = (Real) 1.0e-10;
    params.tolerance = (Real) 1.0e-5;
    params.max_iters = 10;
    if(params_or_null)
        params = *params_or_null;

    //@NOTE: Evil programmer doing evil programming practices
    static int static_N = 0;
    static Real* _r = NULL;
    static Real* _p = NULL;
    static Real* _Ap = NULL;
    if(static_N < N)
    {
        cuda_realloc_in_place((void**) &_r, N*sizeof(Real), static_N*sizeof(Real), 0);
        cuda_realloc_in_place((void**) &_p, N*sizeof(Real), static_N*sizeof(Real), 0);
        cuda_realloc_in_place((void**) &_Ap, N*sizeof(Real), static_N*sizeof(Real), 0);
        static_N = N;
    }
   
    //NVCC seams to struggle with statics in device code
    // (is probably passing them by global reference or something)
    Real* r = _r;
    Real* p = _p;
    Real* Ap = _Ap;

    //@TODO: async + explicit sync
    //@TODO: This is broken since r assumes x to be zero the first iter. Make it not.
    // if(params.initial_value_or_null)
        // CUDA_DEBUG_TEST(hipMemcpy(x, params.initial_value_or_null, sizeof(Real)*N, hipMemcpyDeviceToDevice));
    // else
        CUDA_DEBUG_TEST(hipMemset(x, 0, sizeof(Real)*N));
    CUDA_DEBUG_TEST(hipMemcpy(r, b, sizeof(Real)*N, hipMemcpyDeviceToDevice));
    CUDA_DEBUG_TEST(hipMemcpy(p, b, sizeof(Real)*N, hipMemcpyDeviceToDevice));

    Real r_dot_r = vector_dot_product(r, r, N);
    int iter = 0;
    for(; iter < params.max_iters; iter++)
    {
        matrix_mul_func(Ap, A, p, N);
        
        Real p_dot_Ap = vector_dot_product(p, Ap, N);
        Real alpha = r_dot_r / MAX(p_dot_Ap, params.epsilon);
        
        //@TODO: split into two separate kernels and do async
        //@TODO: add option for capping the values for x in a given range.
        //       this can potentially increase convergence and make the reuslts more
        //       accurate (if the range is correct) 
        cuda_for(0, N, [=]SHARED(int i){
            x[i] = x[i] + alpha*p[i];
            r[i] = r[i] - alpha*Ap[i];
        });

        Real r_dot_r_new = vector_dot_product(r, r, N);
        if(r_dot_r_new/N < params.tolerance*params.tolerance)
        {
            r_dot_r = r_dot_r_new;
            break;
        }

        Real beta = r_dot_r_new / MAX(r_dot_r, params.epsilon);
        cuda_for(0, N, [=]SHARED(int i){
            p[i] = r[i] + beta*p[i]; 
        });

        r_dot_r = r_dot_r_new;
    }

    Conjugate_Gardient_Convergence out = {0};
    out.iters = iter;
    out.converged = iter != params.max_iters;
    out.error = sqrt(r_dot_r/N);
    return out;
} 

void matrix_multiply(Real* output, const Real* A, const Real* B, int A_height, int A_width, int B_height, int B_width)
{
    assert(A_width == B_height);
    for(int y = 0; y < A_height; y++)
    {
        for(int x = 0; x < B_width; x++)
        {
            Real val = 0;
            for(int k = 0; k < A_width; k++)
                val += A[k + y*A_width]*B[x + k*B_width];

            output[x + y*B_width] = val;
        }
    }
}

Real vector_get_dist_norm(const Real* a, const Real* b, int N)
{
    static Real* temp = NULL;
    static int temp_size = 0;
    if(temp_size < N)
    {
        cuda_realloc_in_place((void**) &temp, N*sizeof(Real), temp_size*sizeof(Real), 0);
        temp_size = N;
    }

    Real* t = temp; //Needed for lambda to work (they capture statics differently)
    cuda_for(0, N, [=]SHARED(int i){
        t[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_dot_product(temp, temp, N);
    Real error = sqrt(temp_dot_temp/N);
    return error;
}

bool vector_is_near(const Real* a, const Real* b, Real epsilon, int N)
{
    return vector_get_dist_norm(a, b, N) < epsilon;
}

int semi_implicit_solver_resize(Semi_Implicit_Solver* solver, int n, int m)
{
    if(solver->m != m || solver->n != n)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real** debug_maps = (Real**) (void*) &solver->debug_maps;
        for(int i = 0; i < sizeof(solver->debug_maps) / sizeof(Real); i++)
            debug_maps[i] = cross_matrix_vector_realloc(debug_maps[i], n, m, solver->n, solver->m);

        Real** maps = (Real**) (void*) &solver->maps;
        for(int i = 0; i < sizeof(solver->maps) / sizeof(Real); i++)
            maps[i] = cross_matrix_vector_realloc(maps[i], n, m, solver->n, solver->m);

        solver->m = m;
        solver->n = n;
    }

    return SEMI_IMPLICIT_SOLVER_REQUIRED_HISTORY;
}

void semi_implicit_state_resize(Semi_Implicit_State* state, int n, int m)
{
    if(state->m != m || state->n != n)
    {
        state->F = cross_matrix_vector_realloc(state->F, n, m, state->n, state->m);
        state->U = cross_matrix_vector_realloc(state->U, n, m, state->n, state->m);

        state->m = m;
        state->n = n;
    }
}

extern "C" void semi_implicit_solver_step(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->m;
    Real dy = (Real) params.L0 / solver->n;

    int m = solver->m;
    int n = solver->n;
    int N = m*n;

    Real mK = dx * dy;
    Real a = params.a;
    Real b = params.b;
    Real alpha = params.alpha;
    Real beta = params.beta;
    Real xi = params.xi;
    Real Tm = params.Tm;
    Real L = params.L; 
    Real dt = params.dt;
    Real S = params.S; 
    Real m0 = params.m0; 
    Real theta0 = params.theta0;
    
    Real* F_next = next_state.F;
    Real* U_next = next_state.U;

    Real* F = state.F;
    Real* U = state.U;
    
    Real* b_F = solver->maps.b_F;
    Real* b_U = solver->maps.b_U;

    Anisotrophy_Matrix A_F = {0};
    A_F.scale = solver->maps.scale;
    A_F.C_minus_one = 2*dt/(alpha*dx*dx) + 2*dt/(alpha*dy*dy);
    A_F.X = -dt/(alpha*dx*dx);
    A_F.Y = -dt/(alpha*dy*dy);
    A_F.m = m;
    A_F.n = n;

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);
    A_U.m = m;
    A_U.n = n;

    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        Real T = U[x + y*m];
        Real Phi = F[x + y*m];

        Real Phi_U = *at_mod(F, x, y + 1, m, n);
        Real Phi_D = *at_mod(F, x, y - 1, m, n);
        Real Phi_R = *at_mod(F, x + 1, y, m, n);
        Real Phi_L = *at_mod(F, x - 1, y, m, n);

        Real grad_Phi_x = dy*(Phi_R - Phi_L);
        Real grad_Phi_y = dx*(Phi_U - Phi_D);
        Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);
 
        Real g_theta = 1;
        {
            Real theta = atan2(grad_Phi_y, grad_Phi_x);
            g_theta = 1.0f - S*cosf(m0*theta + theta0);
        }

        // g_theta = 1;

        Real f = g_theta*a*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm/(2*mK);
        A_F.scale[x+y*m] = g_theta;
        b_F[x + y*m] = Phi + dt/(xi*xi*alpha)*f;
    });

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-12;
    solver_params.tolerance = (Real) 1.0e-9;
    solver_params.max_iters = 100;
    solver_params.initial_value_or_null = F;

    //Solve A_F*F_next = b_F
    Conjugate_Gardient_Convergence F_converged = conjugate_gradient_solve(&A_F, F_next, b_F, N, anisotrophy_matrix_multiply, &solver_params);
    printf("%lli F %s in %i iters with error %lf\n", (long long) iter, F_converged.converged ? "converged" : "diverged", F_converged.iters, F_converged.error);

    //Clamp in valid range. This should reduce the error even more
    // cuda_for(0, N, [=]SHARED(int i){
    //     F_next[i] = MAX(MIN(F_next[i], 1), 0);
    // });

    //Calculate b_U
    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        Real T = *at_mod(U, x, y, n, m);
        Real Phi = *at_mod(F, x, y, n, m);
        Real Phi_next = *at_mod(F_next, x, y, n, m);

        b_U[x + y*m] = T + L*(Phi_next - Phi);
    });

    solver_params.initial_value_or_null = U;

    //Solve A_U*U_next = b_U
    Conjugate_Gardient_Convergence U_converged = conjugate_gradient_solve(&A_U, U_next, b_U, N, cross_matrix_static_multiply, &solver_params);
    printf("%lli U %s in %i iters with error %lf\n", (long long) iter, U_converged.converged ? "converged" : "diverged", U_converged.iters, U_converged.error);

    if(do_debug)
    {
        Real* AfF = solver->debug_maps.AfF;
        Real* AuU = solver->debug_maps.AuU;
        //Back test
        if(1)
        {
            anisotrophy_matrix_multiply(AfF, &A_F, F_next, N);
            cross_matrix_static_multiply(AuU, &A_U, U_next, N);

            Real back_error_F = vector_get_dist_norm(AfF, b_F, N);
            Real back_error_U = vector_get_dist_norm(AuU, b_U, N);
            printf("F:" REAL_FMT " U:" REAL_FMT " Epsilon:" REAL_FMT "\n", back_error_F, back_error_U, solver_params.tolerance*2);
        }

        Real* grad_F = solver->debug_maps.grad_phi;
        Real* grad_U = solver->debug_maps.grad_T;
        cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
            Real T = *at_mod(U, x, y, n, m);
            Real Phi = *at_mod(F, x, y, n, m);

            Real Phi_U = *at_mod(F, x, y + 1, n, m);
            Real Phi_D = *at_mod(F, x, y - 1, n, m);
            Real Phi_R = *at_mod(F, x + 1, y, n, m);
            Real Phi_L = *at_mod(F, x - 1, y, n, m);

            Real T_U = *at_mod(U, x, y + 1, n, m);
            Real T_D = *at_mod(U, x, y - 1, n, m);
            Real T_R = *at_mod(U, x + 1, y, n, m);
            Real T_L = *at_mod(U, x - 1, y, n, m);

            Real grad_Phi_x = (Phi_R - Phi_L);
            Real grad_Phi_y = (Phi_U - Phi_D);
            Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);

            Real grad_T_x = (T_R - T_L);
            Real grad_T_y = (T_U - T_D);
            Real grad_T_norm = hypotf(grad_T_x, grad_T_y);
            
            grad_F[x + y*m] = grad_Phi_norm;
            grad_U[x + y*m] = grad_T_norm;
        });
    }
}

void semi_implicit_solver_get_maps(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * map_count);
    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->maps.b_F, "b_F");           
    ASSIGN_MAP_NAMED(solver->debug_maps.AfF, "AfF");           
    ASSIGN_MAP_NAMED(solver->maps.b_U, "b_U");           
    ASSIGN_MAP_NAMED(solver->debug_maps.AuU, "AuU");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");           
    ASSIGN_MAP_NAMED(solver->maps.scale, "Anisotrophy");  
}

struct Semi_Implicit_Coupled_Cross_Matrix {
    Anisotrophy_Matrix A_F; //A anisotrophy scaled cross matrix
    Real* B_U; //A changing diagonal 

    Cross_Matrix_Static A_U; //Static cross matrix
    Real B_F; //A single value diagonal

    int m;
    int n;
};

int semi_implicit_coupled_solver_resize(Semi_Implicit_Coupled_Solver* solver, int n, int m)
{
    if(solver->m != m || solver->n != n)
    {
        int N = n*m;
        int N_old = solver->n*solver->m;
        cuda_realloc_in_place((void**) &solver->b_C, 2*N*sizeof(Real), 2*N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->aniso, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->B_U, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->m = m;
        solver->n = n;
    }

    return SEMI_IMPLICIT_SOLVER_REQUIRED_HISTORY;
}

void semi_implicit_coupled_state_resize(Semi_Implicit_Coupled_State* state, int n, int m)
{
    if(state->m != m || state->n != n)
    {
        int N = n*m;
        int N_old = state->n*state->m;
        cuda_realloc_in_place((void**) &state->C, 2*N*sizeof(Real), 2*N_old*sizeof(Real), REALLOC_ZERO);
        state->m = m;
        state->n = n;
    }
}

void semi_implicit_coupled_matrix_multiply(Real* out, const void* A_, const Real* x, int vec_size)
{
    Semi_Implicit_Coupled_Cross_Matrix A = *(Semi_Implicit_Coupled_Cross_Matrix*)A_;

    int m = A.m;
    int n = A.n;
    int N = m*n;

    Real* F = (Real*) x;
    Real* U = (Real*) x + N; 

    Real* out_F = out;
    Real* out_U = out + N;

    //@TODO: async + explicit sync

    //F equation
    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        int i = x + y*m;
        Real s = A.A_F.scale[i];
        Real X = A.A_F.X*s;
        Real Y = A.A_F.Y*s;
        Real C = 1 + A.A_F.C_minus_one*s;

        Real F_val = F[i]*C;
        F_val += *at_mod(F, x + 1, y, n, m)*X;
        F_val += *at_mod(F, x - 1, y, n, m)*X;
        F_val += *at_mod(F, x , y + 1, n, m)*Y;
        F_val += *at_mod(F, x , y - 1, n, m)*Y;

        Real U_val = A.B_U[i]*U[i]; 

        out_F[i] = F_val + U_val;
    });

    //U equation
    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        int i = x + y*m;

        Real U_val = U[i]*A.A_U.C;
        U_val += *at_mod(U, x + 1, y, n, m)*A.A_U.R;
        U_val += *at_mod(U, x - 1, y, n, m)*A.A_U.L;
        U_val += *at_mod(U, x , y + 1, n, m)*A.A_U.U;
        U_val += *at_mod(U, x , y - 1, n, m)*A.A_U.D;

        Real F_val = F[i]*A.B_F;
        out_U[i] = F_val + U_val;
    });
}

void semi_implicit_coupled_solver_step(Semi_Implicit_Coupled_Solver* solver, Semi_Implicit_Coupled_State state, Semi_Implicit_Coupled_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->m;
    Real dy = (Real) params.L0 / solver->n;

    int m = solver->m;
    int n = solver->n;
    int N = m*n;

    Real mK = dx * dy;
    Real a = params.a;
    Real b = params.b;
    Real alpha = params.alpha;
    Real beta = params.beta;
    Real xi = params.xi;
    Real Tm = params.Tm;
    Real L = params.L; 
    Real dt = params.dt;
    Real S = params.S; 
    Real m0 = params.m0; 
    Real theta0 = params.theta0;

    Real* F = state.C;
    Real* U = state.C + N;
    
    Real* b_F = solver->b_C;
    Real* b_U = solver->b_C + N;

    Real* aniso = solver->aniso;
    Real* B_U = solver->B_U;

    //Prepare dynamic data
    cuda_for_2D(0, 0, m, n, [=]SHARED(int x, int y){
        Real T = U[x + y*m];
        Real Phi = F[x + y*m];

        Real Phi_U = *at_mod(F, x, y + 1, m, n);
        Real Phi_D = *at_mod(F, x, y - 1, m, n);
        Real Phi_R = *at_mod(F, x + 1, y, m, n);
        Real Phi_L = *at_mod(F, x - 1, y, m, n);

        Real grad_Phi_x = (Phi_R - Phi_L)/(2*dx);
        Real grad_Phi_y = (Phi_U - Phi_D)/(2*dy);
        Real grad_Phi_norm = hypotf(grad_Phi_x, grad_Phi_y);
 
        Real g_theta = 1;
        {
            Real theta = atan2(grad_Phi_y, grad_Phi_x);
            g_theta = 1.0f - S*cosf(m0*theta + theta0);
        }

        // g_theta = 1;
        Real f_tilda = b*xi*xi*beta*grad_Phi_norm;
        Real f = g_theta*a*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm;
        Real T_factor = dt*f_tilda/(xi*xi*alpha);
        B_U[x+y*m] = -T_factor;
        aniso[x+y*m] = g_theta;
        b_F[x + y*m] = Phi + dt*g_theta/(xi*xi*alpha)*f0(Phi) - T_factor*Tm;
        b_U[x + y*m] = T + L*Phi;
    });

    Anisotrophy_Matrix A_F = {0};
    A_F.scale = aniso;
    A_F.C_minus_one = 2*dt/(alpha*dx*dx) + 2*dt/(alpha*dy*dy);
    A_F.X = -dt/(alpha*dx*dx);
    A_F.Y = -dt/(alpha*dy*dy);

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);

    Real B_U_norm = vector_dot_product(B_U, B_U, N);
    B_U_norm = sqrt(B_U_norm / N);
    ECHOF(B_U_norm);
    ECHOF(A_F.C_minus_one + 1);
    ECHOF(A_F.X);
    ECHOF(A_U.C);
    ECHOF(A_U.U);

    Semi_Implicit_Coupled_Cross_Matrix A_C = {0};
    A_C.A_F = A_F;
    A_C.B_U = B_U;
    A_C.A_U = A_U;
    A_C.B_F = -L;
    A_C.m = m;
    A_C.n = n;

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-10;
    solver_params.tolerance = (Real) 1.0e-7;
    solver_params.max_iters = 200;
    solver_params.initial_value_or_null = state.C;

    Conjugate_Gardient_Convergence conv = conjugate_gradient_solve(&A_C, next_state.C, solver->b_C, 2*N, semi_implicit_coupled_matrix_multiply, &solver_params);
    printf("%lli C %s in %i iters with error %lf\n", (long long) iter, conv.converged ? "converged" : "diverged", conv.iters, conv.error);
}

void semi_implicit_coupled_solver_get_maps(Semi_Implicit_Coupled_Solver* solver, Semi_Implicit_Coupled_State state, Sim_Map* maps, int map_count)
{
    int N = solver->m*solver->n;
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * map_count);
    ASSIGN_MAP_NAMED(state.C, "Phi");            
    ASSIGN_MAP_NAMED(state.C + N, "T");            
    ASSIGN_MAP_NAMED(solver->b_C, "b_F");           
    ASSIGN_MAP_NAMED(solver->b_C + N, "b_U");           
    ASSIGN_MAP_NAMED(solver->B_U, "B_U");           
    ASSIGN_MAP_NAMED(solver->aniso, "Anisotrophy");  
}

extern "C" void kernel_float_from_double(float* output, const double* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (float) input[i];
    });
}
extern "C" void kernel_double_from_float(double* output, const float* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (double) input[i];
    });
}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify)
{
    if(modify == MODIFY_UPLOAD)
        CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice));
    else
        CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost));
}

extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t size, Sim_Modify modify)
{
    static float* static_device = NULL;
    static size_t static_size = 0;

    if(sizeof(Real) != sizeof(float))
    {
        if(static_size < size)
        {
            cuda_realloc_in_place((void**) &static_device, size*sizeof(float), static_size*sizeof(float), 0);
            static_size = size;
        }

        if(modify == MODIFY_UPLOAD)
        {
            //Upload: host -> static -> device
            CUDA_DEBUG_TEST(hipMemcpy(static_device, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
            kernel_double_from_float((double*) (void*) device_memory, static_device, size);
        }
        else
        {
            //download: device -> static -> host
            kernel_float_from_double(static_device, (double*) (void*) device_memory, size);
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, static_device, size*sizeof(float), hipMemcpyDeviceToHost));
        }
    }
    else
    {
        if(modify == MODIFY_UPLOAD)
            CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
        else
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size*sizeof(float), hipMemcpyDeviceToHost));
    }
}


extern "C" int  sim_solver_reinit(Sim_Solver* solver, Solver_Type type, int n, int m)
{
    if(solver->type != type && solver->type != SOLVER_TYPE_NONE)
        sim_solver_reinit(solver, solver->type, 0, 0);

    int out = 0;
    switch(type) {
        case SOLVER_TYPE_NONE: {
            n = 0;
            m = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            out = explicit_solver_resize(&solver->expli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            out = semi_implicit_solver_resize(&solver->impli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            out = semi_implicit_coupled_solver_resize(&solver->impli_coupled, n, m);
        } break;

        default: {
            assert(false);
        }
    };

    solver->type = type;
    solver->m = m;
    solver->n = n;
    return out;
}

void sim_state_reinit(Sim_State* states, Solver_Type type, int n, int m)
{
    if(states->type != type && states->type != SOLVER_TYPE_NONE)
        sim_state_reinit(states, states->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            n = 0;
            m = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            explicit_state_resize(&states->expli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_state_resize(&states->impli, n, m);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_state_resize(&states->impli_coupled, n, m);
        } break;

        default: {
            assert(false);
        }
    };

    states->type = type;
    states->m = m;
    states->n = n;
}

extern "C" void sim_states_reinit(Sim_State* states, int state_count, Solver_Type type, int n, int m)
{
    for(int i = 0; i < state_count; i++)
        sim_state_reinit(&states[i], type, n, m);
}


void _switch(Solver_Type type)
{
    switch(type) {
        case SOLVER_TYPE_NONE: {

        } break;

        case SOLVER_TYPE_EXPLICIT: {
            
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {

        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {

        } break;

        default: assert(false);
    };
}


extern "C" void sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Allen_Cahn_Params params, bool do_debug)
{

    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            // nothing
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            if(states_count < EXPLICIT_SOLVER_REQUIRED_HISTORY)
                printf("explicit solver requires bigger history\n");
            else
            {
                Sim_State state = states[iter % states_count];
                Sim_State next_state = states[(iter + 1) % states_count];
                if(state.type != SOLVER_TYPE_EXPLICIT || next_state.type != SOLVER_TYPE_EXPLICIT)
                    printf("not matching state provided to explicit solver\n");
                else
                {
                    explicit_solver_rk4_step(&solver->expli, state.expli, &next_state.expli, params, iter, do_debug);
                    // explicit_solver_newton_step(&solver->expli, state.expli, next_state.expli, params, iter, do_debug);
                }
            }
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            if(states_count < EXPLICIT_SOLVER_REQUIRED_HISTORY)
                printf("explicit solver requires bigger history");
            else
            {
                Sim_State state = states[iter % states_count];
                Sim_State next_state = states[(iter + 1) % states_count];
                if(state.type != SOLVER_TYPE_SEMI_IMPLICIT || next_state.type != SOLVER_TYPE_SEMI_IMPLICIT)
                    printf("not matching state provided to semi implicit solver\n");
                else
                    semi_implicit_solver_step(&solver->impli, state.impli, next_state.impli, params, iter, do_debug);
            }
        } break;


        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            if(states_count < EXPLICIT_SOLVER_REQUIRED_HISTORY)
                printf("explicit solver requires bigger history");
            else
            {
                Sim_State state = states[iter % states_count];
                Sim_State next_state = states[(iter + 1) % states_count];
                if(state.type != SOLVER_TYPE_SEMI_IMPLICIT_COUPLED || next_state.type != SOLVER_TYPE_SEMI_IMPLICIT_COUPLED)
                    printf("not matching state provided to semi implicit solver\n");
                else
                    semi_implicit_coupled_solver_step(&solver->impli_coupled, state.impli_coupled, next_state.impli_coupled, params, iter, do_debug);
            }
        } break;

        default: assert(false);
    };
}

extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count)
{
    if(states_count <= 0 || map_count <= 0)
        return;

    Sim_State state = states[MOD(iter, states_count)];
    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            //none
        } break;

        case SOLVER_TYPE_EXPLICIT: {
            explicit_solver_get_maps(&solver->expli, state.expli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_get_maps(&solver->impli, state.impli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_solver_get_maps(&solver->impli_coupled, state.impli_coupled, maps, map_count);
        } break;

        default: assert(false);
    };
}