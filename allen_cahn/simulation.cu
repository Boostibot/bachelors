#include "hip/hip_runtime.h"
// ==================== SWITCHES ==========================

// #define COMPILE_BENCHMARKS
// #define COMPILE_TESTS
#define COMPILE_SIMULATION
// #define COMPILE_THRUST

#define USE_CUSTOM_REDUCE
#define USE_TILED_FOR 
#define USE_FLOATS

//==================== IMPLEMENTATION ===================
#ifdef COMPILE_TESTS
#define TEST_CUDA_ALL
#endif

#include "simulation.h"
#include "cuda_util.cuh"
#include "cuda_reduction.cuh"
#include "cuda_for.cuh"
#include "cuda_random.cuh"

#ifdef COMPILE_SIMULATION

#ifndef USE_CUSTOM_REDUCE
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#endif

#if 1
SHARED Real custom_hypot(Real y, Real x) { return (Real) hypotf((float)y, (float)x); }
SHARED Real custom_atan2(Real y, Real x) { return (Real) atan2f((float) y, (float) x); }
SHARED Real custom_cos(Real theta) { return (Real) cosf((float)theta); }
#else
SHARED Real custom_hypot(Real y, Real x) { return hypot(y, x); }
SHARED Real custom_atan2(Real y, Real x) { return atan2(y, x); }
SHARED Real custom_cos(Real theta) { return cos(theta); }
#endif

Real vector_dot_product(const Real *a, const Real *b, int ny)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_dot_product(a, b, ny);
    #else
    // wrap raw pointers to device memory with device_ptr
    thrust::device_ptr<const Real> d_a(a);
    thrust::device_ptr<const Real> d_b(b);

    // inner_product implements a mathematical dot product
    return thrust::inner_product(d_a, d_a + ny, d_b, 0.0);
    #endif
}

Real vector_max(const Real *a, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_max(a, N);
    #else
    thrust::device_ptr<const Real> d_a(a);
    return *(thrust::max_element(d_a, d_a + N));
    #endif
}

Real vector_get_l2_dist(const Real* a, const Real* b, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_distance(a, b, N)/ sqrt((Real) N);
    #else
    Cache_Tag tag = cache_tag_make();
    Real* temp = cache_alloc(Real, N, &tag);
    cuda_for(0, N, [=]SHARED(int i){
        temp[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_dot_product(temp, temp, N);
    Real error = sqrt(temp_dot_temp/N);
    cache_free(&tag);
    return error;
    #endif
}

Real vector_get_max_dist(const Real* a, const Real* b, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_distance(a, b, N);
    #else
    Cache_Tag tag = cache_tag_make();
    Real* temp = cache_alloc(Real, N, &tag);
    cuda_for(0, N, [=]SHARED(int i){
        temp[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_max(temp, N);
    Real error = sqrt(temp_dot_temp/N);
    cache_free(&tag);
    return error;
    #endif
}


Real vector_euclid_norm(const Real* vector, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_norm(vector, N)/ sqrt((Real) N);
    #else
    Real dot = vector_dot_product(vector, vector, N);
    return sqrt(dot / N);
    #endif
}


SHARED Real* at_mod(Real* map, int x, int y, int nx, int ny)
{
    #define AT_MOD_MODE 1
    #if AT_MOD_MODE == 0
        //95 ms
        int x_mod = MOD(x, nx);
        int y_mod = MOD(y, ny);
    #elif AT_MOD_MODE == 1
        //63 ms
        //@NOTE: this only works for x, y maximaly nx, ny respectively otuside of their proper range.
        // In our application this is enough.
        //@NOTE: this seems to be the fastest mode
        int x_mod = x;
        if(x_mod < 0)
            x_mod += nx;
        else if(x_mod >= nx)
            x_mod -= nx;

        int y_mod = y;
        if(y_mod < 0)
            y_mod += ny;
        else if(y_mod >= ny)
            y_mod -= ny;

    #elif AT_MOD_MODE == 2
        //85 ms
        int x_mod = (x + nx) % nx;
        int y_mod = (y + ny) % ny;
    #endif
    return &map[x_mod + y_mod*nx];
}

void explicit_solver_resize(Explicit_Solver* solver, int nx, int ny)
{
    size_t N = (size_t)nx*(size_t)ny;
    size_t N_old = (size_t)solver->nx*(size_t)solver->ny;
    if(solver->nx != nx || solver->ny != ny)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real* debug_maps = (Real*) (void*) &solver->debug_maps;
        for(uint i = 0; i < sizeof(solver->debug_maps) / sizeof(Real*); i++)
            cuda_realloc_in_place((void**) &debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void explicit_state_resize(Explicit_State* state, int nx, int ny)
{
    size_t N = (size_t)nx*(size_t)ny;
    size_t N_old = (size_t)state->nx*(size_t)state->ny;
    if(state->nx != nx || state->ny != ny)
    {
        cuda_realloc_in_place((void**) &state->F, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &state->U, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        state->nx = nx;
        state->ny = ny;
    }
}

struct Bundled {
    Real Phi;
    Real T;
};

struct Explicit_Solve_Result {
    Real dt_Phi;
    Real dt_T;
};

struct Explicit_Solve_Debug {
    Real grad_Phi;
    Real grad_T;
    Real g_theta;
    Real theta;
    Real reaction_term;
};

SHARED Real f0(Real phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}

extern "C" void explicit_solver_newton_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    int nx = params.nx;
    int ny = params.ny;

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; //Latent heat, not L0 (sym size) ! 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; //anisotrophy strength
    Real m0 = (Real) params.m0; //anisotrophy frequency (?)
    Real theta0 = (Real) params.theta0;
    bool do_corrector_guess = params.do_corrector_guess;

    Real dx = (Real) params.L0 / nx;
    Real dy = (Real) params.L0 / ny;
    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);

    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    Real* in_F = state.F;
    Real* in_U = state.U;

    Real* out_F = next_state->F;
    Real* out_U = next_state->U;

    // Cache_Tag tag = cache_tag_make();
    // int N = params.nx * params.ny;
    // Real* out_F = cache_alloc(Real, N, &tag);
    // Real* out_U = cache_alloc(Real, N, &tag);

    cuda_tiled_for_2D<1, 1, Bundled>(0, 0, params.nx, params.ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
            csize x_mod = x;
            csize y_mod = y;

            if(x_mod < 0)
                x_mod += nx;
            else if(x_mod >= nx)
                x_mod -= nx;

            if(y_mod < 0)
                y_mod += ny;
            else if(y_mod >= ny)
                y_mod -= ny;

            int I = x_mod + y_mod*nx;

            Real Phi = in_F[I];
            Real T = in_U[I];
            return Bundled{Phi, T};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
            Bundled C = shared[tx   + ty*tile_size_x];
            Bundled E = shared[tx+1 + ty*tile_size_x];
            Bundled W = shared[tx-1 + ty*tile_size_x];
            Bundled N = shared[tx   + (ty+1)*tile_size_x];
            Bundled S = shared[tx   + (ty-1)*tile_size_x];

            Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
            Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dx;
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

            Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
            Real laplace_T = (W.T - 2*C.T + E.T)*one_over_dx2 +     (S.T - 2*C.T + N.T)*one_over_dy2;

            Real k0 = g_theta*f0(C.Phi)*k0_factor;
            Real k2 = grad_Phi_norm*k2_factor;
            Real k1 = g_theta*k1_factor;
            Real corr = 1 + k2*dt_L;

            Real dt_Phi = 0;
            if(do_corrector_guess)
                dt_Phi = (k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T))/corr;
            else
                dt_Phi = k1*laplace_Phi + k0 - k2*(C.T - Tm);

            Real dt_T = laplace_T + L*dt_Phi; 

            out_F[x + y*nx] = C.Phi + dt_Phi*dt;
            out_U[x + y*nx] = C.T + dt_T*dt;
        });

    // CUDA_DEBUG_TEST(hipMemcpy(next_state->F, out_F, N*sizeof(Real), hipMemcpyDeviceToDevice));
    // CUDA_DEBUG_TEST(hipMemcpy(next_state->U, out_U, N*sizeof(Real), hipMemcpyDeviceToDevice));
    // cache_free(&tag);
}

struct Explicit_Blend_State {
    Real weight;
    Explicit_State state;
};

template<typename ... States>
void explicit_solver_solve_lin_combination(Explicit_State* out, Allen_Cahn_Params params, States... state_args)
{
    int nx = params.nx;
    int ny = params.ny;
    Real* out_F = out->F;
    Real* out_U = out->U;

    constexpr int state_count = (int) sizeof...(state_args);
    Explicit_Blend_State states[(size_t) state_count] = {state_args...};

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; //Latent heat, not L0 (sym size) ! 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; //anisotrophy strength
    Real m0 = (Real) params.m0; //anisotrophy frequency (?)
    Real theta0 = (Real) params.theta0;
    bool do_corrector_guess = params.do_corrector_guess;

    Real dx = (Real) params.L0 / nx;
    Real dy = (Real) params.L0 / ny;
    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);

    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    #if 1
    cuda_tiled_for_2D<1, 1, Bundled>(0, 0, nx, ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
            csize x_mod = x;
            csize y_mod = y;

            if(x_mod < 0)
                x_mod += nx;
            else if(x_mod >= nx)
                x_mod -= nx;

            if(y_mod < 0)
                y_mod += ny;
            else if(y_mod >= ny)
                y_mod -= ny;

            int I = x_mod + y_mod*nx;

            Real T = 0;
            Real Phi = 0;
            #pragma unroll
            for(int i = 0; i < state_count; i++)
            {
                Phi += states[i].state.F[I] * states[i].weight;
                T += states[i].state.U[I] * states[i].weight;
            }

            return Bundled{Phi, T};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
            Bundled C = shared[tx   + ty*tile_size_x];
            Bundled E = shared[tx+1 + ty*tile_size_x];
            Bundled W = shared[tx-1 + ty*tile_size_x];
            Bundled N = shared[tx   + (ty+1)*tile_size_x];
            Bundled S = shared[tx   + (ty-1)*tile_size_x];

            Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
            Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dx;
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

            Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
            Real laplace_T = (W.T - 2*C.T + E.T)*one_over_dx2 +     (S.T - 2*C.T + N.T)*one_over_dy2;

            Real k0 = g_theta*f0(C.Phi)*k0_factor;
            Real k2 = grad_Phi_norm*k2_factor;
            Real k1 = g_theta*k1_factor;
            Real corr = 1 + k2*dt_L;

            Real dt_Phi = 0;
            if(do_corrector_guess)
                dt_Phi = (k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T))/corr;
            else
                dt_Phi = k1*laplace_Phi + k0 - k2*(C.T - Tm);

            Real dt_T = laplace_T + L*dt_Phi; 

            out_F[x + y*nx] = dt_Phi;
            out_U[x + y*nx] = dt_T;
        });
    #else

    Cache_Tag tag = cache_tag_make();
    int N = params.nx * params.ny;
    Real* F = cache_alloc(Real, N, &tag);
    Real* U = cache_alloc(Real, N, &tag);
    Real* laplace_U = cache_alloc(Real, N, &tag);
    Real* laplace_F = cache_alloc(Real, N, &tag);
    Real* grad_F_x_2dx = cache_alloc(Real, N, &tag);
    Real* grad_F_y_2dy = cache_alloc(Real, N, &tag);

    cuda_tiled_for_2D<1, 1, Bundled>(0, 0, params.nx, params.ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
            csize x_mod = x;
            csize y_mod = y;

            if(x_mod < 0)
                x_mod += nx;
            else if(x_mod >= nx)
                x_mod -= nx;

            if(y_mod < 0)
                y_mod += ny;
            else if(y_mod >= ny)
                y_mod -= ny;

            int I = x_mod + y_mod*nx;

            Real T = 0;
            Real Phi = 0;
            #pragma unroll
            for(int i = 0; i < state_count; i++)
            {
                Phi += states[i].state.F[I] * states[i].weight;
                T += states[i].state.U[I] * states[i].weight;
            }

            U[I] = T;
            F[I] = Phi;
            return Bundled{Phi, T};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
            Bundled C = shared[tx   + ty*tile_size_x];
            Bundled E = shared[tx+1 + ty*tile_size_x];
            Bundled W = shared[tx-1 + ty*tile_size_x];
            Bundled N = shared[tx   + (ty+1)*tile_size_x];
            Bundled S = shared[tx   + (ty-1)*tile_size_x];

            csize i = x + y*nx;
            grad_F_x_2dx[i] = (E.Phi - W.Phi);
            grad_F_y_2dy[i] = (N.Phi - S.Phi);
            laplace_F[i] = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
            laplace_U[i] = (W.T - 2*C.T + E.T)*one_over_dx2 +       (S.T - 2*C.T + N.T)*one_over_dy2;
        });

    cuda_for(0, N, [=]SHARED(csize i){
        Real T =  U[i];
        Real Phi = F[i];

        Real grad_Phi_x = grad_F_x_2dx[i]*one_over_2dx;
        Real grad_Phi_y = grad_F_y_2dy[i]*one_over_2dy;
        Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

        Real laplace_Phi = laplace_F[i];
        Real laplace_T = laplace_U[i];

        Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
        Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

        Real k0 = g_theta*f0(Phi)*k0_factor;
        Real k2 = grad_Phi_norm*k2_factor;
        Real k1 = g_theta*k1_factor;
        Real corr = 1 + k2*dt_L;

        Real dt_Phi = 0;
        if(do_corrector_guess)
            dt_Phi = (k1*laplace_Phi + k0 - k2*(T - Tm + dt*laplace_T))/corr;
        else
            dt_Phi = k1*laplace_Phi + k0 - k2*(T - Tm);

        Real dt_T = laplace_T + L*dt_Phi; 

        out_F[i] = dt_Phi;
        out_U[i] = dt_T;
    });
    cache_free(&tag);
    #endif
}
void explicit_solver_debug_step(Explicit_Solver* solver, Explicit_State state, Allen_Cahn_Params params)
{
    int nx = params.nx;
    int ny = params.ny;
    Real* F = state.F;
    Real* U = state.U;
    Real* grad_F = solver->debug_maps.grad_phi;
    Real* grad_U = solver->debug_maps.grad_T;
    Real* aniso = solver->debug_maps.aniso_factor;
    Real* perlin = solver->debug_maps.perlin;
    Real* simplex = solver->debug_maps.simplex;
    static uint32_t _iter = 0;
    uint32_t iter = _iter++;
    Real S0 = (Real) params.S;
    Real m0 = (Real) params.m0;
    Real theta0 = (Real) params.theta0;
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        Real T = *at_mod(U, x, y, nx, ny);
        Real Phi = *at_mod(F, x, y, nx, ny);

        Real Phi_U = *at_mod(F, x, y + 1, nx, ny);
        Real Phi_D = *at_mod(F, x, y - 1, nx, ny);
        Real Phi_R = *at_mod(F, x + 1, y, nx, ny);
        Real Phi_L = *at_mod(F, x - 1, y, nx, ny);

        Real T_U = *at_mod(U, x, y + 1, nx, ny);
        Real T_D = *at_mod(U, x, y - 1, nx, ny);
        Real T_R = *at_mod(U, x + 1, y, nx, ny);
        Real T_L = *at_mod(U, x - 1, y, nx, ny);

        Real grad_Phi_x = (Phi_R - Phi_L);
        Real grad_Phi_y = (Phi_U - Phi_D);
        Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

        Real grad_T_x = (T_R - T_L);
        Real grad_T_y = (T_U - T_D);
        Real grad_T_norm = custom_hypot(grad_T_x, grad_T_y);
        
        Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
        Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

        grad_F[x + y*nx] = grad_Phi_norm;
        grad_U[x + y*nx] = grad_T_norm; 
        aniso[x + y*nx] = g_theta;
    });

    perlin2d_generate(perlin, nx, ny, 1, 1, 8, iter);
    simplex2d_generate(simplex, nx, ny, 3, 3, 8, iter);
}

void explicit_solver_rk4_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    Explicit_State steps[4] = {0};
    for(int i = 0; i < (int) STATIC_ARRAY_SIZE(steps); i++)
    {
        steps[i].F = cache_alloc(Real, N, &tag);
        steps[i].U = cache_alloc(Real, N, &tag);
        steps[i].nx = params.nx;
        steps[i].ny = params.ny;
    }

    Explicit_State k1 = steps[0];
    Explicit_State k2 = steps[1];
    Explicit_State k3 = steps[2];
    Explicit_State k4 = steps[3];

    Real dt = (Real) params.dt;
    using W = Explicit_Blend_State;
    explicit_solver_solve_lin_combination(&k1, params, W{1, state});
    explicit_solver_solve_lin_combination(&k2, params, W{1, state}, W{dt * (Real) 0.5, k1});
    explicit_solver_solve_lin_combination(&k3, params, W{1, state}, W{dt * (Real) 0.5, k2});
    explicit_solver_solve_lin_combination(&k4, params, W{1, state}, W{dt * 1, k3});

    Real* out_F = next_state->F;
    Real* out_U = next_state->U;
    cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
        out_F[i] =  state.F[i] + dt/6*(k1.F[i] + 2*k2.F[i] + 2*k3.F[i] + k4.F[i]);
        out_U[i] =  state.U[i] + dt/6*(k1.U[i] + 2*k2.U[i] + 2*k3.U[i] + k4.U[i]);
    });

    if(do_debug)
        explicit_solver_debug_step(solver, state, params);

    cache_free(&tag);
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

double explicit_solver_rk4_adaptive_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    static Real _initial_step = 0;
    if(iter == 0)
        _initial_step = (Real) params.dt;

    Real tau = _initial_step;
    Explicit_State steps[5] = {0};
    for(int i = 0; i < (int) STATIC_ARRAY_SIZE(steps); i++)
    {
        steps[i].F = cache_alloc(Real, N, &tag);
        steps[i].U = cache_alloc(Real, N, &tag);
        steps[i].nx = params.nx;
        steps[i].ny = params.ny;
    }

    Real* Epsilon_F = cache_alloc(Real, N, &tag);
    Real* Epsilon_U = cache_alloc(Real, N, &tag);
    Real epsilon_F = 0;
    Real epsilon_U = 0;

    Explicit_State k1 = steps[0];
    Explicit_State k2 = steps[1];
    Explicit_State k3 = steps[2];
    Explicit_State k4 = steps[3];
    Explicit_State k5 = steps[4];

    using W = Explicit_Blend_State;
    explicit_solver_solve_lin_combination(&k1, params, W{1, state});

    bool converged = false;
    int i = 0;
    int max_iters = MAX(MAX(params.T_max_iters, params.Phi_max_iters), 1);
    Real used_tau = tau;
    for(; i < max_iters && converged == false; i++)
    {
        // k1 = f(t, x);
        // k2 = f(t + tau/3, x + tau/3*k1);
        // k3 = f(t + tau/3, x + tau/6*(k1 + k2));
        // k4 = f(t + tau/2, x + tau/8*(k1 + 3*k3));
        // k5 = f(t + tau/1, x + tau*(0.5f*k1 - 1.5f*k3 + 2*k4));
        
        // k1 = f(x);
        // k2 = f(x + tau/3*k1);
        // k3 = f(x + tau/6*k1 + tau/6*k2);
        // k4 = f(x + tau/8*k1 + tau*3/8*k3));
        // k5 = f(x + tau/2*k1 - tau*3/2*k3 + tau*2*k4));
        
        explicit_solver_solve_lin_combination(&k2, params, W{1, state}, W{tau/3, k1});
        explicit_solver_solve_lin_combination(&k3, params, W{1, state}, W{tau/6, k1}, W{tau/6, k2});
        explicit_solver_solve_lin_combination(&k4, params, W{1, state}, W{tau/8, k1}, W{tau*3/8, k3});
        explicit_solver_solve_lin_combination(&k5, params, W{1, state}, W{tau/2, k1}, W{-tau*3/2, k3}, W{tau*2, k4});

        cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
            Real F = (Real)0.2*k1.F[i] - (Real)0.9*k3.F[i] + (Real)0.8*k4.F[i] - (Real)0.1*k5.F[i];
            Real U = (Real)0.2*k1.U[i] - (Real)0.9*k3.U[i] + (Real)0.8*k4.U[i] - (Real)0.1*k5.U[i];

            Epsilon_F[i] = F >= 0 ? F : -F;
            Epsilon_U[i] = U >= 0 ? U : -U;
        });

        epsilon_F = vector_max(Epsilon_F, N);
        epsilon_U = vector_max(Epsilon_U, N);

        if(epsilon_F < params.Phi_tolerance && epsilon_U < params.T_tolerance)
            converged = true;

        Real epsilon = (Real) MAX(epsilon_F + epsilon_U, 1e-8);
        Real delta = (Real) MAX(params.Phi_tolerance + params.T_tolerance, 1e-8);
        used_tau = tau;
        tau = pow(delta / epsilon, (Real)0.2)*4/5*tau;
    }

    Real* next_F = next_state->F;
    Real* next_U = next_state->U;
    cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
        next_F[i] = state.F[i] + used_tau*((Real)1.0/6*(k1.F[i] + k5.F[i]) + (Real)2.0/3*k4.F[i]);
        next_U[i] = state.U[i] + used_tau*((Real)1.0/6*(k1.U[i] + k5.U[i]) + (Real)2.0/3*k4.U[i]);
    });

    LOG("SOLVER", converged ? LOG_DEBUG : LOG_WARN, "rk4-adaptive %s in %i iters with error F:%lf | U:%lf | tau:%e", converged ? "converged" : "diverged", i, (double) epsilon_F, (double) epsilon_U, (double)used_tau);
    _initial_step = tau;

    if(do_debug)
        explicit_solver_debug_step(solver, state, params);

    cache_free(&tag);
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
    return (double) used_tau;
}

double explicit_solver_choose(Solver_Type type, Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter)
{
    if(type == SOLVER_TYPE_EXPLICIT)
    {
        explicit_solver_newton_step(solver, state, next_state, params, iter, params.do_debug);
        return params.dt;
    }
    if(type == SOLVER_TYPE_EXPLICIT_RK4)
    {
        explicit_solver_rk4_step(solver, state, next_state, params, iter, params.do_debug);
        return params.dt;
    }
    if(type == SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE)
    {
        return explicit_solver_rk4_adaptive_step(solver, state, next_state, params, iter, params.do_debug);
    }

    assert(false);
    return false;
}


double explicit_solver_choose_and_copute_step_residual(Solver_Type type, Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Allen_Cahn_Params params, size_t iter, Allen_Cahn_Stats* stats_or_null)
{
    double advance_by = explicit_solver_choose(type, solver, state, next_state, params, iter);
    if(params.do_stats_step_residual && stats_or_null)
    {
        Explicit_State combined_state = state;
        combined_state.U = next_state->U;

        Explicit_State corrected_next = {0};
        Cache_Tag tag = cache_tag_make();
        int N = params.ny * params.nx;
        corrected_next.F = cache_alloc(Real, N, &tag);
        corrected_next.U = cache_alloc(Real, N, &tag);
        corrected_next.nx = params.nx;
        corrected_next.ny = params.ny;

        Allen_Cahn_Params changed_params = params;
        changed_params.do_debug = false;
        changed_params.do_stats = false;
        changed_params.do_stats_step_residual = false;

        explicit_solver_choose(type, solver, combined_state, &corrected_next, changed_params, iter);
        stats_or_null->L2_step_residuals[0] = vector_get_l2_dist(corrected_next.F, next_state->F, N);
        stats_or_null->Lmax_step_residuals[0] = vector_get_max_dist(corrected_next.F, next_state->F, N);
        stats_or_null->step_residuals = 1;

        LOG_DEBUG("SOLVER", "%lli step residual | avg: %e | max: %e", (long long) iter, (double) stats_or_null->L2_step_residuals[0], (double) stats_or_null->Lmax_step_residuals[0]);

        cache_free(&tag);
    }

    return advance_by;
}

void explicit_solver_get_maps(Explicit_Solver* solver, Explicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t) map_count);

    #define ASSIGN_MAP_NAMED(var_ptr, var_name) \
        if(__map_i < map_count) \
        { \
            maps[__map_i].data = var_ptr; \
            maps[__map_i].name = var_name; \
            maps[__map_i].nx = solver->nx; \
            maps[__map_i].ny = solver->ny; \
            __map_i += 1; \
        }\

    #define ASSIGN_MAP(var_ptr) ASSIGN_MAP_NAMED(var_ptr, #var_ptr) 

    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");
    ASSIGN_MAP_NAMED(solver->debug_maps.aniso_factor, "aniso_factor");
    ASSIGN_MAP_NAMED(solver->debug_maps.reaction, "reaction");
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residual, "step_residual");
    ASSIGN_MAP_NAMED(solver->debug_maps.perlin, "perlin");
    ASSIGN_MAP_NAMED(solver->debug_maps.simplex, "simplex");
}

struct Cross_Matrix_Static {
    Real C;
    Real U;
    Real D;
    Real L;
    Real R;

    int nx;
    int ny;
};

struct Cross_Matrix {
    Real* C;
    Real* U;
    Real* D;
    Real* L;
    Real* R;

    int nx;
    int ny;
};

struct Anisotrophy_Matrix {
    Real* anisotrophy;
    Real X;
    Real Y;
    Real C_minus_one;

    int nx;
    int ny;
};

void cross_matrix_static_multiply(Real* out, const void* _A, const Real* vec, int N)
{
    Cross_Matrix_Static A = *(Cross_Matrix_Static*)_A;
    int nx = A.nx;
    int ny = A.ny;

    #ifdef USE_TILED_FOR
    cuda_tiled_for_2D_modular<1, 1, TILED_FOR_PERIODIC_SMALL_R>(vec, nx, ny, 
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
            Real val = shared[tx + ty*tile_size_x]*A.C;
            val += shared[tx+1 + ty*tile_size_x]*A.R;
            val += shared[tx-1 + ty*tile_size_x]*A.L;
            val += shared[tx   + (ty+1)*tile_size_x]*A.U;
            val += shared[tx   + (ty-1)*tile_size_x]*A.D;

            out[x + y*nx] = val;
        }
    );
    #else
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;
        Real val = vec[i]*A.C;
        val += *at_mod((Real*) vec, x+1, y, nx, ny)*A.R;
        val += *at_mod((Real*) vec, x-1, y, nx, ny)*A.L;
        val += *at_mod((Real*) vec, x, y+1, nx, ny)*A.U;
        val += *at_mod((Real*) vec, x, y-1, nx, ny)*A.D;
        out[i] = val;
    });
    #endif
}

void cross_matrix_multiply(Real* out, const void* _A, const Real* x, int N)
{
    Cross_Matrix A = *(Cross_Matrix*)_A;
    int nx = A.nx;
    cuda_for(0, N, [=]SHARED(int i){
        Real val = 0;
        val += x[i]*A.C[i];
        if(i+1 < N)  val += x[i+1]*A.R[i];
        if(i-1 >= 0) val += x[i-1]*A.L[i];
        if(i+nx < N)  val += x[i+nx]*A.U[i];
        if(i-nx >= 0) val += x[i-nx]*A.D[i];

        out[i] = val;
    });
}

void anisotrophy_matrix_multiply(Real* out, const void* _A, const Real* vec, int N)
{
    Anisotrophy_Matrix A = * (Anisotrophy_Matrix*)_A;
    int nx = A.nx;
    int ny = A.ny;

    #ifdef USE_TILED_FOR
    cuda_tiled_for_2D_modular<1, 1, TILED_FOR_PERIODIC_SMALL_R>(vec, nx, ny, 
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
            int i = x + y*nx;
            Real s = A.anisotrophy[i];
            Real X = A.X*s;
            Real Y = A.Y*s;
            Real C = 1 + A.C_minus_one*s;

            Real val = shared[tx + ty*tile_size_x]*C;
            val += shared[tx+1 + ty*tile_size_x]*X;
            val += shared[tx-1 + ty*tile_size_x]*X;
            val += shared[tx   + (ty+1)*tile_size_x]*Y;
            val += shared[tx   + (ty-1)*tile_size_x]*Y;

            out[i] = val;
        }
    );
    #else
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;
        Real s = A.anisotrophy[i];
        Real X = A.X*s;
        Real Y = A.Y*s;
        Real C = 1 + A.C_minus_one*s;

        Real val = vec[i]*C;
        val += *at_mod((Real*) vec, x+1, y, nx, ny)*X;
        val += *at_mod((Real*) vec, x-1, y, nx, ny)*X;
        val += *at_mod((Real*) vec, x, y+1, nx, ny)*Y;
        val += *at_mod((Real*) vec, x, y-1, nx, ny)*Y;
        out[i] = val;
    });
    #endif
}

typedef struct Conjugate_Gardient_Params {
    Real epsilon;
    Real tolerance;
    int max_iters;

    Real* initial_value_or_null;
} Conjugate_Gardient_Params;

typedef struct Conjugate_Gardient_Convergence {
    Real error;
    int iters;
    bool converged;
} Conjugate_Gardient_Convergence;

typedef void(*Matrix_Vector_Mul_Func)(Real* out, const void* A, const Real* x, int N);

Conjugate_Gardient_Convergence conjugate_gradient_solve(const void* A, Real* x, const Real* b, int N, Matrix_Vector_Mul_Func matrix_mul_func, const Conjugate_Gardient_Params* params_or_null)
{
    i64 start = clock_ns();
    Conjugate_Gardient_Params params = {0};
    params.epsilon = (Real) 1.0e-10;
    params.tolerance = (Real) 1.0e-5;
    params.max_iters = 10;
    if(params_or_null)
        params = *params_or_null;

    Cache_Tag tag = cache_tag_make();

    Real scaled_squared_tolerance = params.tolerance*params.tolerance*N;
    Real* r = cache_alloc(Real, N, &tag);
    Real* p = cache_alloc(Real, N, &tag);
    Real* Ap = cache_alloc(Real, N, &tag);
    Real r_dot_r = 0;

    //@TODO: IMPLEMENT FULLY (add launch params for reductions etc.)!
    static hipStream_t stream1 = NULL;
    static hipStream_t stream2 = NULL;
    static hipStream_t stream3 = NULL;
    static hipStream_t stream4 = NULL;
    if(stream1 == NULL)
    {
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
        hipStreamCreate(&stream4);
    }

    //@TODO: streams
    if(params.initial_value_or_null)
    {
        CUDA_DEBUG_TEST(hipMemcpyAsync(x, params.initial_value_or_null, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice));
        matrix_mul_func(Ap, A, params.initial_value_or_null, N);
        cuda_for(0, N, [=]SHARED(int i){
            r[i] = b[i] - Ap[i];
            p[i] = r[i];
        });

        r_dot_r = vector_dot_product(r, r, N);
    }
    else
    {
        CUDA_DEBUG_TEST(hipMemsetAsync(x, 0, sizeof(Real)*(size_t)N, stream1));
        CUDA_DEBUG_TEST(hipMemcpyAsync(r, b, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice, stream2));
        CUDA_DEBUG_TEST(hipMemcpyAsync(p, b, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice, stream3));

        r_dot_r = vector_dot_product(b, b, N);
        // CUDA_DEBUG_TEST(hipDeviceSynchronize());
    }

    int iter = 0;
    if(r_dot_r >= scaled_squared_tolerance || iter == 0)
    {
        for(; iter < params.max_iters; iter++)
        {
            matrix_mul_func(Ap, A, p, N);
            
            Real p_dot_Ap = vector_dot_product(p, Ap, N);
            Real alpha = r_dot_r / MAX(p_dot_Ap, params.epsilon);
            
            cuda_for(0, N, [=]SHARED(int i){
                x[i] = x[i] + alpha*p[i];
                r[i] = r[i] - alpha*Ap[i];
            });

            Real r_dot_r_new = vector_dot_product(r, r, N);
            if(r_dot_r_new < scaled_squared_tolerance)
            {
                r_dot_r = r_dot_r_new;
                break;
            }

            Real beta = r_dot_r_new / MAX(r_dot_r, params.epsilon);
            cuda_for(0, N, [=]SHARED(int i){
                p[i] = r[i] + beta*p[i]; 
            });

            r_dot_r = r_dot_r_new;
        }
    }

    Conjugate_Gardient_Convergence out = {0};
    out.iters = iter;
    out.converged = iter != params.max_iters;
    out.error = sqrt(r_dot_r/N);

    i64 end = clock_ns();
    LOG_DEBUG("KERNEL", "conjugate_gradient_solve(%lli) took: %.2ems", (lli)N, (double)(end - start)*1e-6);

    cache_free(&tag);
    return out;
} 

void matrix_multiply(Real* output, const Real* A, const Real* B, int A_height, int A_width, int B_height, int B_width)
{
    assert(A_width == B_height);
    for(int y = 0; y < A_height; y++)
    {
        for(int x = 0; x < B_width; x++)
        {
            Real val = 0;
            for(int k = 0; k < A_width; k++)
                val += A[k + y*A_width]*B[x + k*B_width];

            output[x + y*B_width] = val;
        }
    }
}

void semi_implicit_solver_resize(Semi_Implicit_Solver* solver, int nx, int ny)
{
    if(solver->nx != nx || solver->ny != ny)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        //@TODO: make this on demand load
        size_t N = (size_t) (ny*nx);
        size_t N_old = (size_t) (solver->ny*solver->nx);

        void** maps = (void**) (void*) &solver->maps;
        for(uint i = 0; i < sizeof(solver->maps) / sizeof(Real*); i++)
            cuda_realloc_in_place(&maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        void** debug_maps = (void**) (void*) &solver->debug_maps;
        for(uint i = 0; i < sizeof(solver->debug_maps) / sizeof(Real*); i++)
            cuda_realloc_in_place(&debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void semi_implicit_solver_step_based(Semi_Implicit_Solver* solver, Real* F, Real* U, Real* U_base, Semi_Implicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->nx;
    Real dy = (Real) params.L0 / solver->ny;

    int nx = solver->nx;
    int ny = solver->ny;
    int N = nx*ny;

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; 
    Real m0 = (Real) params.m0; 
    Real theta0 = (Real) params.theta0;
    Real gamma = (Real) params.gamma;
    
    Real* F_next = next_state.F;
    Real* U_next = next_state.U;
    
    Real* b_F = solver->maps.b_F;
    Real* b_U = solver->maps.b_U;

    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);
    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    Anisotrophy_Matrix A_F = {0};
    A_F.anisotrophy = solver->maps.anisotrophy;
    A_F.C_minus_one = 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_F.X = -dt/(dx*dx);
    A_F.Y = -dt/(dy*dy);
    A_F.nx = nx;
    A_F.ny = ny;

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);
    A_U.nx = nx;
    A_U.ny = ny;

    bool do_corrector_guess = params.do_corrector_guess;
    bool is_tiled = true;
    Cache_Tag tag = cache_tag_make();

    //@TODO: factor out
    static hipEvent_t start = NULL;
    static hipEvent_t stop = NULL;
    if(start == NULL || stop == NULL)
    {
        CUDA_TEST(hipEventCreate(&start));
        CUDA_TEST(hipEventCreate(&stop));
    }
    CUDA_TEST(hipEventRecord(start, 0));

    if(do_corrector_guess)
    {
        #if 1
        Real* laplace_U = cache_alloc(Real, N, &tag);
        Real* laplace_F = cache_alloc(Real, N, &tag);
        Real* grad_F_x_2dx = cache_alloc(Real, N, &tag);
        Real* grad_F_y_2dy = cache_alloc(Real, N, &tag);

        cuda_tiled_for_2D<1, 1, Bundled>(0, 0, nx, ny,
            [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
                csize x_mod = x;
                csize y_mod = y;

                if(x_mod < 0)
                    x_mod += nx;
                else if(x_mod >= nx)
                    x_mod -= nx;

                if(y_mod < 0)
                    y_mod += ny;
                else if(y_mod >= ny)
                    y_mod -= ny;

                Real T = (Real) U[x_mod + y_mod*nx];
                Real Phi = (Real) F[x_mod + y_mod*ny];
                return Bundled{Phi, T};
            },
            [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
                Bundled C = shared[tx   + ty*tile_size_x];
                Bundled E = shared[tx+1 + ty*tile_size_x];
                Bundled W = shared[tx-1 + ty*tile_size_x];
                Bundled N = shared[tx   + (ty+1)*tile_size_x];
                Bundled S = shared[tx   + (ty-1)*tile_size_x];

                csize i = x + y*nx;
                grad_F_x_2dx[i] = (E.Phi - W.Phi);
                grad_F_y_2dy[i] = (N.Phi - S.Phi);
                laplace_F[i] = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
                laplace_U[i] = (W.T - 2*C.T + E.T)*one_over_dx2 +       (S.T - 2*C.T + N.T)*one_over_dy2;
            }
        );
        
        cuda_for(0, N, [=]SHARED(csize i){
            Real T =  U[i];
            Real Phi = F[i];

            Real grad_Phi_x = grad_F_x_2dx[i]*one_over_2dx;
            Real grad_Phi_y = grad_F_y_2dy[i]*one_over_2dy;
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

            Real k0 = g_theta*f0(Phi)*k0_factor;
            Real k2 = grad_Phi_norm*k2_factor;
            Real k1 = g_theta*k1_factor;
            Real corr = 1 + k2*dt_L;

            Real right = Phi + dt/corr*((1-gamma)*k1*laplace_F[i] + k0 - k2*(T - Tm + dt*laplace_U[i]));
            Real factor = gamma/corr*k1; 

            A_F.anisotrophy[i] = (Real) factor;
            b_F[i] = (Real) right;
        });
        
        #else
        cuda_tiled_for_2D<1, 1, Bundled>(0, 0, nx, ny,
            [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
                csize x_mod = x;
                csize y_mod = y;

                if(x_mod < 0)
                    x_mod += nx;
                else if(x_mod >= nx)
                    x_mod -= nx;

                if(y_mod < 0)
                    y_mod += ny;
                else if(y_mod >= ny)
                    y_mod -= ny;

                Real T = (Real) U[x_mod + y_mod*nx];
                Real Phi = (Real) F[x_mod + y_mod*ny];
                return Bundled{Phi, T};
            },
            [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
                Bundled C = shared[tx   + ty*tile_size_x];
                Bundled E = shared[tx+1 + ty*tile_size_x];
                Bundled W = shared[tx-1 + ty*tile_size_x];
                Bundled N = shared[tx   + (ty+1)*tile_size_x];
                Bundled S = shared[tx   + (ty-1)*tile_size_x];

                Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
                Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dy;
                Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

                Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
                Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

                Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
                Real laplace_T =   (W.T - 2*C.T + E.T)*one_over_dx2 +       (S.T - 2*C.T + N.T)*one_over_dy2;

                Real k0 = g_theta*f0(C.Phi)*k0_factor;
                Real k2 = grad_Phi_norm*k2_factor;
                Real k1 = g_theta*k1_factor;
                Real corr = 1 + k2*dt_L;

                Real right = C.Phi + dt/corr*((1-gamma)*k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T));
                Real factor = gamma/corr*k1; 

                A_F.anisotrophy[x+y*nx] = (Real) factor;
                b_F[x + y*nx] = (Real) right;
            }
        );
        #endif
    }
    else
    {
        cuda_tiled_for_2D_modular<1, 1, TILED_FOR_PERIODIC_SMALL_R>(F, nx, ny,
            [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
                Real C_T   = U[x + y*nx];
                Real C_Phi = shared[tx   + ty*tile_size_x];
                Real E_Phi = shared[tx+1 + ty*tile_size_x];
                Real W_Phi = shared[tx-1 + ty*tile_size_x];
                Real N_Phi = shared[tx   + (ty+1)*tile_size_x];
                Real S_Phi = shared[tx   + (ty-1)*tile_size_x];

                Real grad_Phi_x = (E_Phi - W_Phi)/(2*dx);
                Real grad_Phi_y = (N_Phi - S_Phi)/(2*dy);
                Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);


                Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
                Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

                Real laplace_Phi = (W_Phi - 2*C_Phi + E_Phi)*one_over_dx2 + (S_Phi - 2*C_Phi + N_Phi)*one_over_dy2;

                Real k0 = g_theta*f0(C_Phi)*k0_factor;
                Real k2 = grad_Phi_norm*k2_factor;
                Real k1 = g_theta*k1_factor;

                Real right = C_Phi + dt*((1-gamma)*k1*laplace_Phi + k0 - k2*(C_T - Tm));
                Real factor = gamma*k1; 

                A_F.anisotrophy[x+y*nx] = (Real) factor;
                b_F[x + y*nx] = (Real) right;
            }
        );
    }

    CUDA_TEST(hipEventRecord(stop, 0));
    CUDA_TEST(hipEventSynchronize(stop));

    float time = 0;
    CUDA_TEST(hipEventElapsedTime(&time, start, stop));
    LOG_DEBUG("SOLVER", "Prepare kernel time %.2ems corrector_guess:%s tiled:%s", (double)time, 
        do_corrector_guess ? "true" : "false", 
        is_tiled ? "true" : "false");

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-12;
    solver_params.tolerance = (Real) params.Phi_tolerance;
    solver_params.max_iters = params.Phi_max_iters;
    solver_params.initial_value_or_null = F;

    //Solve A_F*F_next = b_F
    Conjugate_Gardient_Convergence F_converged = conjugate_gradient_solve(&A_F, F_next, b_F, N, anisotrophy_matrix_multiply, &solver_params);
    LOG_DEBUG("SOLVER", "%lli F %s in %i iters with error %e\n", (lli) iter, F_converged.converged ? "converged" : "diverged", F_converged.iters, (double)F_converged.error);

    //Calculate b_U
    cuda_for(0, nx*ny, [=]SHARED(csize i){
        Real T =  U_base[i];
        Real Phi = F[i];
        Real Phi_next = F_next[i];

        b_U[i] = (Real) (T + L*(Phi_next - Phi) + dt*(1-gamma)*T);
    });

    solver_params.tolerance = (Real) params.T_tolerance;
    solver_params.max_iters = params.T_max_iters;
    solver_params.initial_value_or_null = U;

    //Solve A_U*U_next = b_U
    Conjugate_Gardient_Convergence U_converged = conjugate_gradient_solve(&A_U, U_next, b_U, N, cross_matrix_static_multiply, &solver_params);
    LOG_DEBUG("SOLVER", "%lli U %s in %i iters with error %e\n", (lli) iter, U_converged.converged ? "converged" : "diverged", U_converged.iters, (double)U_converged.error);

    if(do_debug)
    {
        Real* AfF = solver->debug_maps.AfF;
        Real* AuU = solver->debug_maps.AuU;
        //Back test
        if(1)
        {
            anisotrophy_matrix_multiply(AfF, &A_F, F_next, N);
            cross_matrix_static_multiply(AuU, &A_U, U_next, N);

            Real back_error_F = vector_get_l2_dist(AfF, b_F, N);
            Real back_error_U = vector_get_l2_dist(AuU, b_U, N);

            Real back_error_F_max = vector_get_max_dist(AfF, b_F, N);
            Real back_error_U_max = vector_get_max_dist(AuU, b_U, N);

            LOG_DEBUG("SOLVER", "AVG | F:%e U:%e Epsilon:%e \n", (double) back_error_F, (double) back_error_U, (double) solver_params.tolerance*2);
            LOG_DEBUG("SOLVER", "MAX | F:%e U:%e Epsilon:%e \n", (double) back_error_F_max, (double) back_error_U_max, (double) solver_params.tolerance*2);
        }

        Real* grad_F = solver->debug_maps.grad_phi;
        Real* grad_U = solver->debug_maps.grad_T;
        cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
            Real T = *at_mod(U, x, y, nx, ny);
            Real Phi = *at_mod(F, x, y, nx, ny);

            Real Phi_U = *at_mod(F, x, y + 1, nx, ny);
            Real Phi_D = *at_mod(F, x, y - 1, nx, ny);
            Real Phi_R = *at_mod(F, x + 1, y, nx, ny);
            Real Phi_L = *at_mod(F, x - 1, y, nx, ny);

            Real T_U = *at_mod(U, x, y + 1, nx, ny);
            Real T_D = *at_mod(U, x, y - 1, nx, ny);
            Real T_R = *at_mod(U, x + 1, y, nx, ny);
            Real T_L = *at_mod(U, x - 1, y, nx, ny);

            Real grad_Phi_x = (Phi_R - Phi_L);
            Real grad_Phi_y = (Phi_U - Phi_D);
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real grad_T_x = (T_R - T_L);
            Real grad_T_y = (T_U - T_D);
            Real grad_T_norm = custom_hypot(grad_T_x, grad_T_y);
            
            grad_F[x + y*nx] = grad_Phi_norm;
            grad_U[x + y*nx] = grad_T_norm;
        });
    }

    cache_free(&tag);
}

void semi_implicit_solver_step(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    semi_implicit_solver_step_based(solver, state.F, state.U, state.U, next_state, params, iter, do_debug);
}

void semi_implicit_solver_step_corrector(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Allen_Cahn_Params params, size_t iter, Allen_Cahn_Stats* stats_or_null)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    Explicit_State temp_state = {0};
    temp_state.F = cache_alloc(Real, N, &tag);
    temp_state.U = cache_alloc(Real, N, &tag);
    temp_state.nx = params.nx;
    temp_state.ny = params.ny;

    static int last_placement = 0;

    //Init states in such a way that the resutl will already be in 
    // next_state (thus no need to copy)
    Explicit_State steps[2] = {0};
    if(last_placement % 2 == 0)
    {
        steps[0] = next_state;
        steps[1] = temp_state;
    }
    else
    {
        steps[1] = next_state;
        steps[0] = temp_state;
    }

    Real* step_resiudal = cache_alloc(Real, N, &tag);
    
    Real step_residual_avg_error = 0;
    Real step_residual_max_error = 0;
    bool converged = false;
    USE_VARIABLE(converged);
    
    int k = 0;
    int max_iters = params.corrector_max_iters;
    if(params.do_corrector_loop == false)
        max_iters = 0;

    if(max_iters == 0 && params.do_stats_step_residual)
        max_iters = 1;

    bool do_debug = false;
    if(max_iters == 0)
        do_debug = params.do_debug;

    //Perform first step
    semi_implicit_solver_step(solver, state, steps[0], params, iter, do_debug);
    for(; k < max_iters; k++)
    {
        Explicit_State step_curr = steps[MOD(k, 2)];
        Explicit_State step_next = steps[MOD(k + 1, 2)];

        log_group();
        semi_implicit_solver_step_based(solver, state.F, step_curr.U, state.U, step_next, params, iter, false);

        cuda_for(0, N, [=]SHARED(int i){
            //@NOTE:fabs is broken and linking the wrong function which results in
            // illegal memory access ?!
            //@NOTE: abs mostly for debug view
            Real diff = step_curr.F[i] - step_next.F[i]; 
            step_resiudal[i] = diff >= 0 ? diff : -diff;
        });

        //@NOTE: no explicit sync!
        if(k < (int) STATIC_ARRAY_SIZE(solver->debug_maps.step_residuals))
            CUDA_DEBUG_TEST(hipMemcpyAsync(solver->debug_maps.step_residuals[k], step_resiudal, (size_t)N*sizeof(Real), hipMemcpyDeviceToDevice));

        step_residual_avg_error = vector_euclid_norm(step_resiudal, N);
        step_residual_max_error = vector_max(step_resiudal, N);
        if(k < (int) STATIC_ARRAY_SIZE(stats_or_null->L2_step_residuals) && stats_or_null)
        {
            stats_or_null->L2_step_residuals[k] = step_residual_avg_error;
            stats_or_null->Lmax_step_residuals[k] = step_residual_max_error;
            stats_or_null->step_residuals = k + 1;
        }

        LOG_DEBUG("SOLVER", "step residual loop: %i | avg: %e | max: %e | tolerance: %lf", k, 
            (double) step_residual_avg_error, (double) step_residual_max_error, params.corrector_tolerance);
        if(step_residual_avg_error < params.corrector_tolerance)
        {
            k ++;
            converged = true;
            break;
        }

        log_ungroup();
    }
    
    last_placement = k;

    //Debug only print
    step_residual_max_error = vector_max(step_resiudal, N);
    // if(max_iters > 0)
        // LOG_DEBUG("SOLVER", "step residual %s iters: %i | avg: %lf | max: %lf | tolerance: %lf", 
            // converged ? "converged" : "diverged", k + 1, 
            // (double) step_residual_avg_error, (double) step_residual_max_error, params.corrector_tolerance);

    //If the ended on step is already next_state dont copy anything
    Explicit_State final_step = steps[MOD(k, 2)];
    if(final_step.F != next_state.F)
    {
        CUDA_DEBUG_TEST(hipMemcpyAsync(next_state.F, final_step.F, (size_t)N*sizeof(Real), hipMemcpyDeviceToDevice));
        CUDA_DEBUG_TEST(hipMemcpyAsync(next_state.U, final_step.U, (size_t)N*sizeof(Real), hipMemcpyDeviceToDevice));
    }

    cache_free(&tag);
}

void semi_implicit_solver_get_maps(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t)map_count);
    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    // ASSIGN_MAP_NAMED(solver->maps.b_F, "b_F");           
    // ASSIGN_MAP_NAMED(solver->debug_maps.AfF, "AfF");           
    // ASSIGN_MAP_NAMED(solver->maps.b_U, "b_U");           
    // ASSIGN_MAP_NAMED(solver->debug_maps.AuU, "AuU");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");           
    ASSIGN_MAP_NAMED(solver->maps.anisotrophy, "Anisotrophy");

    CHECK_BOUNDS(2, STATIC_ARRAY_SIZE(solver->debug_maps.step_residuals)); 
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[0], "step_residual1");          
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[1], "step_residual2");           
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[2], "step_residual3");           
}

struct Semi_Implicit_Coupled_Cross_Matrix {
    Anisotrophy_Matrix A_F; //A anisotrophy scaled cross matrix
    Real* B_U; //A changing diagonal 

    Cross_Matrix_Static A_U; //Static cross matrix
    Real B_F; //A single value diagonal

    int nx;
    int ny;
};

void semi_implicit_coupled_solver_resize(Semi_Implicit_Coupled_Solver* solver, int nx, int ny)
{
    if(solver->nx != nx || solver->ny != ny)
    {
        int N = ny*nx;
        int N_old = solver->ny*solver->nx;
        cuda_realloc_in_place((void**) &solver->b_C, 2*(size_t)N*sizeof(Real), 2*(size_t)N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->aniso, (size_t)N*sizeof(Real), (size_t)N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->B_U, (size_t)N*sizeof(Real), (size_t)N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void semi_implicit_coupled_state_resize(Semi_Implicit_Coupled_State* state, int nx, int ny)
{
    if(state->nx != nx || state->ny != ny)
    {
        int N = ny*nx;
        int N_old = state->ny*state->nx;
        cuda_realloc_in_place((void**) &state->C, 2*(size_t)N*isizeof(Real), 2*(size_t)N_old*isizeof(Real), REALLOC_ZERO);
        state->nx = nx;
        state->ny = ny;
    }
}

void semi_implicit_coupled_matrix_multiply(Real* out, const void* A_, const Real* x, int vec_size)
{
    Semi_Implicit_Coupled_Cross_Matrix A = *(Semi_Implicit_Coupled_Cross_Matrix*)A_;

    int nx = A.nx;
    int ny = A.ny;
    int N = nx*ny;

    Real* F = (Real*) x;
    Real* U = (Real*) x + N; 

    Real* out_F = out;
    Real* out_U = out + N;

    //F equation
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;
        Real s = A.A_F.anisotrophy[i];
        Real X = A.A_F.X*s;
        Real Y = A.A_F.Y*s;
        Real C = 1 + A.A_F.C_minus_one*s;

        Real F_val = F[i]*C;
        F_val += *at_mod(F, x + 1, y, nx, ny)*X;
        F_val += *at_mod(F, x - 1, y, nx, ny)*X;
        F_val += *at_mod(F, x , y + 1, nx, ny)*Y;
        F_val += *at_mod(F, x , y - 1, nx, ny)*Y;

        Real U_val = A.B_U[i]*U[i]; 

        out_F[i] = F_val + U_val;
    });

    //U equation
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;

        Real U_val = U[i]*A.A_U.C;
        U_val += *at_mod(U, x + 1, y, nx, ny)*A.A_U.R;
        U_val += *at_mod(U, x - 1, y, nx, ny)*A.A_U.L;
        U_val += *at_mod(U, x , y + 1, nx, ny)*A.A_U.U;
        U_val += *at_mod(U, x , y - 1, nx, ny)*A.A_U.D;

        Real F_val = F[i]*A.B_F;
        out_U[i] = F_val + U_val;
    });
}

void semi_implicit_coupled_solver_step(Semi_Implicit_Coupled_Solver* solver, Semi_Implicit_Coupled_State state, Semi_Implicit_Coupled_State next_state, Allen_Cahn_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->nx;
    Real dy = (Real) params.L0 / solver->ny;

    int nx = solver->nx;
    int ny = solver->ny;
    int N = nx*ny;

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; 
    Real dt = (Real) params.dt;
    Real S = (Real) params.S; 
    Real m0 = (Real) params.m0; 
    Real theta0 = (Real) params.theta0;

    Real* F = state.C;
    Real* U = state.C + N;
    
    Real* b_F = solver->b_C;
    Real* b_U = solver->b_C + N;

    Real* aniso = solver->aniso;
    Real* B_U = solver->B_U;

    //Prepare dynamic data
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        Real T = U[x + y*nx];
        Real Phi = F[x + y*nx];

        Real Phi_U = *at_mod(F, x, y + 1, nx, ny);
        Real Phi_D = *at_mod(F, x, y - 1, nx, ny);
        Real Phi_R = *at_mod(F, x + 1, y, nx, ny);
        Real Phi_L = *at_mod(F, x - 1, y, nx, ny);

        Real grad_Phi_x = (Phi_R - Phi_L)/(2*dx);
        Real grad_Phi_y = (Phi_U - Phi_D)/(2*dy);
        Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);
 
        Real g_theta = 1;
        {
            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            g_theta = 1.0f - S*custom_cos(m0*theta + theta0);
        }

        // g_theta = 1;
        Real f_tilda = b*xi*xi*beta*grad_Phi_norm;
        Real f = g_theta*a*f0(Phi) - b*xi*xi*beta*(T - Tm)*grad_Phi_norm;
        Real T_factor = dt*f_tilda/(xi*xi*alpha);
        B_U[x+y*nx] = -T_factor;
        aniso[x+y*nx] = g_theta;
        b_F[x + y*nx] = Phi + dt*g_theta/(xi*xi*alpha)*f0(Phi) - T_factor*Tm;
        b_U[x + y*nx] = T + L*Phi;
    });

    Anisotrophy_Matrix A_F = {0};
    A_F.anisotrophy = aniso;
    A_F.C_minus_one = 2*dt/(alpha*dx*dx) + 2*dt/(alpha*dy*dy);
    A_F.X = -dt/(alpha*dx*dx);
    A_F.Y = -dt/(alpha*dy*dy);

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);

    #define ECHOF(x)    printf(#x ": %e\n", (double) (x))
    Real B_U_norm = vector_dot_product(B_U, B_U, N);
    B_U_norm = sqrt(B_U_norm / N);
    ECHOF(B_U_norm);
    ECHOF(A_F.C_minus_one + 1);
    ECHOF(A_F.X);
    ECHOF(A_U.C);
    ECHOF(A_U.U);

    Semi_Implicit_Coupled_Cross_Matrix A_C = {0};
    A_C.A_F = A_F;
    A_C.B_U = B_U;
    A_C.A_U = A_U;
    A_C.B_F = -L;
    A_C.nx = nx;
    A_C.ny = ny;

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-10;
    solver_params.tolerance = (Real) 1.0e-7;
    solver_params.max_iters = 200;
    solver_params.initial_value_or_null = state.C;

    Conjugate_Gardient_Convergence conv = conjugate_gradient_solve(&A_C, next_state.C, solver->b_C, 2*N, semi_implicit_coupled_matrix_multiply, &solver_params);
    printf("%lli C %s in %i iters with error %lf\n", (long long) iter, conv.converged ? "converged" : "diverged", conv.iters, (double) conv.error);
}

void semi_implicit_coupled_solver_get_maps(Semi_Implicit_Coupled_Solver* solver, Semi_Implicit_Coupled_State state, Sim_Map* maps, int map_count)
{
    int N = solver->nx*solver->ny;
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t)map_count);
    ASSIGN_MAP_NAMED(state.C, "Phi");            
    ASSIGN_MAP_NAMED(state.C + N, "T");            
    ASSIGN_MAP_NAMED(solver->b_C, "b_F");           
    ASSIGN_MAP_NAMED(solver->b_C + N, "b_U");           
    ASSIGN_MAP_NAMED(solver->B_U, "B_U");           
    ASSIGN_MAP_NAMED(solver->aniso, "Anisotrophy");  
}

extern "C" void kernel_float_from_double(float* output, const double* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (float) input[i];
    });
}
extern "C" void kernel_double_from_float(double* output, const float* input, size_t size)
{
    cuda_for(0, (int) size, [=]SHARED(int i){
        output[i] = (double) input[i];
    });
}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify)
{
    if(modify == MODIFY_UPLOAD)
        CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice));
    else
        CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost));
}

extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t size, Sim_Modify modify)
{
    static float* static_device = NULL;
    static size_t static_size = 0;

    if(sizeof(Real) != sizeof(float))
    {
        if(static_size < size)
        {
            cuda_realloc_in_place((void**) &static_device, size*sizeof(float), static_size*sizeof(float), 0);
            static_size = size;
        }

        if(modify == MODIFY_UPLOAD)
        {
            //Upload: host -> static -> device
            CUDA_DEBUG_TEST(hipMemcpy(static_device, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
            kernel_double_from_float((double*) (void*) device_memory, static_device, size);
        }
        else
        {
            //download: device -> static -> host
            kernel_float_from_double(static_device, (double*) (void*) device_memory, size);
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, static_device, size*sizeof(float), hipMemcpyDeviceToHost));
        }
    }
    else
    {
        if(modify == MODIFY_UPLOAD)
            CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size*sizeof(float), hipMemcpyHostToDevice));
        else
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size*sizeof(float), hipMemcpyDeviceToHost));
    }
}


extern "C" void sim_solver_reinit(Sim_Solver* solver, Solver_Type type, int nx, int ny)
{
    if(solver->type != type && solver->type != SOLVER_TYPE_NONE)
        sim_solver_reinit(solver, solver->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            ny = 0;
            nx = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_solver_resize(&solver->expli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_resize(&solver->impli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_solver_resize(&solver->impli_coupled, nx, ny);
        } break;

        default: {
            assert(false);
        }
    };

    solver->type = type;
    solver->nx = nx;
    solver->ny = ny;
}

void sim_state_reinit(Sim_State* states, Solver_Type type, int nx, int ny)
{
    if(states->type != type && states->type != SOLVER_TYPE_NONE)
        sim_state_reinit(states, states->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            ny = 0;
            nx = 0;
        } break;

        case SOLVER_TYPE_EXPLICIT: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_state_resize(&states->expli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            //For the moemnt these are the same
            explicit_state_resize(&states->impli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_state_resize(&states->impli_coupled, nx, ny);
        } break;

        default: {
            assert(false);
        }
    };

    states->type = type;
    states->nx = nx;
    states->ny = ny;
}

extern "C" void sim_states_reinit(Sim_State* states, int state_count, Solver_Type type, int nx, int ny)
{
    for(int i = 0; i < state_count; i++)
        sim_state_reinit(&states[i], type, nx, ny);
}

extern "C" double sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Allen_Cahn_Params params, Allen_Cahn_Stats* stats_or_null)
{
    int required_history = solver_type_required_history(solver->type);
    const char* solver_name = solver_type_to_cstring(solver->type);
    
    bool okay = true;
    if(states_count < required_history)
    {
        okay = false;
        LOG_INFO("SOLVER", "Step: Not enough history for solver %s! Required %i. Got %i", solver_name, states_count, required_history);
    }
    else
    {
        for(int i = 0; i < states_count; i++)
        {
            if(states[i].type != solver->type)
            {
                LOG_INFO("SOLVER", "Step: state[%i] is of bad type %s. Expected %s", solver_type_to_cstring(states[i].type), solver_name);
                okay = false;
            }
        }
    }
    
    double step_by = 0;
    if(okay)
    {
        step_by = params.dt;
        ASSERT(states_count > 0);
        Sim_State state = states[MOD(iter, states_count)];
        Sim_State next_state = states[MOD(iter + 1, states_count)];
        switch(solver->type) {
            case SOLVER_TYPE_NONE: {
                LOG_INFO("SOLVER", "Step: stepping as solver type none has no effect");
                // nothing
            } break;

            case SOLVER_TYPE_EXPLICIT: 
            case SOLVER_TYPE_EXPLICIT_RK4: 
            case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
                step_by = explicit_solver_choose_and_copute_step_residual(solver->type, &solver->expli, state.expli, &next_state.expli, params, (size_t) iter, stats_or_null);
            } break;

            case SOLVER_TYPE_SEMI_IMPLICIT: {
                semi_implicit_solver_step_corrector(&solver->impli, state.impli, next_state.impli, params, (size_t) iter, stats_or_null);
            } break;

            case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
                semi_implicit_coupled_solver_step(&solver->impli_coupled, state.impli_coupled, next_state.impli_coupled, params, (size_t) iter, params.do_debug);
            } break;

            default: assert(false);
        };
    }

    CUDA_DEBUG_TEST(hipDeviceSynchronize());
    return step_by;
}

extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count)
{
    if(states_count <= 0 || map_count <= 0)
        return;

    Sim_State state = states[MOD(iter, states_count)];
    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            //none
        } break;

        case SOLVER_TYPE_EXPLICIT: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_solver_get_maps(&solver->expli, state.expli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_get_maps(&solver->impli, state.impli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_solver_get_maps(&solver->impli_coupled, state.impli_coupled, maps, map_count);
        } break;

        default: assert(false);
    };
}


#else

extern "C" void sim_solver_reinit(Sim_Solver* solver, Solver_Type type, int nx, int ny) {}
extern "C" void sim_states_reinit(Sim_State* states, int state_count, Solver_Type type, int nx, int ny) {}
extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count) {}
extern "C" double sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Allen_Cahn_Params params, Allen_Cahn_Stats* stats_or_null) {return 0;}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify) {}
extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t size, Sim_Modify modify) {}
extern "C" void sim_modify_double(Real* device_memory, double* host_memory, size_t size, Sim_Modify modify) {}
#endif

#include "cuda_reduction.cuh"
#include "cuda_random.cuh"

#ifdef COMPILE_BENCHMARKS
static void cache_prepare(int count, int item_size, int N)
{
    Cache_Tag tag = cache_tag_make();
    for(int i = 0; i < count; i++)
        _cache_alloc((size_t) (item_size*N), &tag, SOURCE_INFO());
    cache_free(&tag);
}

extern "C" bool run_benchmarks(int N_)
{
    csize N = (csize) N_;
    cache_prepare(3, sizeof(int), N);
    cache_prepare(3, sizeof(float), N);
    cache_prepare(3, sizeof(double), N);

    Cache_Tag tag = cache_tag_make();
    uint* rand_state = cache_alloc(uint, N, &tag);
    random_map_seed_32(rand_state, N, (uint32_t) clock_ns());

    int GB = 1024*1024*1024;
    {
        double* rand_map = cache_alloc(double, N, &tag);
        random_map_32(rand_map, rand_state, N);
        
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_map, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_map, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_map, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(double);
        LOG_OKAY("BENCH", "double (gb/s): cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(double)).str);
        LOG_OKAY("BENCH", "double (time): cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }
    {
        float* rand_map = cache_alloc(float, N, &tag);
        random_map_32(rand_map, rand_state, N);
        
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_map, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_map, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_map, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(float);
        LOG_OKAY("BENCH", "float (gb/s) : cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(float)).str);
        LOG_OKAY("BENCH", "float (time) : cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }

    {
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_state, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_state, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_state, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(uint);
        LOG_OKAY("BENCH", "uint (gb/s)  : cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(uint)).str);
        LOG_OKAY("BENCH", "uint (time)  : cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }

    cache_free(&tag);
    return true;
}
#else
extern "C" bool run_benchmarks(int N)
{
    (void) N;
    return false;
}
#endif

#include "cuda_examples.cuh"
extern "C" bool run_tests()
{
    test_all_examples(3);
    #ifdef TEST_CUDA_FOR_IMPL
    test_tiled_for((uint64_t) clock_ns());
    test_tiled_for_2D((uint64_t) clock_ns());
    #endif
    #ifdef TEST_CUDA_REDUCTION_IMPL
    test_reduce((uint64_t) clock_ns());
    #endif

    return true;
}