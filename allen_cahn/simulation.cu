#include "hip/hip_runtime.h"
#include "simulation.h"

#ifdef COMPILE_TESTS
#define TEST_CUDA_ALL
#endif

#include "exact.h"
#include "cuda_util.cuh"
#include "cuda_reduction.cuh"
#include "cuda_for.cuh"
#include "cuda_random.cuh"
#include <assert.h>

#ifdef COMPILE_SIMULATION

#ifndef USE_CUSTOM_REDUCE
#include <thrust/inner_product.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#endif

#if 1
SHARED Real custom_hypot(Real y, Real x) { return (Real) hypotf((float)y, (float)x); }
SHARED Real custom_atan2(Real y, Real x) { return (Real) atan2f((float) y, (float) x); }
SHARED Real custom_cos(Real theta) { return (Real) cosf((float)theta); }
#else
SHARED Real custom_hypot(Real y, Real x) { return hypot(y, x); }
SHARED Real custom_atan2(Real y, Real x) { return atan2(y, x); }
SHARED Real custom_cos(Real theta) { return cos(theta); }
#endif

Real vector_dot_product(const Real *a, const Real *b, int ny)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_dot_product(a, b, ny);
    #else
    // wrap raw pointers to device memory with device_ptr
    thrust::device_ptr<const Real> d_a(a);
    thrust::device_ptr<const Real> d_b(b);

    // inner_product implements a mathematical dot product
    return thrust::inner_product(d_a, d_a + ny, d_b, 0.0);
    #endif
}

Real vector_max(const Real *a, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_max(a, N);
    #else
    thrust::device_ptr<const Real> d_a(a);
    return *(thrust::max_element(d_a, d_a + N));
    #endif
}

Real vector_get_l2_dist(const Real* a, const Real* b, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_distance(a, b, N)/ sqrt((Real) N);
    #else
    Cache_Tag tag = cache_tag_make();
    Real* temp = cache_alloc(Real, N, &tag);
    cuda_for(0, N, [=]SHARED(int i){
        temp[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_dot_product(temp, temp, N);
    Real error = sqrt(temp_dot_temp/N);
    cache_free(&tag);
    return error;
    #endif
}

Real vector_get_max_dist(const Real* a, const Real* b, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_distance(a, b, N);
    #else
    Cache_Tag tag = cache_tag_make();
    Real* temp = cache_alloc(Real, N, &tag);
    cuda_for(0, N, [=]SHARED(int i){
        temp[i] = a[i] - b[i];
    });

    Real temp_dot_temp = vector_max(temp, N);
    Real error = sqrt(temp_dot_temp/N);
    cache_free(&tag);
    return error;
    #endif
}


Real vector_euclid_norm(const Real* vector, int N)
{
    #ifdef USE_CUSTOM_REDUCE
    return cuda_L2_norm(vector, N)/ sqrt((Real) N);
    #else
    Real dot = vector_dot_product(vector, vector, N);
    return sqrt(dot / N);
    #endif
}

SHARED Real* at_mod(Real* map, int x, int y, int nx, int ny)
{
    int x_mod = x;
    if(x_mod < 0)
        x_mod += nx;
    else if(x_mod >= nx)
        x_mod -= nx;

    int y_mod = y;
    if(y_mod < 0)
        y_mod += ny;
    else if(y_mod >= ny)
        y_mod -= ny;

    return &map[x_mod + y_mod*nx];
}

SHARED Real boundary_sample(const Real* map, int x, int y, int nx, int ny, Sim_Boundary_Type bound)
{
    switch(bound)
    {
        case BOUNDARY_PERIODIC: {
            int x_mod = x;
            if(x_mod < 0)
                x_mod += nx;
            else if(x_mod >= nx)
                x_mod -= nx;

            int y_mod = y;
            if(y_mod < 0)
                y_mod += ny;
            else if(y_mod >= ny)
                y_mod -= ny;

            return map[x_mod + y_mod*nx];
        } break;

        case BOUNDARY_DIRICHLET_ZERO: {
            if(0 <= x && x < nx 
                && 0 <= y && y < ny)
                return map[x + y*nx];
            else
            {
                int clampx = CLAMP(x, 0, nx - 1);
                int clampy = CLAMP(y, 0, ny - 1);
                return -map[clampx + clampy*nx];
            }              
        } break;

        case BOUNDARY_NEUMANN_ZERO: {
            int clampx = CLAMP(x, 0, nx - 1);
            int clampy = CLAMP(y, 0, ny - 1);
            return map[clampx + clampy*nx];
        } break;
    }

    return 0;
}

void explicit_solver_resize(Explicit_Solver* solver, int nx, int ny)
{
    size_t N = (size_t)nx*(size_t)ny;
    size_t N_old = (size_t)solver->nx*(size_t)solver->ny;
    if(solver->nx != nx || solver->ny != ny)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        Real* debug_maps = (Real*) (void*) &solver->debug_maps;
        for(uint i = 0; i < sizeof(solver->debug_maps) / sizeof(Real*); i++)
            cuda_realloc_in_place((void**) &debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void explicit_state_resize(Explicit_State* state, int nx, int ny)
{
    size_t N = (size_t)nx*(size_t)ny;
    size_t N_old = (size_t)state->nx*(size_t)state->ny;
    if(state->nx != nx || state->ny != ny)
    {
        cuda_realloc_in_place((void**) &state->F, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &state->U, N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        state->nx = nx;
        state->ny = ny;
    }
}

struct Bundled {
    Real Phi;
    Real T;
};

struct Explicit_Solve_Result {
    Real dt_Phi;
    Real dt_T;
};

struct Explicit_Solve_Debug {
    Real grad_Phi;
    Real grad_T;
    Real g_theta;
    Real theta;
    Real reaction_term;
};

SHARED Real f0(Real phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}

extern "C" void explicit_solver_newton_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    int nx = params.nx;
    int ny = params.ny;

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; //Latent heat, not L0 (sym size) ! 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; //anisotrophy strength
    Real m0 = (Real) params.m0; //anisotrophy frequency (?)
    Real theta0 = (Real) params.theta0;
    bool do_corrector_guess = params.do_corrector_guess;

    Real dx = (Real) params.L0 / nx;
    Real dy = (Real) params.L0 / ny;
    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);

    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    Real* in_F = state.F;
    Real* in_U = state.U;

    Real* out_F = next_state->F;
    Real* out_U = next_state->U;

    Real fu = 0;
    if(params.do_exact)
    {
        Exact_Params exact_params = get_static_exact_params(params);
        fu = exact_fu(step_info.sim_time, exact_params);
    }

    cuda_tiled_for_2D<1, 1, Bundled>(0, 0, params.nx, params.ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
            csize x_mod = x;
            csize y_mod = y;

            if(x_mod < 0)
                x_mod += nx;
            else if(x_mod >= nx)
                x_mod -= nx;

            if(y_mod < 0)
                y_mod += ny;
            else if(y_mod >= ny)
                y_mod -= ny;

            int I = x_mod + y_mod*nx;

            Real Phi = in_F[I];
            Real T = in_U[I];
            return Bundled{Phi, T};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
            Bundled C = shared[tx   + ty*tile_size_x];
            Bundled E = shared[tx+1 + ty*tile_size_x];
            Bundled W = shared[tx-1 + ty*tile_size_x];
            Bundled N = shared[tx   + (ty+1)*tile_size_x];
            Bundled S = shared[tx   + (ty-1)*tile_size_x];

            Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
            Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dx;
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

            Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
            Real laplace_T = (W.T - 2*C.T + E.T)*one_over_dx2 +     (S.T - 2*C.T + N.T)*one_over_dy2;

            Real k0 = g_theta*f0(C.Phi)*k0_factor;
            Real k2 = grad_Phi_norm*k2_factor;
            Real k1 = g_theta*k1_factor;
            Real corr = 1 + k2*dt_L;

            Real dt_Phi = 0;
            if(do_corrector_guess)
                dt_Phi = (k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T))/corr;
            else
                dt_Phi = k1*laplace_Phi + k0 - k2*(C.T - Tm);

            Real dt_T = laplace_T + L*dt_Phi + fu; 

            out_F[x + y*nx] = C.Phi + dt_Phi*dt;
            out_U[x + y*nx] = C.T + dt_T*dt;
        });

    if(params.do_stats && stats_or_null)
    {
        float_array_push(&stats_or_null->vectors.phi_iters, 1);
        float_array_push(&stats_or_null->vectors.T_iters, 1);
    }
}

struct Explicit_Blend_State {
    Real weight;
    Explicit_State state;
};

template<typename ... States>
void explicit_solver_solve_lin_combination(Explicit_State* out, Sim_Params params, Sim_Step_Info step_info, States... state_args)
{
    int nx = params.nx;
    int ny = params.ny;
    Real* out_F = out->F;
    Real* out_U = out->U;

    constexpr int state_count = (int) sizeof...(state_args);
    Explicit_Blend_State states[(size_t) state_count] = {state_args...};

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; //Latent heat, not L0 (sym size) ! 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; //anisotrophy strength
    Real m0 = (Real) params.m0; //anisotrophy frequency (?)
    Real theta0 = (Real) params.theta0;
    bool do_corrector_guess = params.do_corrector_guess;

    Real dx = (Real) params.L0 / nx;
    Real dy = (Real) params.L0 / ny;
    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);

    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    Real fu = 0;
    if(params.do_exact)
    {
        Exact_Params exact_params = get_static_exact_params(params);
        fu = exact_fu(step_info.sim_time, exact_params);
    }

    Sim_Boundary_Type U_bound = params.T_boundary;
    Sim_Boundary_Type F_bound = params.Phi_boundary;
    cuda_tiled_for_2D<1, 1, Bundled>(0, 0, nx, ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled{
            Real U = 0;
            Real F = 0;
            #pragma unroll
            for(int i = 0; i < state_count; i++)
            {
                F += boundary_sample(states[i].state.F, x, y, nx, ny, F_bound) * states[i].weight;
                U += boundary_sample(states[i].state.U, x, y, nx, ny, U_bound) * states[i].weight;
            }

            return Bundled{F, U};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
            Bundled C = shared[tx   + ty*tile_size_x];
            Bundled E = shared[tx+1 + ty*tile_size_x];
            Bundled W = shared[tx-1 + ty*tile_size_x];
            Bundled N = shared[tx   + (ty+1)*tile_size_x];
            Bundled S = shared[tx   + (ty-1)*tile_size_x];

            Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
            Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dx;
            Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

            Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
            Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

            Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
            Real laplace_T = (W.T - 2*C.T + E.T)*one_over_dx2 +     (S.T - 2*C.T + N.T)*one_over_dy2;

            Real k0 = g_theta*f0(C.Phi)*k0_factor;
            Real k2 = grad_Phi_norm*k2_factor;
            Real k1 = g_theta*k1_factor;
            Real corr = 1 + k2*dt_L;

            Real dt_Phi = 0;
            if(do_corrector_guess)
                dt_Phi = (k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T))/corr;
            else
                dt_Phi = k1*laplace_Phi + k0 - k2*(C.T - Tm);

            Real dt_T = laplace_T + L*dt_Phi + fu;

            out_F[x + y*nx] = dt_Phi;
            out_U[x + y*nx] = dt_T;
        });   
}

void calc_debug_values(const Real* F, const Real* U, Real* grad_F, Real* grad_U, Real* aniso, Sim_Params params)
{
    int nx = params.nx;
    int ny = params.ny;
    Real S0 = (Real) params.S;
    Real m0 = (Real) params.m0;
    Real theta0 = (Real) params.theta0;
    Sim_Boundary_Type Phi_bound = params.Phi_boundary;
    Sim_Boundary_Type T_bound = params.T_boundary;
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        Real Phi_U = boundary_sample(F, x, y + 1, nx, ny, Phi_bound);
        Real Phi_D = boundary_sample(F, x, y - 1, nx, ny, Phi_bound);
        Real Phi_R = boundary_sample(F, x + 1, y, nx, ny, Phi_bound);
        Real Phi_L = boundary_sample(F, x - 1, y, nx, ny, Phi_bound);

        Real T_U = boundary_sample(U, x, y + 1, nx, ny, T_bound);
        Real T_D = boundary_sample(U, x, y - 1, nx, ny, T_bound);
        Real T_R = boundary_sample(U, x + 1, y, nx, ny, T_bound);
        Real T_L = boundary_sample(U, x - 1, y, nx, ny, T_bound);

        Real grad_Phi_x = (Phi_R - Phi_L);
        Real grad_Phi_y = (Phi_U - Phi_D);
        Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

        Real grad_T_x = (T_R - T_L);
        Real grad_T_y = (T_U - T_D);
        Real grad_T_norm = custom_hypot(grad_T_x, grad_T_y);
        
        Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
        Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

        grad_F[x + y*nx] = grad_Phi_norm;
        grad_U[x + y*nx] = grad_T_norm; 
        aniso[x + y*nx] = g_theta;
    });
}
void explicit_solver_rk4_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    Explicit_State steps[4] = {0};
    for(int i = 0; i < (int) ARRAY_LEN(steps); i++)
    {
        steps[i].F = cache_alloc(Real, N, &tag);
        steps[i].U = cache_alloc(Real, N, &tag);
        steps[i].nx = params.nx;
        steps[i].ny = params.ny;
    }

    Explicit_State k1 = steps[0];
    Explicit_State k2 = steps[1];
    Explicit_State k3 = steps[2];
    Explicit_State k4 = steps[3];

    Real dt = (Real) params.dt;
    using W = Explicit_Blend_State;
    explicit_solver_solve_lin_combination(&k1, params, step_info, W{1, state});
    explicit_solver_solve_lin_combination(&k2, params, step_info, W{1, state}, W{dt * (Real) 0.5, k1});
    explicit_solver_solve_lin_combination(&k3, params, step_info, W{1, state}, W{dt * (Real) 0.5, k2});
    explicit_solver_solve_lin_combination(&k4, params, step_info, W{1, state}, W{dt * 1, k3});

    Real* out_F = next_state->F;
    Real* out_U = next_state->U;
    cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
        out_F[i] = state.F[i] + dt/6*(k1.F[i] + 2*k2.F[i] + 2*k3.F[i] + k4.F[i]);
        out_U[i] = state.U[i] + dt/6*(k1.U[i] + 2*k2.U[i] + 2*k3.U[i] + k4.U[i]);
    });

    if(params.do_debug)
        calc_debug_values(next_state->F, next_state->U, solver->debug_maps.grad_phi, solver->debug_maps.grad_T, solver->debug_maps.aniso_factor, params);

    if(params.do_stats && stats_or_null)
    {
        float_array_push(&stats_or_null->vectors.phi_iters, 1);
        float_array_push(&stats_or_null->vectors.T_iters, 1);
    }

    cache_free(&tag);
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
}

double explicit_solver_rk4_adaptive_step(Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    static Real _initial_step = 0;
    if(step_info.iter == 0)
        _initial_step = (Real) params.dt;

    Real tau = _initial_step;
    Explicit_State steps[5] = {0};
    for(int i = 0; i < (int) ARRAY_LEN(steps); i++)
    {
        steps[i].F = cache_alloc(Real, N, &tag);
        steps[i].U = cache_alloc(Real, N, &tag);
        steps[i].nx = params.nx;
        steps[i].ny = params.ny;
    }

    Real* Epsilon_F = cache_alloc(Real, N, &tag);
    Real* Epsilon_U = cache_alloc(Real, N, &tag);
    Real epsilon_F = 0;
    Real epsilon_U = 0;

    Explicit_State k1 = steps[0];
    Explicit_State k2 = steps[1];
    Explicit_State k3 = steps[2];
    Explicit_State k4 = steps[3];
    Explicit_State k5 = steps[4];

    using W = Explicit_Blend_State;
    explicit_solver_solve_lin_combination(&k1, params, step_info, W{1, state});

    bool converged = false;
    int i = 0;
    int max_iters = MAX(MAX(params.T_max_iters, params.Phi_max_iters), 1);
    Real used_tau = tau;
    for(; i < max_iters && converged == false; i++)
    {
        // k1 = f(t, x);
        // k2 = f(t + tau/3, x + tau/3*k1);
        // k3 = f(t + tau/3, x + tau/6*(k1 + k2));
        // k4 = f(t + tau/2, x + tau/8*(k1 + 3*k3));
        // k5 = f(t + tau/1, x + tau*(0.5f*k1 - 1.5f*k3 + 2*k4));
        
        // k1 = f(x);
        // k2 = f(x + tau/3*k1);
        // k3 = f(x + tau/6*k1 + tau/6*k2);
        // k4 = f(x + tau/8*k1 + tau*3/8*k3));
        // k5 = f(x + tau/2*k1 - tau*3/2*k3 + tau*2*k4));
        
        explicit_solver_solve_lin_combination(&k2, params, step_info, W{1, state}, W{tau/3, k1});
        explicit_solver_solve_lin_combination(&k3, params, step_info, W{1, state}, W{tau/6, k1}, W{tau/6, k2});
        explicit_solver_solve_lin_combination(&k4, params, step_info, W{1, state}, W{tau/8, k1}, W{tau*3/8, k3});
        explicit_solver_solve_lin_combination(&k5, params, step_info, W{1, state}, W{tau/2, k1}, W{-tau*3/2, k3}, W{tau*2, k4});

        cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
            Real F = (Real)0.2*k1.F[i] - (Real)0.9*k3.F[i] + (Real)0.8*k4.F[i] - (Real)0.1*k5.F[i];
            Real U = (Real)0.2*k1.U[i] - (Real)0.9*k3.U[i] + (Real)0.8*k4.U[i] - (Real)0.1*k5.U[i];

            Epsilon_F[i] = F >= 0 ? F : -F;
            Epsilon_U[i] = U >= 0 ? U : -U;
        });

        epsilon_F = vector_max(Epsilon_F, N);
        epsilon_U = vector_max(Epsilon_U, N);

        if(epsilon_F < params.Phi_tolerance && epsilon_U < params.T_tolerance)
            converged = true;

        Real epsilon = (Real) MAX(epsilon_F + epsilon_U, 1e-8);
        Real delta = (Real) MAX(params.Phi_tolerance + params.T_tolerance, 1e-8);
        used_tau = tau;
        tau = pow(delta / epsilon, (Real)0.2)*4/5*tau;
    }

    Real* next_F = next_state->F;
    Real* next_U = next_state->U;
    cuda_for(0, params.ny*params.nx, [=]SHARED(int i){
        next_F[i] = state.F[i] + used_tau*((Real)1.0/6*(k1.F[i] + k5.F[i]) + (Real)2.0/3*k4.F[i]);
        next_U[i] = state.U[i] + used_tau*((Real)1.0/6*(k1.U[i] + k5.U[i]) + (Real)2.0/3*k4.U[i]);
    });

    LOG("SOLVER", converged ? LOG_DEBUG : LOG_WARN, "rk4-adaptive %s in %i iters with error F:%lf | U:%lf | tau:%e", converged ? "converged" : "diverged", i, (double) epsilon_F, (double) epsilon_U, (double)used_tau);
    _initial_step = tau;

    if(params.do_debug)
        calc_debug_values(next_state->F, next_state->U, solver->debug_maps.grad_phi, solver->debug_maps.grad_T, solver->debug_maps.aniso_factor, params);

    if(params.do_stats && stats_or_null)
    {
        float_array_push(&stats_or_null->vectors.phi_iters, i);
        float_array_push(&stats_or_null->vectors.T_iters, i);
    }

    cache_free(&tag);
    CUDA_DEBUG_TEST(hipDeviceSynchronize());
    return (double) used_tau;
}

double explicit_solver_choose(Sim_Solver_Type type, Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    if(type == SOLVER_TYPE_EXPLICIT_EULER)
    {
        explicit_solver_newton_step(solver, state, next_state, params, step_info, stats_or_null);
        return params.dt;
    }
    if(type == SOLVER_TYPE_EXPLICIT_RK4)
    {
        explicit_solver_rk4_step(solver, state, next_state, params, step_info, stats_or_null);
        return params.dt;
    }
    if(type == SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE)
    {
        return explicit_solver_rk4_adaptive_step(solver, state, next_state, params, step_info, stats_or_null);
    }

    assert(false);
    return false;
}


double explicit_solver_choose_and_copute_step_residual(Sim_Solver_Type type, Explicit_Solver* solver, Explicit_State state, Explicit_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    double advance_by = explicit_solver_choose(type, solver, state, next_state, params, step_info, stats_or_null);
    if(params.do_stats_step_residual && stats_or_null)
    {
        Explicit_State combined_state = state;
        combined_state.U = next_state->U;

        Explicit_State corrected_next = {0};
        Cache_Tag tag = cache_tag_make();
        int N = params.ny * params.nx;
        corrected_next.F = cache_alloc(Real, N, &tag);
        corrected_next.U = cache_alloc(Real, N, &tag);
        corrected_next.nx = params.nx;
        corrected_next.ny = params.ny;

        Sim_Params changed_params = params;
        changed_params.do_debug = false;
        changed_params.do_stats = false;
        changed_params.do_stats_step_residual = false;

        explicit_solver_choose(type, solver, combined_state, &corrected_next, changed_params, step_info, stats_or_null);

        Reduce::Stats<Real> step_res_stats = cuda_stats_delta(corrected_next.F, next_state->F, state.nx*state.ny);
        Sim_Stats_Vectors* vectors = &stats_or_null->vectors;
        float_array_push(&vectors->step_res_L1[0], step_res_stats.L1);
        float_array_push(&vectors->step_res_L2[0], step_res_stats.L2);
        float_array_push(&vectors->step_res_min[0], step_res_stats.min);
        float_array_push(&vectors->step_res_max[0], step_res_stats.max);
        stats_or_null->step_res_count = 1;

        // LOG_DEBUG("SOLVER", "%lli step residual | avg: %e | max: %e", (long long) iter, (double) stats_or_null->L2_step_residuals[0], (double) stats_or_null->Lmax_step_residuals[0]);

        cache_free(&tag);
    }

    if(params.do_stats && stats_or_null)
    {
        Reduce::Stats<Real> F_stats = cuda_stats_delta(state.F, next_state->F, state.nx*state.ny);
        Reduce::Stats<Real> U_stats = cuda_stats_delta(state.U, next_state->U, state.nx*state.ny);
        Sim_Stats_Vectors* vectors = &stats_or_null->vectors;

        float_array_push(&vectors->iter, (double) step_info.iter);
        float_array_push(&vectors->time, (double) step_info.sim_time);

        float_array_push(&vectors->phi_delta_L1, F_stats.L1);
        float_array_push(&vectors->phi_delta_L2, F_stats.L2);
        float_array_push(&vectors->phi_delta_min, F_stats.min);
        float_array_push(&vectors->phi_delta_max, F_stats.max);

        float_array_push(&vectors->T_delta_L1, U_stats.L1);
        float_array_push(&vectors->T_delta_L2, U_stats.L2);
        float_array_push(&vectors->T_delta_min, U_stats.min);
        float_array_push(&vectors->T_delta_max, U_stats.max);
    }

    return advance_by;
}

void explicit_solver_get_maps(Explicit_Solver* solver, Explicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t) map_count);

    #define ASSIGN_MAP_NAMED(var_ptr, var_name) \
        if(__map_i < map_count) \
        { \
            maps[__map_i].data = var_ptr; \
            maps[__map_i].name = var_name; \
            maps[__map_i].nx = solver->nx; \
            maps[__map_i].ny = solver->ny; \
            __map_i += 1; \
        }\

    #define ASSIGN_MAP(var_ptr) ASSIGN_MAP_NAMED(var_ptr, #var_ptr) 

    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");
    ASSIGN_MAP_NAMED(solver->debug_maps.aniso_factor, "aniso_factor");
    ASSIGN_MAP_NAMED(solver->debug_maps.reaction, "reaction");
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residual, "step_residual");
    ASSIGN_MAP_NAMED(solver->debug_maps.perlin, "perlin");
    ASSIGN_MAP_NAMED(solver->debug_maps.simplex, "simplex");
}

struct Cross_Matrix_Static {
    Real C;
    Real U;
    Real D;
    Real L;
    Real R;

    int nx;
    int ny;

    Sim_Boundary_Type boundary;
};

struct Anisotrophy_Matrix {
    Real* anisotrophy;
    Real X;
    Real Y;
    Real C_minus_one;

    int nx;
    int ny;

    Sim_Boundary_Type boundary;
};

void cross_matrix_static_multiply(Real* out, const void* _A, const Real* vec, int N)
{
    Cross_Matrix_Static A = *(Cross_Matrix_Static*)_A;
    int nx = A.nx;
    int ny = A.ny;

    #ifdef USE_TILED_FOR
    cuda_tiled_for_2D<1, 1, Real>(0, 0, nx, ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Real {
            return boundary_sample(vec, x, y, nx, ny, A.boundary);
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
            Real val = shared[tx + ty*tile_size_x]*A.C;
            val += shared[tx+1 + ty*tile_size_x]*A.R;
            val += shared[tx-1 + ty*tile_size_x]*A.L;
            val += shared[tx   + (ty+1)*tile_size_x]*A.U;
            val += shared[tx   + (ty-1)*tile_size_x]*A.D;

            out[x + y*nx] = val;
        }
    );
    #else
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;
        Real val = vec[i]*A.C;
        val += boundary_sample(vec, x+1, y, nx, ny, A.boundary)*A.R;
        val += boundary_sample(vec, x-1, y, nx, ny, A.boundary)*A.L;
        val += boundary_sample(vec, x, y+1, nx, ny, A.boundary)*A.U;
        val += boundary_sample(vec, x, y-1, nx, ny, A.boundary)*A.D;
        out[i] = val;
    });
    #endif
}

void anisotrophy_matrix_multiply(Real* out, const void* _A, const Real* vec, int N)
{
    Anisotrophy_Matrix A = * (Anisotrophy_Matrix*)_A;
    int nx = A.nx;
    int ny = A.ny;

    #ifdef USE_TILED_FOR
    cuda_tiled_for_2D<1, 1, Real>(0, 0, nx, ny,
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Real {
            return boundary_sample(vec, x, y, nx, ny, A.boundary);
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
            int i = x + y*nx;
            Real s = A.anisotrophy[i];
            Real X = A.X*s;
            Real Y = A.Y*s;
            Real C = 1 + A.C_minus_one*s;

            Real val = shared[tx + ty*tile_size_x]*C;
            val += shared[tx+1 + ty*tile_size_x]*X;
            val += shared[tx-1 + ty*tile_size_x]*X;
            val += shared[tx   + (ty+1)*tile_size_x]*Y;
            val += shared[tx   + (ty-1)*tile_size_x]*Y;

            out[i] = val;
        }
    );
    #else
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int x, int y){
        int i = x + y*nx;
        Real s = A.anisotrophy[i];
        Real X = A.X*s;
        Real Y = A.Y*s;
        Real C = 1 + A.C_minus_one*s;

        Real val = vec[i]*C;
        val += boundary_sample(vec, x+1, y, nx, ny, A.boundary)*X;
        val += boundary_sample(vec, x-1, y, nx, ny, A.boundary)*X;
        val += boundary_sample(vec, x, y+1, nx, ny, A.boundary)*Y;
        val += boundary_sample(vec, x, y-1, nx, ny, A.boundary)*Y;
        out[i] = val;
    });
    #endif
}

typedef struct Conjugate_Gardient_Params {
    Real epsilon;
    Real tolerance;
    int max_iters;

    Real* initial_value_or_null;
} Conjugate_Gardient_Params;

typedef struct Conjugate_Gardient_Convergence {
    Real error;
    int iters;
    bool converged;
} Conjugate_Gardient_Convergence;

typedef void(*Matrix_Vector_Mul_Func)(Real* out, const void* A, const Real* x, int N);

Conjugate_Gardient_Convergence conjugate_gradient_solve(const void* A, Real* x, const Real* b, int N, Matrix_Vector_Mul_Func matrix_mul_func, const Conjugate_Gardient_Params* params_or_null)
{
    i64 start = clock_ns();
    Conjugate_Gardient_Params params = {0};
    params.epsilon = (Real) 1.0e-10;
    params.tolerance = (Real) 1.0e-5;
    params.max_iters = 10;
    if(params_or_null)
        params = *params_or_null;

    Cache_Tag tag = cache_tag_make();

    Real scaled_squared_tolerance = params.tolerance*params.tolerance*N;
    Real* r = cache_alloc(Real, N, &tag);
    Real* p = cache_alloc(Real, N, &tag);
    Real* Ap = cache_alloc(Real, N, &tag);
    Real r_dot_r = 0;

    //@TODO: IMPLEMENT FULLY (add launch params for reductions etc.)!
    static hipStream_t stream1 = NULL;
    static hipStream_t stream2 = NULL;
    static hipStream_t stream3 = NULL;
    static hipStream_t stream4 = NULL;
    if(stream1 == NULL)
    {
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
        hipStreamCreate(&stream4);
    }

    //@TODO: streams
    if(params.initial_value_or_null)
    {
        CUDA_DEBUG_TEST(hipMemcpyAsync(x, params.initial_value_or_null, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice));
        matrix_mul_func(Ap, A, params.initial_value_or_null, N);
        cuda_for(0, N, [=]SHARED(int i){
            r[i] = b[i] - Ap[i];
            p[i] = r[i];
        });

        r_dot_r = vector_dot_product(r, r, N);
    }
    else
    {
        CUDA_DEBUG_TEST(hipMemsetAsync(x, 0, sizeof(Real)*(size_t)N, stream1));
        CUDA_DEBUG_TEST(hipMemcpyAsync(r, b, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice, stream2));
        CUDA_DEBUG_TEST(hipMemcpyAsync(p, b, sizeof(Real)*(size_t)N, hipMemcpyDeviceToDevice, stream3));

        r_dot_r = vector_dot_product(b, b, N);
        // CUDA_DEBUG_TEST(hipDeviceSynchronize());
    }

    int iter = 0;
    if(r_dot_r >= scaled_squared_tolerance || iter == 0)
    {
        for(; iter < params.max_iters; iter++)
        {
            matrix_mul_func(Ap, A, p, N);
            
            Real p_dot_Ap = vector_dot_product(p, Ap, N);
            Real alpha = r_dot_r / MAX(p_dot_Ap, params.epsilon);
            
            cuda_for(0, N, [=]SHARED(int i){
                x[i] = x[i] + alpha*p[i];
                r[i] = r[i] - alpha*Ap[i];
            });

            Real r_dot_r_new = vector_dot_product(r, r, N);
            if(r_dot_r_new < scaled_squared_tolerance)
            {
                r_dot_r = r_dot_r_new;
                break;
            }

            Real beta = r_dot_r_new / MAX(r_dot_r, params.epsilon);
            cuda_for(0, N, [=]SHARED(int i){
                p[i] = r[i] + beta*p[i]; 
            });

            r_dot_r = r_dot_r_new;
        }
    }

    Conjugate_Gardient_Convergence out = {0};
    out.iters = iter;
    out.converged = iter != params.max_iters;
    out.error = sqrt(r_dot_r/N);

    i64 end = clock_ns();
    LOG_DEBUG("KERNEL", "conjugate_gradient_solve(%lli) took: %.2ems", (lli)N, (double)(end - start)*1e-6);

    cache_free(&tag);
    return out;
} 

void matrix_multiply(Real* output, const Real* A, const Real* B, int A_height, int A_width, int B_height, int B_width)
{
    assert(A_width == B_height);
    for(int y = 0; y < A_height; y++)
    {
        for(int x = 0; x < B_width; x++)
        {
            Real val = 0;
            for(int k = 0; k < A_width; k++)
                val += A[k + y*A_width]*B[x + k*B_width];

            output[x + y*B_width] = val;
        }
    }
}

void semi_implicit_solver_resize(Semi_Implicit_Solver* solver, int nx, int ny)
{
    if(solver->nx != nx || solver->ny != ny)
    {
        //Big evil programming practices because we are cool and we know
        // what we are doing and dont care much about what others have to
        // say
        //@TODO: make this on demand load
        size_t N = (size_t) (ny*nx);
        size_t N_old = (size_t) (solver->ny*solver->nx);

        void** maps = (void**) (void*) &solver->maps;
        for(uint i = 0; i < sizeof(solver->maps) / sizeof(Real*); i++)
            cuda_realloc_in_place(&maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        void** debug_maps = (void**) (void*) &solver->debug_maps;
        for(uint i = 0; i < sizeof(solver->debug_maps) / sizeof(Real*); i++)
            cuda_realloc_in_place(&debug_maps[i], N*sizeof(Real), N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void semi_implicit_solver_step_based(Semi_Implicit_Solver* solver, Real* F, Real* U, Real* U_base, Semi_Implicit_State next_state, Sim_Params params, size_t iter, bool do_debug)
{
    Real dx = (Real) params.L0 / solver->nx;
    Real dy = (Real) params.L0 / solver->ny;

    int nx = solver->nx;
    int ny = solver->ny;
    int N = nx*ny;

    Real a = (Real) params.a;
    Real b = (Real) params.b;
    Real alpha = (Real) params.alpha;
    Real beta = (Real) params.beta;
    Real xi = (Real) params.xi;
    Real Tm = (Real) params.Tm;
    Real L = (Real) params.L; 
    Real dt = (Real) params.dt;
    Real S0 = (Real) params.S; 
    Real m0 = (Real) params.m0; 
    Real theta0 = (Real) params.theta0;
    Real gamma = (Real) params.gamma;
    
    Real* F_next = next_state.F;
    Real* U_next = next_state.U;
    
    Real* b_F = solver->maps.b_F;
    Real* b_U = solver->maps.b_U;

    Real one_over_2dx = 1/(2*dx);
    Real one_over_2dy = 1/(2*dy);
    Real one_over_dx2 = 1/(dx*dx);
    Real one_over_dy2 = 1/(dy*dy);
    Real k0_factor = a/(xi*xi * alpha);
    Real k2_factor = b*beta/alpha;
    Real k1_factor = 1/alpha;
    Real dt_L = dt*L;

    Sim_Boundary_Type U_bound = params.T_boundary;
    Sim_Boundary_Type F_bound = params.Phi_boundary;

    Anisotrophy_Matrix A_F = {0};
    A_F.anisotrophy = solver->maps.anisotrophy;
    A_F.C_minus_one = 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_F.X = -dt/(dx*dx);
    A_F.Y = -dt/(dy*dy);
    A_F.nx = nx;
    A_F.ny = ny;

    Cross_Matrix_Static A_U = {0};
    A_U.C = 1 + 2*dt/(dx*dx) + 2*dt/(dy*dy);
    A_U.R = -dt/(dx*dx);
    A_U.L = -dt/(dx*dx);
    A_U.U = -dt/(dy*dy);
    A_U.D = -dt/(dy*dy);
    A_U.nx = nx;
    A_U.ny = ny;

    bool do_corrector_guess = params.do_corrector_guess;
    bool is_tiled = true;
    Cache_Tag tag = cache_tag_make();

    //@TODO: factor out
    static hipEvent_t start = NULL;
    static hipEvent_t stop = NULL;
    if(start == NULL || stop == NULL)
    {
        CUDA_TEST(hipEventCreate(&start));
        CUDA_TEST(hipEventCreate(&stop));
    }
    CUDA_TEST(hipEventRecord(start, 0));

    if(do_corrector_guess)
    {
        cuda_tiled_for_2D<1, 1, Bundled>(0, 0, nx, ny,
            [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Bundled {
                Real Phi = boundary_sample(F, x, y, nx, ny, F_bound);
                Real T = boundary_sample(U, x, y, nx, ny, U_bound);
                return Bundled{Phi, T};
            },
            [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Bundled* shared){
                Bundled C = shared[tx   + ty*tile_size_x];
                Bundled E = shared[tx+1 + ty*tile_size_x];
                Bundled W = shared[tx-1 + ty*tile_size_x];
                Bundled N = shared[tx   + (ty+1)*tile_size_x];
                Bundled S = shared[tx   + (ty-1)*tile_size_x];

                Real grad_Phi_x = (E.Phi - W.Phi)*one_over_2dx;
                Real grad_Phi_y = (N.Phi - S.Phi)*one_over_2dy;
                Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);

                Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
                Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

                Real laplace_Phi = (W.Phi - 2*C.Phi + E.Phi)*one_over_dx2 + (S.Phi - 2*C.Phi + N.Phi)*one_over_dy2;
                Real laplace_T =   (W.T - 2*C.T + E.T)*one_over_dx2 +       (S.T - 2*C.T + N.T)*one_over_dy2;

                Real k0 = g_theta*f0(C.Phi)*k0_factor;
                Real k2 = grad_Phi_norm*k2_factor;
                Real k1 = g_theta*k1_factor;
                Real corr = 1 + k2*dt_L;

                Real right = C.Phi + dt/corr*((1-gamma)*k1*laplace_Phi + k0 - k2*(C.T - Tm + dt*laplace_T));
                Real factor = gamma/corr*k1; 

                A_F.anisotrophy[x+y*nx] = (Real) factor;
                b_F[x + y*nx] = (Real) right;
            }
        );
    }
    else
    {
        cuda_tiled_for_2D<1, 1, Real>(0, 0, nx, ny,
            [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry) -> Real {
                return boundary_sample(F, x, y, nx, ny, F_bound);
            },
            [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Real* shared){
                Real C_T   = U[x + y*nx];
                Real C_Phi = shared[tx   + ty*tile_size_x];
                Real E_Phi = shared[tx+1 + ty*tile_size_x];
                Real W_Phi = shared[tx-1 + ty*tile_size_x];
                Real N_Phi = shared[tx   + (ty+1)*tile_size_x];
                Real S_Phi = shared[tx   + (ty-1)*tile_size_x];

                Real grad_Phi_x = (E_Phi - W_Phi)/(2*dx);
                Real grad_Phi_y = (N_Phi - S_Phi)/(2*dy);
                Real grad_Phi_norm = custom_hypot(grad_Phi_x, grad_Phi_y);


                Real theta = custom_atan2(grad_Phi_y, grad_Phi_x);
                Real g_theta = (Real) 1 - S0*custom_cos(m0*theta + theta0);

                Real laplace_Phi = (W_Phi - 2*C_Phi + E_Phi)*one_over_dx2 + (S_Phi - 2*C_Phi + N_Phi)*one_over_dy2;

                Real k0 = g_theta*f0(C_Phi)*k0_factor;
                Real k2 = grad_Phi_norm*k2_factor;
                Real k1 = g_theta*k1_factor;

                Real right = C_Phi + dt*((1-gamma)*k1*laplace_Phi + k0 - k2*(C_T - Tm));
                Real factor = gamma*k1; 

                A_F.anisotrophy[x+y*nx] = (Real) factor;
                b_F[x + y*nx] = (Real) right;
            }
        );
    }

    CUDA_TEST(hipEventRecord(stop, 0));
    CUDA_TEST(hipEventSynchronize(stop));

    float time = 0;
    CUDA_TEST(hipEventElapsedTime(&time, start, stop));
    LOG_DEBUG("SOLVER", "Prepare kernel time %.2ems corrector_guess:%s tiled:%s", (double)time, 
        do_corrector_guess ? "true" : "false", 
        is_tiled ? "true" : "false");

    Conjugate_Gardient_Params solver_params = {0};
    solver_params.epsilon = (Real) 1.0e-12;
    solver_params.tolerance = (Real) params.Phi_tolerance;
    solver_params.max_iters = params.Phi_max_iters;
    solver_params.initial_value_or_null = F;

    //Solve A_F*F_next = b_F
    Conjugate_Gardient_Convergence F_converged = conjugate_gradient_solve(&A_F, F_next, b_F, N, anisotrophy_matrix_multiply, &solver_params);
    LOG_DEBUG("SOLVER", "%lli F %s in %i iters with error %e\n", (lli) iter, F_converged.converged ? "converged" : "diverged", F_converged.iters, (double)F_converged.error);

    //Calculate b_U
    cuda_for(0, nx*ny, [=]SHARED(csize i){
        Real T =  U_base[i];
        Real Phi = F[i];
        Real Phi_next = F_next[i];

        b_U[i] = (Real) (T + L*(Phi_next - Phi) + dt*(1-gamma)*T);
    });

    solver_params.tolerance = (Real) params.T_tolerance;
    solver_params.max_iters = params.T_max_iters;
    solver_params.initial_value_or_null = U;

    //Solve A_U*U_next = b_U
    Conjugate_Gardient_Convergence U_converged = conjugate_gradient_solve(&A_U, U_next, b_U, N, cross_matrix_static_multiply, &solver_params);
    LOG_DEBUG("SOLVER", "%lli U %s in %i iters with error %e\n", (lli) iter, U_converged.converged ? "converged" : "diverged", U_converged.iters, (double)U_converged.error);

    if(do_debug)
    {
        Real* AfF = solver->debug_maps.AfF;
        Real* AuU = solver->debug_maps.AuU;
        //Back test
        if(1)
        {
            anisotrophy_matrix_multiply(AfF, &A_F, F_next, N);
            cross_matrix_static_multiply(AuU, &A_U, U_next, N);

            Real back_error_F = vector_get_l2_dist(AfF, b_F, N);
            Real back_error_U = vector_get_l2_dist(AuU, b_U, N);

            Real back_error_F_max = vector_get_max_dist(AfF, b_F, N);
            Real back_error_U_max = vector_get_max_dist(AuU, b_U, N);

            LOG_DEBUG("SOLVER", "AVG | F:%e U:%e Epsilon:%e \n", (double) back_error_F, (double) back_error_U, (double) solver_params.tolerance*2);
            LOG_DEBUG("SOLVER", "MAX | F:%e U:%e Epsilon:%e \n", (double) back_error_F_max, (double) back_error_U_max, (double) solver_params.tolerance*2);
        }
        
        calc_debug_values(next_state.F, next_state.U, solver->debug_maps.grad_phi, solver->debug_maps.grad_T, solver->debug_maps.aniso_factor, params);
    }

    cache_free(&tag);
}

void semi_implicit_solver_step(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Sim_Params params, size_t iter, bool do_debug)
{
    semi_implicit_solver_step_based(solver, state.F, state.U, state.U, next_state, params, iter, do_debug);
}

void semi_implicit_solver_step_corrector(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Semi_Implicit_State next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    Cache_Tag tag = cache_tag_make();
    int N = params.ny * params.nx;

    Explicit_State temp_state = {0};
    temp_state.F = cache_alloc(Real, N, &tag);
    temp_state.U = cache_alloc(Real, N, &tag);
    temp_state.nx = params.nx;
    temp_state.ny = params.ny;

    static int last_num_steps = 0;

    //Init states in such a way that the resutl will already be in 
    // next_state (thus no need to copy)
    Explicit_State steps[2] = {0};
    if(last_num_steps % 2 == 0)
    {
        steps[0] = next_state;
        steps[1] = temp_state;
    }
    else
    {
        steps[1] = next_state;
        steps[0] = temp_state;
    }

    size_t max_iters = (size_t) params.corrector_max_iters;
    if(params.do_corrector_loop == false)
        max_iters = 0;
    else if(max_iters == 0 && params.do_stats_step_residual)
        max_iters = 1;

    //Perform first step
    semi_implicit_solver_step(solver, state, steps[0], params, step_info.iter, params.do_debug);
    for(size_t k = 0; k < max_iters; k++)
    {
        Explicit_State step_curr = steps[MOD(k, 2)];
        Explicit_State step_next = steps[MOD(k + 1, 2)];

        log_group();
        semi_implicit_solver_step_based(solver, state.F, step_curr.U, state.U, step_next, params, step_info.iter, false);
        if(params.do_stats_step_residual && stats_or_null)
        {
            Reduce::Stats<Real> stats = cuda_stats_delta(step_curr.F, step_next.F, N);
            Real step_residual_max_error = MAX(stats.max, -stats.min);

            Sim_Stats_Vectors* vectors = &stats_or_null->vectors;
            float_array_push(&vectors->step_res_L1[k], stats.L1);
            float_array_push(&vectors->step_res_L2[k], stats.L2);
            float_array_push(&vectors->step_res_min[k], stats.min);
            float_array_push(&vectors->step_res_max[k], stats.max);
            stats_or_null->step_res_count = max_iters;
            
            LOG_DEBUG("SOLVER", "step residual loop: %i | avg: %e | max: %e", k, 
                (double) stats.L1, (double) step_residual_max_error, params.corrector_tolerance);
        }
        log_ungroup();
        last_num_steps = k;
    }

    //If the ended on step is already next_state dont copy anything
    Explicit_State final_step = steps[MOD(last_num_steps, 2)];
    if(final_step.F != next_state.F)
    {
        CUDA_DEBUG_TEST(hipMemcpyAsync(next_state.F, final_step.F, (size_t)N*sizeof(Real), hipMemcpyDeviceToDevice));
        CUDA_DEBUG_TEST(hipMemcpyAsync(next_state.U, final_step.U, (size_t)N*sizeof(Real), hipMemcpyDeviceToDevice));
    }

    if(params.do_stats && stats_or_null)
    {
        Reduce::Stats<Real> F_stats = cuda_stats_delta(state.F, next_state.F, state.nx*state.ny);
        Reduce::Stats<Real> U_stats = cuda_stats_delta(state.U, next_state.U, state.nx*state.ny);
        Sim_Stats_Vectors* vectors = &stats_or_null->vectors;

        float_array_push(&vectors->iter, (double) step_info.iter);
        float_array_push(&vectors->time, (double) step_info.sim_time);
        float_array_push(&vectors->phi_iters, last_num_steps);
        float_array_push(&vectors->T_iters, last_num_steps);

        float_array_push(&vectors->phi_delta_L1, F_stats.L1);
        float_array_push(&vectors->phi_delta_L2, F_stats.L2);
        float_array_push(&vectors->phi_delta_min, F_stats.min);
        float_array_push(&vectors->phi_delta_max, F_stats.max);

        float_array_push(&vectors->T_delta_L1, U_stats.L1);
        float_array_push(&vectors->T_delta_L2, U_stats.L2);
        float_array_push(&vectors->T_delta_min, U_stats.min);
        float_array_push(&vectors->T_delta_max, U_stats.max);
    }

    cache_free(&tag);
}

void semi_implicit_solver_get_maps(Semi_Implicit_Solver* solver, Semi_Implicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t)map_count);
    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
    // ASSIGN_MAP_NAMED(solver->maps.b_F, "b_F");           
    // ASSIGN_MAP_NAMED(solver->debug_maps.AfF, "AfF");           
    // ASSIGN_MAP_NAMED(solver->maps.b_U, "b_U");           
    // ASSIGN_MAP_NAMED(solver->debug_maps.AuU, "AuU");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_phi, "grad_phi");           
    ASSIGN_MAP_NAMED(solver->debug_maps.grad_T, "grad_T");           
    ASSIGN_MAP_NAMED(solver->maps.anisotrophy, "Anisotrophy");

    CHECK_BOUNDS(2, ARRAY_LEN(solver->debug_maps.step_residuals)); 
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[0], "step_residual1");          
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[1], "step_residual2");           
    ASSIGN_MAP_NAMED(solver->debug_maps.step_residuals[2], "step_residual3");           
}

struct Semi_Implicit_Coupled_Cross_Matrix {
    Anisotrophy_Matrix A_F; //A anisotrophy scaled cross matrix
    Real* B_U; //A changing diagonal 

    Cross_Matrix_Static A_U; //Static cross matrix
    Real B_F; //A single value diagonal

    int nx;
    int ny;
};

void semi_implicit_coupled_solver_resize(Semi_Implicit_Coupled_Solver* solver, int nx, int ny)
{
    if(solver->nx != nx || solver->ny != ny)
    {
        int N = ny*nx;
        int N_old = solver->ny*solver->nx;
        cuda_realloc_in_place((void**) &solver->b_C, 2*(size_t)N*sizeof(Real), 2*(size_t)N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->aniso, (size_t)N*sizeof(Real), (size_t)N_old*sizeof(Real), REALLOC_ZERO);
        cuda_realloc_in_place((void**) &solver->B_U, (size_t)N*sizeof(Real), (size_t)N_old*sizeof(Real), REALLOC_ZERO);

        solver->nx = nx;
        solver->ny = ny;
    }
}

void semi_implicit_coupled_state_resize(Semi_Implicit_Coupled_State* state, int nx, int ny)
{
    if(state->nx != nx || state->ny != ny)
    {
        int N = ny*nx;
        int N_old = state->ny*state->nx;
        cuda_realloc_in_place((void**) &state->C, 2*(size_t)N*isizeof(Real), 2*(size_t)N_old*isizeof(Real), REALLOC_ZERO);
        state->nx = nx;
        state->ny = ny;
    }
}

template <typename T>
void sim_modify_T(Real* device_memory, T* host_memory, size_t count, Sim_Modify modify)
{
    static T* static_device = NULL;
    static size_t static_size = 0;

    if(sizeof(Real) != sizeof(T))
    {
        if(static_size < count)
        {
            cuda_realloc_in_place((void**) &static_device, count*sizeof(T), static_size*sizeof(T), 0);
            static_size = count;
        }

        T* temp_device = static_device;
        if(modify == MODIFY_UPLOAD)
        {
            //Upload: host -> static -> device
            CUDA_DEBUG_TEST(hipMemcpy(temp_device, host_memory, count*sizeof(T), hipMemcpyHostToDevice));
            cuda_for(0, (int) count, [=]SHARED(int i){
                device_memory[i] = (Real) temp_device[i];
            });
        }
        else
        {
            //download: device -> static -> host
            cuda_for(0, (int) count, [=]SHARED(int i){
                temp_device[i] = (T) device_memory[i];
            });
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, temp_device, count*sizeof(T), hipMemcpyDeviceToHost));
        }
    }
    else
    {
        if(modify == MODIFY_UPLOAD)
            CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, count*sizeof(T), hipMemcpyHostToDevice));
        else
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, count*sizeof(T), hipMemcpyDeviceToHost));
    }
}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify)
{
    if(modify == MODIFY_UPLOAD)
        CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice));
    else
        CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost));
}

extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t count, Sim_Modify modify)
{   
    sim_modify_T(device_memory, host_memory, count, modify);
}

extern "C" void sim_modify_double(Real* device_memory, double* host_memory, size_t count, Sim_Modify modify)
{   
    sim_modify_T(device_memory, host_memory, count, modify);
}

extern "C" void sim_solver_reinit(Sim_Solver* solver, Sim_Solver_Type type, int nx, int ny)
{
    if(solver->type != type && solver->type != SOLVER_TYPE_NONE)
        sim_solver_reinit(solver, solver->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            ny = 0;
            nx = 0;
        } break;

        case SOLVER_TYPE_EXACT: {
            //void
        } break;

        case SOLVER_TYPE_EXPLICIT_EULER: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_solver_resize(&solver->expli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_resize(&solver->impli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_solver_resize(&solver->impli_coupled, nx, ny);
        } break;

        default: {
            assert(false);
        }
    };

    solver->type = type;
    solver->nx = nx;
    solver->ny = ny;
}

void sim_state_reinit(Sim_State* states, Sim_Solver_Type type, int nx, int ny)
{
    if(states->type != type && states->type != SOLVER_TYPE_NONE)
        sim_state_reinit(states, states->type, 0, 0);

    switch(type) {
        case SOLVER_TYPE_NONE: {
            ny = 0;
            nx = 0;
        } break;

        case SOLVER_TYPE_EXACT: 
        case SOLVER_TYPE_EXPLICIT_EULER: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_state_resize(&states->expli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            explicit_state_resize(&states->impli, nx, ny);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT_COUPLED: {
            semi_implicit_coupled_state_resize(&states->impli_coupled, nx, ny);
        } break;

        default: {
            assert(false);
        }
    };

    states->type = type;
    states->nx = nx;
    states->ny = ny;
}

extern "C" void sim_states_reinit(Sim_State* states, int state_count, Sim_Solver_Type type, int nx, int ny)
{
    for(int i = 0; i < state_count; i++)
        sim_state_reinit(&states[i], type, nx, ny);
}

void exact_solver_step(Sim_Solver* solver, Exact_State state, Exact_State* next_state, Sim_Params params, Sim_Step_Info step_info, Sim_Stats* stats_or_null)
{
    Real dx = (Real) params.L0 / solver->nx;
    Real dy = (Real) params.L0 / solver->ny;
    Real L0 = params.L0;
    int nx = solver->nx;
    int ny = solver->ny;
    
    Exact_Params exact_params = get_static_exact_params(params);

    Real* U_next = next_state->U;
    Real* F_next = next_state->F;
    cuda_for_2D(0, 0, nx, ny, [=]SHARED(int xi, int yi){
        Real x = ((Real) xi + 0.5)*dx - L0/2;
        Real y = ((Real) yi + 0.5)*dy - L0/2;
        Real r = hypot(x, y);


        Real u = exact_u(step_info.sim_time, r, exact_params);
        Real phi = exact_phi(step_info.sim_time, r, exact_params);
        U_next[xi + yi*nx] = u;
        F_next[xi + yi*nx] = phi;
    });


    if(params.do_stats && stats_or_null)
    {
        Reduce::Stats<Real> F_stats = cuda_stats_delta(state.F, next_state->F, state.nx*state.ny);
        Reduce::Stats<Real> U_stats = cuda_stats_delta(state.U, next_state->U, state.nx*state.ny);
        Sim_Stats_Vectors* vectors = &stats_or_null->vectors;

        float_array_push(&vectors->iter, (double) step_info.iter);
        float_array_push(&vectors->time, (double) step_info.sim_time);

        float_array_push(&vectors->phi_delta_L1, F_stats.L1);
        float_array_push(&vectors->phi_delta_L2, F_stats.L2);
        float_array_push(&vectors->phi_delta_min, F_stats.min);
        float_array_push(&vectors->phi_delta_max, F_stats.max);

        float_array_push(&vectors->T_delta_L1, U_stats.L1);
        float_array_push(&vectors->T_delta_L2, U_stats.L2);
        float_array_push(&vectors->T_delta_min, U_stats.min);
        float_array_push(&vectors->T_delta_max, U_stats.max);
    }
}

void exact_solver_get_maps(Sim_Solver* solver, Explicit_State state, Sim_Map* maps, int map_count)
{
    int __map_i = 0;
    memset(maps, 0, sizeof maps[0] * (size_t) map_count);
    ASSIGN_MAP_NAMED(state.F, "Phi");            
    ASSIGN_MAP_NAMED(state.U, "T");            
}
extern "C" double sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, Sim_Step_Info info, Sim_Params params, Sim_Stats* stats_or_null)
{
    int required_history = solver_type_required_history(solver->type);
    const char* solver_name = solver_type_to_cstring(solver->type);
    
    bool okay = true;
    if(states_count < required_history)
    {
        okay = false;
        LOG_INFO("SOLVER", "Step: Not enough history for solver %s! Required %i. Got %i", solver_name, states_count, required_history);
    }
    else
    {
        for(int i = 0; i < states_count; i++)
        {
            if(states[i].type != solver->type)
            {
                LOG_INFO("SOLVER", "Step: state[%i] is of bad type %s. Expected %s", solver_type_to_cstring(states[i].type), solver_name);
                okay = false;
            }
        }
    }
    
    double step_by = 0;
    if(okay)
    {
        step_by = params.dt;
        ASSERT(states_count > 0);
        Sim_State state = states[MOD(info.iter, (size_t) states_count)];
        Sim_State next_state = states[MOD(info.iter + 1, (size_t) states_count)];
        switch(solver->type) {
            case SOLVER_TYPE_NONE: {
                LOG_INFO("SOLVER", "Step: stepping as solver type none has no effect");
                // nothing
            } break;

            case SOLVER_TYPE_EXACT: {
                exact_solver_step(solver, state.exact, &next_state.exact, params, info, stats_or_null);
            } break;

            case SOLVER_TYPE_EXPLICIT_EULER: 
            case SOLVER_TYPE_EXPLICIT_RK4: 
            case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
                step_by = explicit_solver_choose_and_copute_step_residual(solver->type, &solver->expli, state.expli, &next_state.expli, params, info, stats_or_null);
            } break;

            case SOLVER_TYPE_SEMI_IMPLICIT: {
                semi_implicit_solver_step_corrector(&solver->impli, state.impli, next_state.impli, params, info, stats_or_null);
            } break;
            default: assert(false);
        };
    }

    CUDA_DEBUG_TEST(hipDeviceSynchronize());
    return step_by;
}

extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count)
{
    if(states_count <= 0 || map_count <= 0)
        return;

    Sim_State state = states[MOD(iter, states_count)];
    switch(solver->type) {
        case SOLVER_TYPE_NONE: {
            //none
        } break;

        case SOLVER_TYPE_EXACT: {
            exact_solver_get_maps(solver, state.exact, maps, map_count);
        } break;

        case SOLVER_TYPE_EXPLICIT_EULER: 
        case SOLVER_TYPE_EXPLICIT_RK4:
        case SOLVER_TYPE_EXPLICIT_RK4_ADAPTIVE: {
            explicit_solver_get_maps(&solver->expli, state.expli, maps, map_count);
        } break;

        case SOLVER_TYPE_SEMI_IMPLICIT: {
            semi_implicit_solver_get_maps(&solver->impli, state.impli, maps, map_count);
        } break;

        default: assert(false);
    };
}


#else

extern "C" void sim_solver_reinit(Sim_Solver* solver, Sim_Solver_Type type, int nx, int ny) {}
extern "C" void sim_states_reinit(Sim_State* states, int state_count, Sim_Solver_Type type, int nx, int ny) {}
extern "C" void sim_solver_get_maps(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Map* maps, int map_count) {}
extern "C" double sim_solver_step(Sim_Solver* solver, Sim_State* states, int states_count, int iter, Sim_Params params, Sim_Stats* stats_or_null) {return 0;}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify) {}
extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t size, Sim_Modify modify) {}
extern "C" void sim_modify_double(Real* device_memory, double* host_memory, size_t size, Sim_Modify modify) {}
#endif

#include "cuda_reduction.cuh"
#include "cuda_random.cuh"

#ifdef COMPILE_BENCHMARKS
static void cache_prepare(int count, int item_size, int N)
{
    Cache_Tag tag = cache_tag_make();
    for(int i = 0; i < count; i++)
        _cache_alloc((size_t) (item_size*N), &tag, SOURCE_INFO());
    cache_free(&tag);
}

extern "C" bool run_benchmarks(int N_)
{
    csize N = (csize) N_;
    cache_prepare(3, sizeof(int), N);
    cache_prepare(3, sizeof(float), N);
    cache_prepare(3, sizeof(double), N);

    Cache_Tag tag = cache_tag_make();
    uint* rand_state = cache_alloc(uint, N, &tag);
    random_map_seed_32(rand_state, N, (uint32_t) clock_ns());

    int GB = 1024*1024*1024;
    {
        double* rand_map = cache_alloc(double, N, &tag);
        random_map_32(rand_map, rand_state, N);
        
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_map, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_map, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_map, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(double);
        LOG_OKAY("BENCH", "double (gb/s): cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(double)).str);
        LOG_OKAY("BENCH", "double (time): cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }
    {
        float* rand_map = cache_alloc(float, N, &tag);
        random_map_32(rand_map, rand_state, N);
        
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_map, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_map, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_map, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(float);
        LOG_OKAY("BENCH", "float (gb/s) : cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(float)).str);
        LOG_OKAY("BENCH", "float (time) : cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }

    {
        double cpu_time = benchmark(3, [=]{ cpu_reduce(rand_state, N, Reduce::ADD); });
        double thrust_time = benchmark(3, [=]{ thrust_reduce(rand_state, N, Reduce::ADD); });
        double custom_time = benchmark(3, [=]{ cuda_reduce(rand_state, N, Reduce::ADD); });
        double total_gb = (double) N / GB * sizeof(uint);
        LOG_OKAY("BENCH", "uint (gb/s)  : cpu %5.2lf | thrust: %5.2lf | custom: %5.2lf (N:%i %s)", 
            total_gb/cpu_time, total_gb/thrust_time, total_gb/custom_time, N, format_bytes((size_t)N * sizeof(uint)).str);
        LOG_OKAY("BENCH", "uint (time)  : cpu: %e | thrust: %e | custom: %e", N, cpu_time, thrust_time, custom_time);
    }

    cache_free(&tag);
    return true;
}
#else
extern "C" bool run_benchmarks(int N)
{
    (void) N;
    return false;
}
#endif

#include "cuda_examples.cuh"
extern "C" bool run_tests()
{
    test_all_examples(3);
    #ifdef TEST_CUDA_FOR_IMPL
    test_tiled_for((uint64_t) clock_ns());
    test_tiled_for_2D((uint64_t) clock_ns());
    #endif
    #ifdef TEST_CUDA_REDUCTION_IMPL
    test_reduce((uint64_t) clock_ns());
    #endif

    return true;
}