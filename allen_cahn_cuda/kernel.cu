#include "hip/hip_runtime.h"

#define _CRT_SECURE_NO_WARNINGS
#define JOT_ALL_IMPL
#define JMAPI __host__ __device__ static inline

#include "config.h"
#include "integration_methods.h"
#include "render.h"
#include "cuprintf.cuh"
#include "cuprintf.cu"

#include "lib/platform.h"
#include "lib/log.h"
#include "lib/logger_file.h"
#include "lib/allocator_debug.h"
#include "lib/allocator_malloc.h"
#include "lib/error.h"
#include "lib/time.h"
#include "lib/image.h"
#include "lib/format_netbpm.h"
#include "lib/math.h"

#include "glfw/glfw3.h"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <>

const i32 SCR_WIDTH = 1000;
const i32 SCR_HEIGHT = 1000;

const f64 FPS_DISPLAY_FREQ = 50000;
const f64 RENDER_FREQ = 30;

const f64 FREE_RUN_SYM_FPS = 200;

#define CONFIG_FILE "config.lpf"

typedef struct App_State {
    GLFWwindow* window;

    bool is_in_step_mode;
    bool render_phi;
    f64 remaining_steps;
    f64 step_by;
} App_State;

void app_state_init(App_State* state, GLFWwindow* window)
{
    state->window = window;
    state->is_in_step_mode = true;
    state->remaining_steps = 0;
    state->step_by = 1;
    state->render_phi = true;
}

JMAPI f32 map_at(const f32* map, int x, int y, Allen_Cahn_Params params)
{
    int x_mod = x % params.mesh_size_x;
    int y_mod = y % params.mesh_size_y;

    return map[x_mod + y_mod*params.mesh_size_x];
}

JMAPI f32 allen_cahn_reaction_term_0(f32 phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}

JMAPI f32 allen_cahn_reaction_term_1(f32 phi, f32 T, f32 xi, Allen_Cahn_Params params)
{
    f32 mK = 1;
	return (params.a*allen_cahn_reaction_term_0(phi) - params.b*params.beta*xi*(T - params.Tm))*mK;
}

JMAPI f32 allen_cahn_reaction_term_2(f32 phi, f32 T, f32 xi, Vec2 grad_phi, Allen_Cahn_Params params)
{
    f32 mK = 1;
	f32 grad_val = vec2_len(grad_phi);
	return (params.a*allen_cahn_reaction_term_0(phi) - params.b*params.beta*xi*xi*grad_val*(T - params.Tm))*mK;
}

__global__ void allen_cahn_simulate(f32* phi_map_next, f32* T_map_next, const f32* phi_map, const f32* T_map, Allen_Cahn_Params params, isize iter)
{
    f32 tau = 1;
    f32 mK = 1;
    f32 grad_interp = 0.5f;

    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < params.mesh_size_x; x += blockDim.x * gridDim.x) 
    {
        for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < params.mesh_size_x; y += blockDim.y * gridDim.y) 
        {
	        f32 T = map_at(T_map, x, y, params);
	        f32 phi = map_at(phi_map, x, y, params);

	        f32 phi_py = map_at(phi_map, x, y + 1, params);
	        f32 phi_my = map_at(phi_map, x, y - 1, params);
	        f32 phi_px = map_at(phi_map, x + 1, y, params);
	        f32 phi_mx = map_at(phi_map, x - 1, y, params);

	        f32 T_py = map_at(T_map, x, y + 1, params);
	        f32 T_my = map_at(T_map, x, y - 1, params);
	        f32 T_px = map_at(T_map, x + 1, y, params);
	        f32 T_mx = map_at(T_map, x - 1, y, params);
            
            if(y == 0)
                if(T > 0 || phi > 0)
                {
                    //cuPrintf("%lli (%d, %d) {T: %16.8f phi: %16.8f} \n", iter, x, y, T, phi);
                }

	        f32 sum_phi_neigbours = 0
		        + tau*(phi_py - phi)
		        + tau*(phi_my - phi)
		        + tau*(phi_px - phi)
		        + tau*(phi_mx - phi);
		
	        f32 sum_T_neigbours = 0
		        + tau*(T_py - T)
		        + tau*(T_my - T)
		        + tau*(T_px - T)
		        + tau*(T_mx - T);

	        Vec2 grad_phi = {
		        (phi_px - phi_mx) * grad_interp,
		        (phi_py - phi_my) * grad_interp
	        };
        
	        f32 reaction_term = allen_cahn_reaction_term_2(phi, T, params.xi, grad_phi, params);
	        //f32 reaction_term = 0;
	        f32 phi_dt = (sum_phi_neigbours/mK + reaction_term/(params.xi*params.xi)) / params.alpha;
	        f32 T_dt = sum_T_neigbours / mK + params.L * phi_dt;

	        f32 phi_next = phi_dt * params.dt + phi;
	        f32 T_next = T_dt * params.dt + T;
		
            phi_map_next[x + y*params.mesh_size_x] = phi_next;
            T_map_next[x + y*params.mesh_size_x] = T_next;
        }
    }
}

void _test_cuda(hipError_t error, const char* expression, Source_Info info)
{
    if(error != hipSuccess)
    {
        assertion_report(expression, info, "cuda failed with error %s", hipGetErrorString(error));
        platform_trap();
        platform_abort();
    }
}

#define CUDA_TEST(status) _test_cuda((status), #status, SOURCE_INFO())
#define CUDA_ERR_AND(err) (err) != hipSuccess ? (err) :

Error cuda_error(hipError_t error);
String cuda_translate_error(u32 code, void* context);
void render_cuda_memory(App_State* app, Compute_Texture texture, const f32* cuda_memory, f32 min, f32 max);
void allen_cahn_custom_config(Allen_Cahn_Config* out_config);

bool file_change_func(void* context)
{
    Allen_Cahn_Config* config = (Allen_Cahn_Config*) context;
    Allocator_Set prev_set = allocator_set_both(config->allocator, config->allocator);
    TEST(allen_cahn_read_file_config(config, CONFIG_FILE));
    allocator_set(prev_set);

    return true;
}

void allen_cahn_set_initial_conditions(f32* initial_phi_map, f32* initial_T_map, Allen_Cahn_Config config)
{
    Allen_Cahn_Params params = config.params;
    Allen_Cahn_Initial_Conditions initial = config.initial_conditions;
    for(isize y = 0; y < params.mesh_size_y; y++)
    {
        for(isize x = 0; x < params.mesh_size_x; x++)
        {
            Vec2 pos = vec2((f32) x / params.mesh_size_x * params.sym_size, (f32) y / params.mesh_size_y * params.sym_size); 
            isize i = x + y*params.mesh_size_x;

		    if(((initial.square_from.x <= pos.x && pos.x < initial.square_to.x) && 
			    (initial.square_from.y <= pos.y && pos.y < initial.square_to.y))
			    || vec2_len(vec2_sub(initial.circle_center, pos)) < initial.circle_radius)
		    {
                initial_phi_map[i] = initial.inside_phi;
                initial_T_map[i] = initial.inside_T;
		    }
            else
            {
                initial_phi_map[i] = initial.outside_phi;
                initial_T_map[i] = initial.outside_T;
            }
        }
    }
}

void run_func_allen_cahn_cuda(void* context)
{
    compare_rk4();
    cudaPrintfInit();
    
    Allen_Cahn_Config config = {0};
    config.allocator = allocator_get_default();
    allen_cahn_custom_config(&config);
    file_change_func(&config);
    //TEST(allen_cahn_read_file_config(&config, CONFIG_FILE));
    
    Platform_File_Watch file_watch = {0};
    Error watch_error = error_from_platform(platform_file_watch(&file_watch, ".", PLATFORM_FILE_WATCH_CHANGE, file_change_func, &config));
    ASSERT_MSG(error_is_ok(watch_error), "file watch failed %s", error_code(watch_error).data);

    int device_id = 0;
    CUDA_TEST(hipSetDevice(device_id));
    
    int device_processor_count = 0;
    hipDeviceGetAttribute(&device_processor_count, hipDeviceAttributeMultiprocessorCount, device_id);
    
    isize pixel_count = config.params.mesh_size_x * config.params.mesh_size_y;

    f32* initial_phi_map = (f32*) allocator_allocate_cleared(allocator_get_default(), pixel_count * sizeof(f32), DEF_ALIGN, SOURCE_INFO());
    f32* initial_T_map = (f32*) allocator_allocate_cleared(allocator_get_default(), pixel_count * sizeof(f32), DEF_ALIGN, SOURCE_INFO());

    f32* phi_map = NULL;
    f32* T_map = NULL;
    f32* next_phi_map = NULL;
    f32* next_T_map = NULL;

    f32* display_map1 = NULL;
    f32* display_map2 = NULL;
    f32* display_map3 = NULL;

    CUDA_TEST(hipMalloc((void**)&phi_map,          pixel_count * sizeof(f32)));
    CUDA_TEST(hipMalloc((void**)&T_map,            pixel_count * sizeof(f32)));
    CUDA_TEST(hipMalloc((void**)&next_phi_map,     pixel_count * sizeof(f32)));
    CUDA_TEST(hipMalloc((void**)&next_T_map,       pixel_count * sizeof(f32)));
    //CUDA_TEST(hipMalloc((void**)&display_map1,     pixel_count * sizeof(f32)));
    //CUDA_TEST(hipMalloc((void**)&display_map2,     pixel_count * sizeof(f32)));
    //CUDA_TEST(hipMalloc((void**)&display_map3,     pixel_count * sizeof(f32)));

    allen_cahn_set_initial_conditions(initial_phi_map, initial_T_map, config);

    CUDA_TEST(hipMemcpy(phi_map, initial_phi_map, pixel_count * sizeof(f32), hipMemcpyHostToDevice));
    CUDA_TEST(hipMemcpy(T_map, initial_T_map, pixel_count * sizeof(f32), hipMemcpyHostToDevice));

    Compute_Texture output_phi_map  = compute_texture_make(config.params.mesh_size_x, config.params.mesh_size_y, PIXEL_FORMAT_F32, 1);
    Compute_Texture output_T_map    = compute_texture_make(config.params.mesh_size_x, config.params.mesh_size_y, PIXEL_FORMAT_F32, 1);

    Platform_Calendar_Time calendar_time = platform_epoch_time_to_calendar_time(platform_local_epoch_time());
    String_Builder serialized_image = {0};

    platform_directory_create(config.snapshots.folder.data);

    i64 save_counter = 0;
    i64 frame_counter = 0;
    f64 frame_time_sum = 0;
    
    f64 fps_display_last_time_sum = 0;
    f64 fps_display_last_time = 0;
    i64 fps_display_last_frames = 0;
    
    f64 render_last_time = 0;
    f64 simulated_last_time = 0;

    f64 simulation_time_sum = 0;

    GLFWwindow* window = (GLFWwindow*) context;
    App_State* app = (App_State*) glfwGetWindowUserPointer(window); (void) app;
	while (!glfwWindowShouldClose(window))
    {
        f64 now = clock_s();
        //if(0)
        if(now - render_last_time > 1.0/RENDER_FREQ)
        {
            render_last_time = now;
            if(app->render_phi)
                render_cuda_memory(app, output_phi_map, phi_map, 0, 1);
            else
                render_cuda_memory(app, output_T_map, T_map, 0, 1.5);
                
            glfwSwapBuffers(app->window);
        }

        if(now - fps_display_last_time > 1.0/FPS_DISPLAY_FREQ)
        {
            f64 time_sum_delta = frame_time_sum - fps_display_last_time_sum;
            f64 counter_delta = (f64) (frame_counter - fps_display_last_frames);
            f64 avg_fps = 0;
            if(time_sum_delta != 0)
            {
                avg_fps = counter_delta / time_sum_delta;
                glfwSetWindowTitle(window, format_ephemeral("iter %lli", (lli) frame_counter).data);
            }

            fps_display_last_time = now;
            fps_display_last_frames = frame_counter;
            fps_display_last_time_sum = frame_time_sum;
        }


        bool step_sym = false;
        if(app->is_in_step_mode)
            step_sym = app->remaining_steps > 0.5;
        else
            step_sym = now - simulated_last_time > 1.0/app->step_by/FREE_RUN_SYM_FPS;

        if(step_sym)
        {
            simulated_last_time = now;
            app->remaining_steps -= 1;

            f64 frame_start_time = clock_s();

            dim3 bs(64, 1);
            dim3 grid(device_processor_count, 1);
            allen_cahn_simulate<<<grid, bs>>>(next_phi_map, next_T_map, phi_map, T_map, config.params, frame_counter);
            CUDA_TEST(hipGetLastError());
            CUDA_TEST(hipDeviceSynchronize());
            cudaPrintfDisplay();
            
            f64 end_start_time = clock_s();

            f64 delta = end_start_time - frame_start_time;
            
            #if 0
            if(SAVE_EVERY > 0 && frame_counter % SAVE_EVERY == 0)
            {
                PERF_COUNTER_START(image_saving);
                Image_Builder pixels = {0};
                String file_name = {0};
                image_builder_init(&pixels, NULL, 1, PIXEL_FORMAT_U8);

                {
                    compute_texture_get_pixels_converted(&pixels, next_phi_map);
                    netbpm_format_pgm_write_into(&serialized_image, image_from_builder(pixels));
                
                    file_name = format_ephemeral("%s/%s_%lld-%lld-%lld_%lld-%lld-%lld_iter_%lld_phi.pgm", SAVE_FOLDER, SAVE_PREFIX, 
                        (lli) calendar_time.year, (lli) calendar_time.month, (lli) calendar_time.day, 
                        (lli) calendar_time.hour, (lli) calendar_time.minute, (lli) calendar_time.second, 
                        (lli) frame_counter);

                    file_write_entire(file_name, string_from_builder(serialized_image));
                }

                {
                    compute_texture_get_pixels_converted(&pixels, next_T_map);
                    netbpm_format_pgm_write_into(&serialized_image, image_from_builder(pixels));
                
                    file_name = format_ephemeral("%s/%s_%lld-%lld-%lld_%lld-%lld-%lld_iter_%lld_T.pgm", SAVE_FOLDER, SAVE_PREFIX, 
                        (lli) calendar_time.year, (lli) calendar_time.month, (lli) calendar_time.day, 
                        (lli) calendar_time.hour, (lli) calendar_time.minute, (lli) calendar_time.second, 
                        (lli) frame_counter);

                    file_write_entire(file_name, string_from_builder(serialized_image));
                }

                image_builder_deinit(&pixels);
                PERF_COUNTER_END(image_saving);
                
                save_counter ++;
                if(save_counter > PAUSE_AFTER_SAVES)
                {
                    save_counter = 0;
                    app->is_in_step_mode = true;
                    app->remaining_steps = 0;
                }
            }
            #endif

            frame_time_sum += delta;
            frame_counter += 1;
            simulation_time_sum += config.params.dt;

            SWAP(&phi_map, &next_phi_map, f32*);
            SWAP(&T_map, &next_T_map, f32*);

        }
        
		glfwPollEvents();
    }

    platform_file_unwatch(&file_watch);
}

void* glfw_malloc_func(size_t size, void* user);
void* glfw_realloc_func(void* block, size_t size, void* user);
void glfw_free_func(void* block, void* user);
void glfw_error_func(int code, const char* description);
void glfw_resize_func(GLFWwindow* window, int width, int heigth);
void glfw_key_func(GLFWwindow* window, int key, int scancode, int action, int mods);

void run_func(void* context);
void error_func(void* context, Platform_Sandox_Error error_code);

int main()
{
    platform_init();
    Malloc_Allocator static_allocator = {0};
    malloc_allocator_init(&static_allocator);
    allocator_set_static(&static_allocator.allocator);
    
    Malloc_Allocator malloc_allocator = {0};
    malloc_allocator_init_use(&malloc_allocator, 0);
    
    error_system_init(&static_allocator.allocator);
    file_logger_init_use(&global_logger, &malloc_allocator.allocator, &malloc_allocator.allocator);

    Debug_Allocator debug_alloc = {0};
    debug_allocator_init_use(&debug_alloc, DEBUG_ALLOCATOR_DEINIT_LEAK_CHECK | DEBUG_ALLOCATOR_CAPTURE_CALLSTACK);

    GLFWallocator allocator = {0};
    allocator.allocate = glfw_malloc_func;
    allocator.reallocate = glfw_realloc_func;
    allocator.deallocate = glfw_free_func;
    allocator.user = &malloc_allocator;
 
    glfwInitAllocator(&allocator);
    glfwSetErrorCallback(glfw_error_func);
    TEST_MSG(glfwInit(), "Failed to init glfw");

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_DEBUG_CONTEXT, true);  
 
    GLFWmonitor* monitor = glfwGetPrimaryMonitor();
    const GLFWvidmode* mode = glfwGetVideoMode(monitor);
    ASSERT(monitor && mode);
    if(monitor != NULL && mode != NULL)
    {
        glfwWindowHint(GLFW_RED_BITS, mode->redBits);
        glfwWindowHint(GLFW_GREEN_BITS, mode->greenBits);
        glfwWindowHint(GLFW_BLUE_BITS, mode->blueBits);
        glfwWindowHint(GLFW_REFRESH_RATE, mode->refreshRate);
    }
 
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Render", NULL, NULL);
    TEST_MSG(window != NULL, "Failed to make glfw window");

    App_State app = {0};
    app_state_init(&app, window);
    glfwSetWindowUserPointer(window, &app);
    glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, glfw_resize_func);
    glfwSetKeyCallback(window, glfw_key_func);
    glfwSwapInterval(0);

    int version = gladLoadGL((GLADloadfunc) glfwGetProcAddress);
    TEST_MSG(version != 0, "Failed to load opengl with glad");

    gl_debug_output_enable();

    platform_exception_sandbox(
        run_func_allen_cahn_cuda, window, 
        error_func, window);

    glfwDestroyWindow(window);
    glfwTerminate();

    debug_allocator_deinit(&debug_alloc);
    
    file_logger_deinit(&global_logger);
    error_system_deinit();

    ASSERT(malloc_allocator.bytes_allocated == 0);
    malloc_allocator_deinit(&malloc_allocator);
    platform_deinit();

    return 0;    
}

void* glfw_malloc_func(size_t size, void* user)
{
    return malloc_allocator_malloc((Malloc_Allocator*) user, size);
}

void* glfw_realloc_func(void* block, size_t size, void* user)
{
    return malloc_allocator_realloc((Malloc_Allocator*) user, block, size);
}

void glfw_free_func(void* block, void* user)
{
    malloc_allocator_free((Malloc_Allocator*) user, block);
}

void glfw_error_func(int code, const char* description)
{
    LOG_ERROR("APP", "GLWF error %d with message: %s", code, description);
}
void glfw_resize_func(GLFWwindow* window, int width, int heigth)
{
    (void) window;
	// make sure the viewport matches the new window dimensions; note that width and 
	// heigth will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, heigth);
}

void glfw_key_func(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    (void) mods;
    (void) scancode;
    (void) window;
    App_State* app = (App_State*) glfwGetWindowUserPointer(window); (void) app;
    if(action == GLFW_RELEASE)
    {
        if(key == GLFW_KEY_ENTER)
            app->remaining_steps = app->step_by;
        if(key == GLFW_KEY_SPACE)
            app->is_in_step_mode = !app->is_in_step_mode;
        if(key == GLFW_KEY_F1)
        {
            app->render_phi = !app->render_phi;
            LOG_INFO("APP", "Rendering %s", app->render_phi ? "phi" : "T");
        }

        if(key == GLFW_KEY_C)
        {
            for(Global_Perf_Counter* counter = profile_get_counters(); counter != NULL; counter = counter->next)
            {
                Perf_Counter_Stats stats = perf_counter_get_stats(counter->counter, 1);
		        LOG_INFO("APP", "total: %15.8lf avg: %12.8lf runs: %-8lli σ/μ %13.6lf [%13.6lf %13.6lf] (ms) from %-4lli %s \"%s\"", 
			        stats.total_s*1000,
			        stats.average_s*1000,
                    (lli) stats.runs,
                    stats.normalized_standard_deviation_s,
			        stats.min_s*1000,
			        stats.max_s*1000,
			        (lli) counter->line,
			        counter->function,
			        counter->name
		        );
            }
        }
        
        f64 iters_before = app->step_by;
        if(key == GLFW_KEY_O)
            app->step_by = app->step_by*1.3 + 1;
        if(key == GLFW_KEY_P)
            app->step_by = MAX((app->step_by - 1)/1.3, 1.0);

        if(iters_before != app->step_by)
            LOG_INFO("APP", "Steps per iter %lf", app->step_by);
    }
}

void error_func(void* context, Platform_Sandox_Error error_code)
{
    (void) context;
    const char* msg = platform_sandbox_error_to_string(error_code);
    
    LOG_ERROR("APP", "%s exception occured", msg);
    LOG_TRACE("APP", "printing trace:");
    log_group_push();
    log_callstack("APP", LOG_TYPE_ERROR, -1, 1);
    log_group_pop();
}

void render_cuda_memory(App_State* app, Compute_Texture texture, const f32* cuda_memory, f32 min, f32 max)
{
    struct hipGraphicsResource *cuda_resource = 0; 
            
    //@TODO: the registering and unregistering has a cost!
    //@TODO: fastest would be to write directly into the texture!
    CUDA_TEST(hipGraphicsGLRegisterImage(&cuda_resource, texture.id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_TEST(hipGraphicsMapResources(1, &cuda_resource, 0));
    
    #if 1
        //for texture objects
        hipArray_t mapped_array = {0};
        isize texture_size = texture.width * texture.heigth * texture.format.pixel_size;
        CUDA_TEST(hipGraphicsSubResourceGetMappedArray(&mapped_array, cuda_resource, 0, 0));
        CUDA_TEST(hipMemcpyToArray(mapped_array, 0, 0, cuda_memory, texture_size, hipMemcpyDeviceToDevice));
    #else
        //For gl buffers
        size_t num_bytes = 0;
        unsigned int *mapped_map = NULL;
        CUDA_TEST(hipGraphicsResourceGetMappedPointer((void **)&mapped_map, &num_bytes, cuda_resource));
        ASSERT(num_bytes <= texture_size);
        hipMemcpy(mapped_map, cuda_memory, texture_size, hipMemcpyDeviceToDevice);
    #endif

    CUDA_TEST(hipGraphicsUnmapResources(1, &cuda_resource, 0));
    CUDA_TEST(hipGraphicsUnregisterResource(cuda_resource)); 
    (void) app;
    render_sci_texture(texture, min, max);
}

void allen_cahn_custom_config(Allen_Cahn_Config* out_config)
{
    const i32 _SIZE_X = 1024;
    const i32 _SIZE_Y = _SIZE_X;
    const f32 _dt = 1.0f/200;
    const f32 _alpha = 0.5;
    const f32 _L = 2;
    const f32 _xi = 0.00411f;
    const f32 _a = 2;
    const f32 _b = 1;
    const f32 _beta = 8;
    const f32 _Tm = 1;
    const f32 _Tini = 0;
    const f32 _L0 = 4;

    Allen_Cahn_Scale scale = {0};
    scale.L0 = _L0 / (f32) _SIZE_X;
    scale.Tini = _Tini;
    scale.Tm = _Tm;
    scale.c = 1;
    scale.rho = 1;
    scale.lambda = 1;
    
    Allen_Cahn_Params params = {0};
    params.sym_size = _L0;
    params.mesh_size_x = _SIZE_X;
    params.mesh_size_y = _SIZE_Y;
    params.L = allen_cahn_scale_latent_heat(_L, scale);
    params.xi = allen_cahn_scale_xi(_xi, scale);
    params.dt = _dt;
    params.a = _a;
    params.b = _b;
    params.alpha = allen_cahn_scale_alpha(_alpha, scale);
    params.beta = allen_cahn_scale_latent_heat(_beta, scale);
    params.Tm = _Tm;
    
    Allen_Cahn_Initial_Conditions initial_conditions = {0};
    initial_conditions.inside_phi = 1;
    initial_conditions.inside_T = 0;
    initial_conditions.outside_phi = 0;
    initial_conditions.outside_T = 0;
    initial_conditions.circle_center = vec2(_L0 / 4, _L0 / 4);
    initial_conditions.circle_radius = _L0 / 8;
    initial_conditions.square_from = vec2(_L0/2 - 0.3f, _L0/2 - 0.3f);
    initial_conditions.square_to = vec2(_L0/2 + 0.3f, _L0/2 + 0.3f);

    Allen_Cahn_Snapshots snapshots = {0};
    snapshots.folder = builder_from_cstring("snapshots", NULL);
    snapshots.prefix = builder_from_cstring("v1", NULL);
    snapshots.every = 0.1f;
    snapshots.sym_time = -1;
    
    out_config->config_name = builder_from_cstring("from_code_config", NULL);
    out_config->initial_conditions = initial_conditions;
    out_config->params = params;
    out_config->snapshots = snapshots;
}

#include "lib/platform_windows.c"