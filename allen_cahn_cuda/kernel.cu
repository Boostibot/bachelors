#include "hip/hip_runtime.h"

#define JOT_ALL_IMPL
#define JMAPI __host__ __device__ static inline
#include "lib/platform.h"

#include "config.h"
#include "integration_methods.h"
#include "render.h"
#include "cuprintf.cuh"
#include "cuprintf.cu"
#include "cuda_util.cuh"

#include "lib/log.h"
#include "lib/logger_file.h"
#include "lib/allocator_debug.h"
#include "lib/allocator_malloc.h"
#include "lib/error.h"
#include "lib/time.h"
#include "lib/image.h"
#include "lib/format_netbpm.h"
#include "lib/math.h"

#include "glfw/glfw3.h"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <>

#include <stddef.h>

const i32 SCR_WIDTH = 1000;
const i32 SCR_HEIGHT = 1000;

const f64 FPS_DISPLAY_FREQ = 50000;
const f64 RENDER_FREQ = 30;
const f64 POLL_FREQ = 30;

const f64 FREE_RUN_SYM_FPS = 200;

#define CONFIG_FILE "config.lpf"


typedef struct Simulation_State {
    Allen_Cahn_Config config;
    Allen_Cahn_Config last_config;
    i64 epoch_time;
    Platform_Calendar_Time calendar_time;
    Allocator* alloc;
    
    f32* initial_phi_map;
    f32* initial_T_map;

    f32* phi_map;
    f32* T_map;
    f32* next_phi_map;
    f32* next_T_map;

    f32* display_map1;
    f32* display_map2;
    f32* display_map3;
    
    Compute_Texture output_phi_map = {0};
    Compute_Texture output_T_map = {0};
} Simulation_State;

typedef struct App_State {
    GLFWwindow* window;

    bool is_in_step_mode;
    bool render_phi;
    f64 remaining_steps;
    f64 step_by;

    i64 queued_reloads;
    i64 queued_reloads_last;

    Simulation_State simulation_state;
} App_State;

bool simulation_state_is_hard_reload(const Allen_Cahn_Config* config, const Allen_Cahn_Config* last_config)
{
    if(config->params.mesh_size_x != last_config->params.mesh_size_x || config->params.mesh_size_y != last_config->params.mesh_size_y)
        return true;

    //@HACK: proper comparison of fields is replaced by comparison of the flat memory because I am lazy.
    String_Builder null_sttring_builder = {0};
    Allen_Cahn_Initial_Conditions ini = config->initial_conditions;
    Allen_Cahn_Initial_Conditions ini_last = last_config->initial_conditions;
    ini.start_snapshot = null_sttring_builder;
    ini_last.start_snapshot = null_sttring_builder;

    if(memcmp(&ini, &ini_last, sizeof ini_last) != 0)
        return true;

    if(builder_is_equal(config->initial_conditions.start_snapshot, last_config->initial_conditions.start_snapshot) == false)
        return true;

    return false;
}

void simulation_state_deinit(Simulation_State* state)
{
    (void) state;
}

void allen_cahn_set_initial_conditions(f32* initial_phi_map, f32* initial_T_map, Allen_Cahn_Config config);

void simulation_state_reload(Simulation_State* state, Allen_Cahn_Config* config)
{
    if(state->alloc == NULL)
        state->alloc = allocator_get_default();

    if(config == NULL || simulation_state_is_hard_reload(&state->last_config, config))
    {
        isize old_pixel_count = state->last_config.params.mesh_size_x * state->last_config.params.mesh_size_y;
        allocator_deallocate(state->alloc, state->initial_phi_map, old_pixel_count*sizeof(f32), DEF_ALIGN, SOURCE_INFO());
        allocator_deallocate(state->alloc, state->initial_T_map, old_pixel_count*sizeof(f32), DEF_ALIGN, SOURCE_INFO());

        hipFree(state->phi_map);
        hipFree(state->T_map);
        hipFree(state->next_phi_map);
        hipFree(state->next_T_map);

        compute_texture_deinit(&state->output_phi_map);
        compute_texture_deinit(&state->output_T_map);

        if(config != NULL)
        {
            isize pixel_count_x = config->params.mesh_size_x;
            isize pixel_count_y = config->params.mesh_size_y;
            isize pixel_count = pixel_count_x * pixel_count_y;

            state->initial_phi_map = (f32*) allocator_allocate_cleared(state->alloc, pixel_count * sizeof(f32), DEF_ALIGN, SOURCE_INFO());
            state->initial_T_map = (f32*) allocator_allocate_cleared(state->alloc, pixel_count * sizeof(f32), DEF_ALIGN, SOURCE_INFO());
            allen_cahn_set_initial_conditions(state->initial_phi_map, state->initial_T_map, *config);

            CUDA_TEST(hipMalloc((void**)&state->phi_map,          pixel_count * sizeof(f32)));
            CUDA_TEST(hipMalloc((void**)&state->T_map,            pixel_count * sizeof(f32)));
            CUDA_TEST(hipMalloc((void**)&state->next_phi_map,     pixel_count * sizeof(f32)));
            CUDA_TEST(hipMalloc((void**)&state->next_T_map,       pixel_count * sizeof(f32)));

            CUDA_TEST(hipMemcpy(state->phi_map, state->initial_phi_map, pixel_count * sizeof(f32), hipMemcpyHostToDevice));
            CUDA_TEST(hipMemcpy(state->T_map, state->initial_T_map, pixel_count * sizeof(f32), hipMemcpyHostToDevice));
    
            state->output_phi_map  = compute_texture_make(pixel_count_x, pixel_count_y, PIXEL_FORMAT_F32, 1);
            state->output_T_map    = compute_texture_make(pixel_count_x, pixel_count_y, PIXEL_FORMAT_F32, 1);
        }
    }

    //@TODO: not leak memeory here
    if(state->epoch_time == 0)
        state->epoch_time = platform_local_epoch_time();

    state->calendar_time = platform_epoch_time_to_calendar_time(state->epoch_time);
    if(config)
    {
        state->config = *config;
        state->last_config = *config;
        platform_directory_create(string_from_builder(config->snapshots.folder));
    }
}

void app_state_init(App_State* state, GLFWwindow* window)
{
    state->window = window;
    state->is_in_step_mode = true;
    state->remaining_steps = 0;
    state->step_by = 1;
    state->render_phi = true;
}

JMAPI f32 map_at(const f32* map, int x, int y, Allen_Cahn_Params params)
{
    int x_mod = x % params.mesh_size_x;
    int y_mod = y % params.mesh_size_y;

    return map[x_mod + y_mod*params.mesh_size_x];
}

JMAPI f32 allen_cahn_reaction_term_0(f32 phi)
{
	return phi*(1 - phi)*(phi - 1.0f/2);
}

JMAPI f32 allen_cahn_reaction_term_1(f32 phi, f32 T, f32 xi, Allen_Cahn_Params params)
{
    f32 mK = 1;
	return (params.a*allen_cahn_reaction_term_0(phi) - params.b*params.beta*xi*(T - params.Tm))*mK;
}

JMAPI f32 allen_cahn_reaction_term_2(f32 phi, f32 T, f32 xi, Vec2 grad_phi, Allen_Cahn_Params params)
{
    f32 mK = 1;
	f32 grad_val = vec2_len(grad_phi);
	return (params.a*allen_cahn_reaction_term_0(phi) - params.b*params.beta*xi*xi*grad_val*(T - params.Tm))*mK;
}

__global__ void allen_cahn_simulate(f32* phi_map_next, f32* T_map_next, const f32* phi_map, const f32* T_map, Allen_Cahn_Params params, isize iter)
{
    f32 dx = (f32) params.sym_size / params.mesh_size_x;
    f32 dy = (f32) params.sym_size / params.mesh_size_y;
    f32 mK = dx * dy;
    
    //uniform grid
    f32 tau_x = 1;
    f32 tau_y = 1;
    //f32 tau_x = dy / dx;
    //f32 tau_y = dx / dy;

    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < params.mesh_size_x; x += blockDim.x * gridDim.x) 
    {
        for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < params.mesh_size_x; y += blockDim.y * gridDim.y) 
        {
	        f32 T = map_at(T_map, x, y, params);
	        f32 phi = map_at(phi_map, x, y, params);

	        f32 phi_py = map_at(phi_map, x, y + 1, params);
	        f32 phi_my = map_at(phi_map, x, y - 1, params);
	        f32 phi_px = map_at(phi_map, x + 1, y, params);
	        f32 phi_mx = map_at(phi_map, x - 1, y, params);

	        f32 T_py = map_at(T_map, x, y + 1, params);
	        f32 T_my = map_at(T_map, x, y - 1, params);
	        f32 T_px = map_at(T_map, x + 1, y, params);
	        f32 T_mx = map_at(T_map, x - 1, y, params);

	        f32 sum_phi_neigbours = 0
		        + tau_y*(phi_py - phi)
		        + tau_y*(phi_my - phi)
		        + tau_x*(phi_px - phi)
		        + tau_x*(phi_mx - phi);
		
	        f32 sum_T_neigbours = 0
		        + tau_y*(T_py - T)
		        + tau_y*(T_my - T)
		        + tau_x*(T_px - T)
		        + tau_x*(T_mx - T);

	        Vec2 grad_phi = {
		        (phi_px - phi_mx) * dx / (2 * mK),
		        (phi_py - phi_my) * dy / (2 * mK)
	        };
        
	        f32 reaction_term = allen_cahn_reaction_term_2(phi, T, params.xi, grad_phi, params);
	        f32 phi_dt = (sum_phi_neigbours/mK + reaction_term/(params.xi*params.xi)) / params.alpha;
	        f32 T_dt = sum_T_neigbours / mK + params.L * phi_dt;

	        f32 phi_next = phi_dt * params.dt + phi;
	        f32 T_next = T_dt * params.dt + T;
		
            phi_map_next[x + y*params.mesh_size_x] = phi_next;
            T_map_next[x + y*params.mesh_size_x] = T_next;

            
            phi_map_next[x + y*params.mesh_size_x] = 0;
            T_map_next[x + y*params.mesh_size_x] = 0;
        }
    }
}


__global__ void allen_cahn_simulate_empty(f32* phi_map_next, f32* T_map_next, const f32* phi_map, const f32* T_map, Allen_Cahn_Params params, isize iter)
{
    f32 dx = (f32) params.sym_size / params.mesh_size_x;
    f32 dy = (f32) params.sym_size / params.mesh_size_y;
    f32 mK = dx * dy;
    
    //uniform grid
    f32 tau_x = 1;
    f32 tau_y = 1;
    //f32 tau_x = dy / dx;
    //f32 tau_y = dx / dy;

    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < params.mesh_size_x; x += blockDim.x * gridDim.x) 
    {
        for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < params.mesh_size_x; y += blockDim.y * gridDim.y) 
        {
            phi_map_next[x + y*params.mesh_size_x] = 0;
            T_map_next[x + y*params.mesh_size_x] = 0;
        }
    }
}


Error cuda_error(hipError_t error);
String cuda_translate_error(u32 code, void* context);
void render_cuda_memory(App_State* app, Compute_Texture texture, const f32* cuda_memory, f32 min, f32 max);
void allen_cahn_custom_config(Allen_Cahn_Config* out_config);

bool app_poll_reloads(App_State* state)
{
    if(state->queued_reloads > platform_interlocked_excahnge64(&state->queued_reloads_last, state->queued_reloads))
        return true;
    else
        return false;
}

bool queue_file_reload(void* context)
{
    App_State* state = (App_State*) context;
    platform_interlocked_increment64(&state->queued_reloads);
    return true;
}

void allen_cahn_set_initial_conditions(f32* initial_phi_map, f32* initial_T_map, Allen_Cahn_Config config)
{
    Allen_Cahn_Params params = config.params;
    Allen_Cahn_Initial_Conditions initial = config.initial_conditions;
    for(isize y = 0; y < params.mesh_size_y; y++)
    {
        for(isize x = 0; x < params.mesh_size_x; x++)
        {
            Vec2 pos = vec2((f32) x / params.mesh_size_x * params.sym_size, (f32) y / params.mesh_size_y * params.sym_size); 
            isize i = x + y*params.mesh_size_x;

		    if(((initial.square_from.x <= pos.x && pos.x < initial.square_to.x) && 
			    (initial.square_from.y <= pos.y && pos.y < initial.square_to.y))
			    || vec2_len(vec2_sub(initial.circle_center, pos)) < initial.circle_radius)
		    {
                initial_phi_map[i] = initial.inside_phi;
                initial_T_map[i] = initial.inside_T;
		    }
            else
            {
                initial_phi_map[i] = initial.outside_phi;
                initial_T_map[i] = initial.outside_T;
            }
        }
    }
}
EXPORT MODIFIER_FORMAT_FUNC(format, 3) void log_message(const char* module, Log_Type type, Source_Info source, MODIFIER_FORMAT_ARG const char* format, ...);
void run_func_allen_cahn_cuda(void* context)
{
    LOG_INFO("App", "current working dir: '%s'", platform_directory_get_current_working());

    GLFWwindow* window = (GLFWwindow*) context;
    App_State* app = (App_State*) glfwGetWindowUserPointer(window); (void) app;
    Simulation_State* simualtion = &app->simulation_state;
    
    queue_file_reload(app);
    cudaPrintfInit();

    Platform_File_Watch file_watch = {0};
    Error watch_error = error_from_platform(platform_file_watch(&file_watch, STRING("."), PLATFORM_FILE_WATCH_CHANGE, queue_file_reload, app));
    ASSERT_MSG(error_is_ok(watch_error), "file watch failed %s", error_code(watch_error));

    int device_id = 0;
    CUDA_TEST(hipSetDevice(device_id));
    
    int device_processor_count = 0;
    hipDeviceGetAttribute(&device_processor_count, hipDeviceAttributeMultiprocessorCount, device_id);

    //compare_rk4();
    //String_Builder serialized_image = {0};

    i64 save_counter = 0;
    i64 frame_counter = 0;
    f64 frame_time_sum = 0;
    
    f64 fps_display_last_time_sum = 0;
    f64 fps_display_last_time = 0;
    i64 fps_display_last_frames = 0;
    
    f64 poll_last_time = 0;
    f64 render_last_time = 0;
    f64 simulated_last_time = 0;

    f64 simulation_time_sum = 0;

	while (!glfwWindowShouldClose(window))
    {
        f64 now = clock_s();

        if(app_poll_reloads(app))
        {
            Allen_Cahn_Config config = {0};
            TEST(allen_cahn_read_file_config(&config, CONFIG_FILE));
            
            if(config.snapshots.folder.size > 0)
                platform_directory_create(string_from_builder(config.snapshots.folder));

            simulation_state_reload(simualtion, &config);
        }

        //if(0)
        if(now - render_last_time > 1.0/RENDER_FREQ)
        {
            PERF_COUNTER_START(render);
            render_last_time = now;
            if(app->render_phi)
                render_cuda_memory(app, simualtion->output_phi_map, simualtion->phi_map, 0, 1);
            else
                render_cuda_memory(app, simualtion->output_T_map, simualtion->T_map, 0, 1.5);
                
            glfwSwapBuffers(app->window);
            PERF_COUNTER_END(render);
        }

        if(now - fps_display_last_time > 1.0/FPS_DISPLAY_FREQ)
        {
            f64 time_sum_delta = frame_time_sum - fps_display_last_time_sum;
            f64 counter_delta = (f64) (frame_counter - fps_display_last_frames);
            f64 avg_fps = 0;
            if(time_sum_delta != 0)
            {
                avg_fps = counter_delta / time_sum_delta;
                glfwSetWindowTitle(window, format_ephemeral("iter %lli", (lli) frame_counter).data);
            }

            fps_display_last_time = now;
            fps_display_last_frames = frame_counter;
            fps_display_last_time_sum = frame_time_sum;
        }


        bool step_sym = false;
        if(app->is_in_step_mode)
            step_sym = app->remaining_steps > 0.5;
        else
            step_sym = now - simulated_last_time > 1.0/app->step_by/FREE_RUN_SYM_FPS;

        if(step_sym)
        {
            PERF_COUNTER_START(simulation_step);

            simulated_last_time = now;
            app->remaining_steps -= 1;

            f64 frame_start_time = clock_s();

            dim3 bs(64, 1);
            dim3 grid(device_processor_count, 1);
            allen_cahn_simulate<<<grid, bs>>>(simualtion->next_phi_map, simualtion->next_T_map, simualtion->phi_map, simualtion->T_map, simualtion->config.params, frame_counter);
            CUDA_TEST(hipGetLastError());
            CUDA_TEST(hipDeviceSynchronize());
            PERF_COUNTER_START(cudaPrintfDisplay_c);
            cudaPrintfDisplay();
            PERF_COUNTER_END(cudaPrintfDisplay_c);
            
            f64 end_start_time = clock_s();

            f64 delta = end_start_time - frame_start_time;
            
            #if 0
            if(SAVE_EVERY > 0 && frame_counter % SAVE_EVERY == 0)
            {
                PERF_COUNTER_START(image_saving);
                Image_Builder pixels = {0};
                String file_name = {0};
                image_builder_init(&pixels, NULL, 1, PIXEL_FORMAT_U8);

                {
                    compute_texture_get_pixels_converted(&pixels, next_phi_map);
                    netbpm_format_pgm_write_into(&serialized_image, image_from_builder(pixels));
                
                    file_name = format_ephemeral("%s/%s_%lld-%lld-%lld_%lld-%lld-%lld_iter_%lld_phi.pgm", SAVE_FOLDER, SAVE_PREFIX, 
                        (lli) calendar_time.year, (lli) calendar_time.month, (lli) calendar_time.day, 
                        (lli) calendar_time.hour, (lli) calendar_time.minute, (lli) calendar_time.second, 
                        (lli) frame_counter);

                    file_write_entire(file_name, string_from_builder(serialized_image));
                }

                {
                    compute_texture_get_pixels_converted(&pixels, next_T_map);
                    netbpm_format_pgm_write_into(&serialized_image, image_from_builder(pixels));
                
                    file_name = format_ephemeral("%s/%s_%lld-%lld-%lld_%lld-%lld-%lld_iter_%lld_T.pgm", SAVE_FOLDER, SAVE_PREFIX, 
                        (lli) calendar_time.year, (lli) calendar_time.month, (lli) calendar_time.day, 
                        (lli) calendar_time.hour, (lli) calendar_time.minute, (lli) calendar_time.second, 
                        (lli) frame_counter);

                    file_write_entire(file_name, string_from_builder(serialized_image));
                }

                image_builder_deinit(&pixels);
                PERF_COUNTER_END(image_saving);
                
                save_counter ++;
                if(save_counter > PAUSE_AFTER_SAVES)
                {
                    save_counter = 0;
                    app->is_in_step_mode = true;
                    app->remaining_steps = 0;
                }
            }
            #endif

            frame_time_sum += delta;
            frame_counter += 1;
            simulation_time_sum += simualtion->config.params.dt;

            SWAP(&simualtion->phi_map, &simualtion->next_phi_map, f32*);
            SWAP(&simualtion->T_map, &simualtion->next_T_map, f32*);
            
            PERF_COUNTER_END(simulation_step);
        }
        
        if(now - poll_last_time > 1.0/POLL_FREQ)
        {
            PERF_COUNTER_START(poll_evennts);
		    glfwPollEvents();
            PERF_COUNTER_END(poll_evennts);
        }
    }

    platform_file_unwatch(&file_watch);
}

void* glfw_malloc_func(size_t size, void* user);
void* glfw_realloc_func(void* block, size_t size, void* user);
void glfw_free_func(void* block, void* user);
void glfw_error_func(int code, const char* description);
void glfw_resize_func(GLFWwindow* window, int width, int heigth);
void glfw_key_func(GLFWwindow* window, int key, int scancode, int action, int mods);

void run_func(void* context);
void error_func(void* context, Platform_Sandbox_Error error_code);

void platform_test_func()
{
    Platform_Directory_Entry* entries = NULL;
    isize entries_count = 0;
    platform_directory_list_contents_alloc(STRING("."), &entries, &entries_count, 3);

    platform_file_info(STRING("temp.h"), NULL);
    platform_file_info(STRING("main.h"), NULL);
    platform_file_info(STRING("config.h"), NULL);
    platform_file_info(STRING("temp.h"), NULL);

    LOG_INFO("platform", "executable path:     '%s'", platform_get_executable_path());
    LOG_INFO("platform", "current working dir: '%s'", platform_directory_get_current_working());

    String_Builder dir_padding = {0};

    //String_Builder complete_list = {0};

    for(isize i = 0; i < entries_count; i++)
    {
        Platform_Directory_Entry entry = entries[i];
        array_clear(&dir_padding);
        for(isize j = 0; j < entry.directory_depth; j++)
            builder_append(&dir_padding, STRING("  "));

        //builder_append(&complete_list)
        LOG_INFO("dirs", "%lli %s%s", i, cstring_from_builder(dir_padding), entry.path);
    }

    //LOG_INFO("dirs", complete_list.data);
    array_deinit(&dir_padding);
    platform_directory_list_contents_free(entries);
}


int main()
{
    platform_init();
    Malloc_Allocator static_allocator = {0};
    malloc_allocator_init(&static_allocator);
    allocator_set_static(&static_allocator.allocator);
    
    Malloc_Allocator malloc_allocator = {0};
    malloc_allocator_init_use(&malloc_allocator, 0);
    
    Debug_Allocator debug_alloc = {0};
    debug_allocator_init_use(&debug_alloc, &malloc_allocator.allocator, DEBUG_ALLOCATOR_DEINIT_LEAK_CHECK | DEBUG_ALLOCATOR_CAPTURE_CALLSTACK);

    error_system_init(&static_allocator.allocator);
    file_logger_init_use(&global_logger, &malloc_allocator.allocator, &malloc_allocator.allocator);

    GLFWallocator allocator = {0};
    allocator.allocate = glfw_malloc_func;
    allocator.reallocate = glfw_realloc_func;
    allocator.deallocate = glfw_free_func;
    allocator.user = &malloc_allocator;
 
    glfwInitAllocator(&allocator);
    glfwSetErrorCallback(glfw_error_func);
    TEST_MSG(glfwInit(), "Failed to init glfw");

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_DEBUG_CONTEXT, true);  
 
    GLFWmonitor* monitor = glfwGetPrimaryMonitor();
    const GLFWvidmode* mode = glfwGetVideoMode(monitor);
    ASSERT(monitor && mode);
    if(monitor != NULL && mode != NULL)
    {
        glfwWindowHint(GLFW_RED_BITS, mode->redBits);
        glfwWindowHint(GLFW_GREEN_BITS, mode->greenBits);
        glfwWindowHint(GLFW_BLUE_BITS, mode->blueBits);
        glfwWindowHint(GLFW_REFRESH_RATE, mode->refreshRate);
    }
 
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "Render", NULL, NULL);
    TEST_MSG(window != NULL, "Failed to make glfw window");

    App_State app = {0};
    app_state_init(&app, window);
    glfwSetWindowUserPointer(window, &app);
    glfwMakeContextCurrent(window);
	glfwSetFramebufferSizeCallback(window, glfw_resize_func);
    glfwSetKeyCallback(window, glfw_key_func);
    glfwSwapInterval(0);

    int version = gladLoadGL((GLADloadfunc) glfwGetProcAddress);
    TEST_MSG(version != 0, "Failed to load opengl with glad");

    gl_debug_output_enable();

    platform_exception_sandbox(
        run_func_allen_cahn_cuda, window, 
        error_func, window);

    glfwDestroyWindow(window);
    glfwTerminate();

    debug_allocator_deinit(&debug_alloc);
    
    file_logger_deinit(&global_logger);
    error_system_deinit();

    ASSERT(malloc_allocator.bytes_allocated == 0);
    malloc_allocator_deinit(&malloc_allocator);
    platform_deinit();

    return 0;    
}

void* glfw_malloc_func(size_t size, void* user)
{
    return malloc_allocator_malloc((Malloc_Allocator*) user, size);
}

void* glfw_realloc_func(void* block, size_t size, void* user)
{
    return malloc_allocator_realloc((Malloc_Allocator*) user, block, size);
}

void glfw_free_func(void* block, void* user)
{
    malloc_allocator_free((Malloc_Allocator*) user, block);
}

void glfw_error_func(int code, const char* description)
{
    LOG_ERROR("APP", "GLWF error %d with message: %s", code, description);
}
void glfw_resize_func(GLFWwindow* window, int width, int heigth)
{
    (void) window;
	// make sure the viewport matches the new window dimensions; note that width and 
	// heigth will be significantly larger than specified on retina displays.
	glViewport(0, 0, width, heigth);
}

void glfw_key_func(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    (void) mods;
    (void) scancode;
    (void) window;
    App_State* app = (App_State*) glfwGetWindowUserPointer(window); (void) app;
    if(action == GLFW_RELEASE)
    {
        if(key == GLFW_KEY_ENTER)
            app->remaining_steps = app->step_by;
        if(key == GLFW_KEY_SPACE)
            app->is_in_step_mode = !app->is_in_step_mode;
        if(key == GLFW_KEY_F1)
        {
            app->render_phi = !app->render_phi;
            LOG_INFO("APP", "Rendering %s", app->render_phi ? "phi" : "T");
        }

        if(key == GLFW_KEY_C)
        {
            for(Global_Perf_Counter* counter = profile_get_counters(); counter != NULL; counter = counter->next)
            {
                Perf_Counter_Stats stats = perf_counter_get_stats(counter->counter, 1);
		        LOG_INFO("APP", "total: %15.8lf avg: %12.8lf runs: %-8lli σ/μ %13.6lf [%13.6lf %13.6lf] (ms) from %-4lli %s \"%s\"", 
			        stats.total_s*1000,
			        stats.average_s*1000,
                    (lli) stats.runs,
                    stats.normalized_standard_deviation_s,
			        stats.min_s*1000,
			        stats.max_s*1000,
			        (lli) counter->line,
			        counter->function,
			        counter->name
		        );
            }
        }
        
        f64 iters_before = app->step_by;
        if(key == GLFW_KEY_O)
            app->step_by = app->step_by*1.3 + 1;
        if(key == GLFW_KEY_P)
            app->step_by = MAX((app->step_by - 1)/1.3, 1.0);

        if(iters_before != app->step_by)
            LOG_INFO("APP", "Steps per iter %lf", app->step_by);
    }
}

void error_func(void* context, Platform_Sandbox_Error error)
{
    (void) context;
    const char* msg = platform_exception_to_string(error.exception);
    
    LOG_ERROR("APP", "%s exception occured", msg);
    LOG_TRACE("APP", "printing trace:");
    log_group_push();
    log_translated_callstack("APP", LOG_TYPE_TRACE, error.call_stack, error.call_stack_size);
    log_group_pop();
}

void render_cuda_memory(App_State* app, Compute_Texture texture, const f32* cuda_memory, f32 min, f32 max)
{
    struct hipGraphicsResource *cuda_resource = 0; 
            
    //@TODO: the registering and unregistering has a cost!
    //@TODO: fastest would be to write directly into the texture!
    CUDA_TEST(hipGraphicsGLRegisterImage(&cuda_resource, texture.id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_TEST(hipGraphicsMapResources(1, &cuda_resource, 0));
    
    #if 1
        //for texture objects
        hipArray_t mapped_array = {0};
        isize texture_size = texture.width * texture.heigth * texture.format.pixel_size;
        CUDA_TEST(hipGraphicsSubResourceGetMappedArray(&mapped_array, cuda_resource, 0, 0));
        CUDA_TEST(hipMemcpyToArray(mapped_array, 0, 0, cuda_memory, texture_size, hipMemcpyDeviceToDevice));
    #else
        //For gl buffers
        size_t num_bytes = 0;
        unsigned int *mapped_map = NULL;
        CUDA_TEST(hipGraphicsResourceGetMappedPointer((void **)&mapped_map, &num_bytes, cuda_resource));
        ASSERT(num_bytes <= texture_size);
        hipMemcpy(mapped_map, cuda_memory, texture_size, hipMemcpyDeviceToDevice);
    #endif

    CUDA_TEST(hipGraphicsUnmapResources(1, &cuda_resource, 0));
    CUDA_TEST(hipGraphicsUnregisterResource(cuda_resource)); 
    (void) app;
    render_sci_texture(texture, min, max);
}

void allen_cahn_custom_config(Allen_Cahn_Config* out_config)
{
    const i32 _SIZE_X = 1024;
    const i32 _SIZE_Y = _SIZE_X;
    const f32 _dt = 1.0f/200;
    const f32 _alpha = 0.5;
    const f32 _L = 2;
    const f32 _xi = 0.00411f;
    const f32 _a = 2;
    const f32 _b = 1;
    const f32 _beta = 8;
    const f32 _Tm = 1;
    const f32 _Tini = 0;
    const f32 _L0 = 4;

    Allen_Cahn_Scale scale = {0};
    scale.L0 = _L0 / (f32) _SIZE_X;
    scale.Tini = _Tini;
    scale.Tm = _Tm;
    scale.c = 1;
    scale.rho = 1;
    scale.lambda = 1;
    
    Allen_Cahn_Params params = {0};
    params.sym_size = _L0;
    params.mesh_size_x = _SIZE_X;
    params.mesh_size_y = _SIZE_Y;
    params.L = allen_cahn_scale_latent_heat(_L, scale);
    params.xi = allen_cahn_scale_xi(_xi, scale);
    params.dt = _dt;
    params.a = _a;
    params.b = _b;
    params.alpha = allen_cahn_scale_alpha(_alpha, scale);
    params.beta = allen_cahn_scale_latent_heat(_beta, scale);
    params.Tm = _Tm;
    
    Allen_Cahn_Initial_Conditions initial_conditions = {0};
    initial_conditions.inside_phi = 1;
    initial_conditions.inside_T = 0;
    initial_conditions.outside_phi = 0;
    initial_conditions.outside_T = 0;
    initial_conditions.circle_center = vec2(_L0 / 4, _L0 / 4);
    initial_conditions.circle_radius = _L0 / 8;
    initial_conditions.square_from = vec2(_L0/2 - 0.3f, _L0/2 - 0.3f);
    initial_conditions.square_to = vec2(_L0/2 + 0.3f, _L0/2 + 0.3f);

    Allen_Cahn_Snapshots snapshots = {0};
    snapshots.folder = builder_from_cstring("snapshots", NULL);
    snapshots.prefix = builder_from_cstring("v1", NULL);
    snapshots.every = 0.1f;
    snapshots.sym_time = -1;
    
    out_config->config_name = builder_from_cstring("from_code_config", NULL);
    out_config->initial_conditions = initial_conditions;
    out_config->params = params;
    out_config->snapshots = snapshots;
}

#if 0
    #if PLATFORM_OS == PLATFORM_OS_WINDOWS
        #include "lib/platform_windows.c"
    #elif PLATFORM_OS == PLATFORM_OS_UNIX
        #include "lib/platform_linux.c"
    #else
        #error Provide support for this operating system or define PLATFORM_OS to one of the values in platform.h
    #endif
#endif